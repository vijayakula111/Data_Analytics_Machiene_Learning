#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/


#include "../NativeOps.h"
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

#include <buffer.h>
#include <helpers/shape.h>
#include "../Environment.h"
#include <helpers/TAD.h>

#include <ops/specials.h>
#include <loops/reduce3.h>

#include <loops/indexreduce.h>
#include <loops/summarystatsreduce.h>
#include <loops/random.h>


#include <loops/broadcasting.h>
#include <loops/broadcasting_bool.h>

#include <loops/scalar.h>
#include <loops/scalar_bool.h>

#include <loops/pairwise_transform.h>
#include <loops/pairwise_bool.h>

#include <loops/transform_same.h>
#include <loops/transform_float.h>
#include <loops/transform_strict.h>
#include <loops/transform_bool.h>
#include <loops/transform_any.h>

#include <loops/reduce_float.h>
#include <loops/reduce_same.h>
#include <loops/reduce_bool.h>
#include <loops/reduce_long.h>

//#include <thread>
#include <map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <pointercast.h>
#include <stdio.h>
#include <stdlib.h>
#include <loops/type_conversions.h>
#include <op_boilerplate.h>
#include <loops/aggregates.h>
#include <helpers/threshold.h>
#include <ShapeList.h>
#include <Context.h>
#include <ops/specials_cuda.h>

#include <graph/exceptions/datatype_exception.h>

#include <helpers/CudaLaunchHelper.h>

// FIXME: we need cuda-specific implementations
#include <helpers/logger.h>
#include <NDArray.h>
#include <GraphExecutioner.h>
#include <graph/GraphHolder.h>
#include <graph/VariablesSet.h>
#include <ops/declarable/OpRegistrator.h>
#include <ops/declarable/CustomOperations.h>



//#include <sys/time.h>

#include <hiprand.h>
#include <Status.h>
#include <helpers/DebugHelper.h>

using namespace nd4j;

#include <loops/special_kernels.h>

hipDeviceProp_t *deviceProperties;
hipFuncAttributes *funcAttributes = new hipFuncAttributes[64];
int blockLimit = 128;
int maxThreads = 512;
bool allowedP2P = false;
bool supportedP2P = false;
#ifdef __ND4J_EXPERIMENTAL__
bool experimentalSupport = true;
#else
bool experimentalSupport = false;
#endif

int minThreads = 32;

__constant__ char deviceConstantMemory[49152];

typedef struct {
    long streamId;
    long callId;
} __syncInfo;

typedef __syncInfo SyncInfo;


/**
* This is utility kernel, that updates given special buffer with proper values in device memory
*/
extern "C" __global__ void prepareShapeBuffer(int *dimension, int *maxDimension, Nd4jLong *specialPointer, int rows, nd4j::DataType dataType) {
    Nd4jLong tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > 0)
        return;

    dimension[0] = 0;
    maxDimension[0] = 1;

    specialPointer[0] = 2;
    specialPointer[1] = rows;
    specialPointer[2] = 1;
    specialPointer[3] = 1;
    specialPointer[4] = 1;
    specialPointer[5] = 0;
    specialPointer[6] = 1;
    specialPointer[7] = 99;

    ArrayOptions::setDataType(specialPointer, dataType);

    //printf("special[0]: [%lld]\n", (long long) specialPointer[0]);
    //shape::printShapeInfoLinear("prepareShapeBuffer", specialPointer);
}


// this method isn't used, left here for legacy and caution purposes
// TLDR: don't use this way, it sucks
void CUDART_CB syncCallback(hipStream_t stream, hipError_t status, void *data){
    SyncInfo *sync = reinterpret_cast<SyncInfo *>(data);

    //printf("Finished stream: [%i], kernel call: [%i]\n", sync->streamId, sync->callId);
}

// this method just does type conversion in fancy way
int getDeviceId(Nd4jPointer ptrToDeviceId) {
    return (int)(Nd4jLong)ptrToDeviceId;
}

template <typename T>
dim3 getOptimalDimensions(Nd4jLong n,hipFuncAttributes attributes, hipDeviceProp_t properties) {

	// we can combine the two to compute a block size
	int num_threads = block_size_with_maximum_potential_occupancy(attributes, properties);

	// no real sense launching more threads, then number of elements we have
	if (num_threads > n) num_threads = n;

	if (maxThreads > 0 && num_threads > maxThreads) num_threads = maxThreads;

	// compute the number of blocks of size num_threads to launch
	int num_blocks = n / num_threads;

	// check for partial block at the end

	if (num_blocks > blockLimit) num_blocks = blockLimit;

	if (num_blocks < 4 && n > 128) {
		num_blocks = 4;
		num_threads = n / num_blocks;
	}

	if (num_threads >= 768) {
		num_blocks = num_blocks * 2;
		num_threads = num_threads / 2;
	}

	if(n % num_threads && num_blocks < blockLimit) ++num_blocks;
    //(num_threads * sizeof(T)) + attributes.sharedSizeBytes);
	return dim3(num_blocks,num_threads, 3000);
}

int getBaseMemorySize(int xRank, hipFuncAttributes funcAttr) {
	int memory_limit = 256; //funcAttr.sharedSizeBytes;

	// TODO: remove this later
	memory_limit += sizeof(UnifiedSharedMemory) + 32; // sizeof(shape::TAD) + (xRank * 4 * 4)
/*
	if (xRank == 0) xRank = 2;

	memory_limit += (xRank * 2 + 4) * 3 * 4; // we reserve memory for xShape + T1/T2 shapes
	memory_limit += yRank == 0 ? 0 : (yRank * 2 + 4) * 4;
	memory_limit += zRank == 0 ? 0 : (zRank * 2 + 4) * 4;
	memory_limit += (xRank * 4) * 6;
	memory_limit += MAX_RANK * 4; // special case, needed roughtly in one pase
*/
	return memory_limit;
}

/*
 * Basic CUDA constants here: number of blocks per MP
 */
int getDeviceBlockThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	int blockThreshold = 8;

	if (ccMajor >= 5)
		blockThreshold = 32;
	else if (ccMajor == 3)
		blockThreshold = 16;
	else if (ccMajor < 3)
		blockThreshold = 8;

	return blockThreshold;
}

dim3 getBasicLaunchParams(int deviceId, long problemLength, int sharedMemoryPerThread, hipFuncAttributes funcAttr) {
	int countMP = deviceProperties[deviceId].multiProcessorCount;
	int blockThreshold = getDeviceBlockThreshold(deviceId);

	int num_threads = problemLength / (countMP * blockThreshold);
    num_threads = nd4j::math::nd4j_min<int>(num_threads, maxThreads);
    num_threads = nd4j::math::nd4j_max<int>(num_threads, 64);
    num_threads = nd4j::math::nd4j_max<int>(num_threads, minThreads);

	int num_blocks = nd4j::math::nd4j_max<int>(problemLength / num_threads, 1);
    num_blocks = nd4j::math::nd4j_min<int>(num_blocks, blockLimit);

	int memory_limit = (sharedMemoryPerThread * num_threads) + getBaseMemorySize(1, funcAttr);

	dim3 launchDims = dim3(num_blocks, num_threads, memory_limit);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("Preliminary basic launch params: gridSize: [%i], blockSize: [%i], base shmem: [%i]\n", num_blocks, num_threads, memory_limit);


	return launchDims;
}

/*
 * This message returns shared memory threshold value. default overflow ratio is 0.3
 */
int getDeviceSharedThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	// please note threshold isn't multiple of 32, and that's NOT a mistake

	int shmemThreshold;
	if (ccMajor == 6 && ccMinor == 0)
		shmemThreshold = 65536;
	else if (ccMajor == 6 && ccMinor == 1)
		shmemThreshold = 49152;
	else if (ccMajor == 5 && ccMinor == 2)
		shmemThreshold = 98304;
	else if (ccMajor == 5)
		shmemThreshold = 65536;
	else if (ccMajor == 3 && ccMinor == 7)
		shmemThreshold = 114688;
	else shmemThreshold = 49152;

	return shmemThreshold / 0.3;
}


dim3 getBetterDimensions(int deviceId, int numTads, int tadLength, int xRank, hipFuncAttributes funcAttr, int dimensionLength, int elementSize, int reduction) {

	int num_threads = nd4j::math::nd4j_min<int>(tadLength, maxThreads);



	int countMP = deviceProperties[deviceId].multiProcessorCount;
	int regPerBlock = deviceProperties[deviceId].regsPerBlock;
	int warpSize = deviceProperties[deviceId].warpSize;

	int blockThreshold = getDeviceBlockThreshold(deviceId);
	int shmemThreshold = getDeviceSharedThreshold(deviceId);

	// round num_threads to nearest warpSize
	num_threads -= num_threads % warpSize;

	num_threads = nd4j::math::nd4j_max<int>(1, num_threads);
    if (num_threads < warpSize && tadLength < warpSize)
        num_threads = tadLength;

	// since we use shared memory as fast memory for some cases - we need to count that in
	int memory_limit = getBaseMemorySize(xRank, funcAttr);
	int memory_floor = memory_limit;
	int effective_block_limit =  countMP * blockThreshold;

	int num_blocks =  numTads; //nd4j::math::nd4j_min<int>(numTads, effective_block_limit);

	int desiredShared = shmemThreshold / nd4j::math::nd4j_max<int>((num_blocks / countMP), 1);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("Launch context: numBlocks: [%i], numThreads: [%i], countMap: [%i], shmemThreshold: [%i], desiredShared: [%i], elementSize: [%i]\n", num_blocks, num_threads, countMP, shmemThreshold, desiredShared, elementSize);

	// at this moment we've stored all required information for things. time to count in reduction multipliers
	int reduction_per_block = 0;
	bool found = false;
	if (reduction > 0)
		while (!found) {
			reduction_per_block = (num_threads * elementSize * reduction);
			if (memory_limit + reduction_per_block < desiredShared) {
				memory_limit += reduction_per_block;
				found = true;
			} else {
				if (num_threads > minThreads) {
					num_threads -= 32;
				} else {
					memory_limit += reduction_per_block;
					found = true;
				}
			}
		}

	// at this moment we know total memory used per block, and we also know per-mp limit.
	int max_active_blocks = shmemThreshold / nd4j::math::nd4j_max<int>(memory_limit, 1);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("MAB: [%i], memory_floor: [%i], memory_limit: [%i], reductionPerBlock: [%i]\n", max_active_blocks, memory_floor, memory_limit, reduction_per_block);

	// we don't want to spawn more blocks, that gpu can actually handle without queue

	//num_blocks = nd4j::math::nd4j_min<int>(num_blocks, max_active_blocks);
	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, blockLimit);

//	if (num_blocks > countMP)
//    	num_blocks = num_blocks - (num_blocks % countMP);

	num_blocks = nd4j::math::nd4j_max<int>(num_blocks, 1);

	int targetBlocksPerMP = num_blocks / countMP;

	// now we know desired number of blocks wrt to shared memory. So, now we should take in account number of threads per SM
	if (targetBlocksPerMP * num_threads > 2048) {
		while (targetBlocksPerMP * num_threads > 2048) {
			if (num_threads <= minThreads)
				break;

			num_threads -= 32;
		}

		reduction_per_block = (num_threads * elementSize * reduction);
		memory_limit = memory_floor + reduction_per_block;
	}




	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("Preliminary reduce launch params: gridSize: [%i], blockSize: [%i], base shmem: [%i], reduction_per_block: [%i], blocksPerMP: [%i]\n", num_blocks, num_threads, memory_limit, reduction_per_block, targetBlocksPerMP);

	return dim3(num_blocks,num_threads, memory_limit);
}

/*
 * This method returns kernel launch param for linear memory access
 */
dim3 getFlatLaunchParams(int deviceId, Nd4jLong *dXShapeInfo, Nd4jLong *dYShapeInfo, hipFuncAttributes funcAttr) {
	auto xRank = shape::rank(dXShapeInfo);
	auto yRank = dYShapeInfo == nullptr ? 0 : shape::rank(dYShapeInfo);
	auto zRank = 0;

	int memory_limit = getBaseMemorySize(xRank, funcAttr);

	int countMP = deviceProperties[deviceId].multiProcessorCount;
	int regPerBlock = deviceProperties[deviceId].regsPerBlock;

	int blockThreshold = getDeviceBlockThreshold(deviceId);
	int shmemThreshold = getDeviceSharedThreshold(deviceId);

	auto xLength = shape::length(dXShapeInfo);
	int effective_block_limit =  countMP * blockThreshold;

	// for flat calls we just want as much concurrent blocks, as possible, and we're not tied to TAD here
	int num_threads = xLength / effective_block_limit;
	if (num_threads < minThreads)
		num_threads = minThreads;

	num_threads = num_threads - (num_threads % 32);

	int memory_floor = memory_limit;

	int num_blocks = xLength / num_threads;
	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, blockLimit);
//	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, effective_block_limit);
	num_blocks = nd4j::math::nd4j_max<int>(num_blocks, 1);

	int targetBlocksPerMP = num_blocks / countMP;

	// now we know desired number of blocks wrt to shared memory. So, now we should take in account number of threads per SM
	if (targetBlocksPerMP * num_threads > 2048 && num_threads >= 128) {
		while (targetBlocksPerMP * num_threads > 2048) {
			if (num_threads <= minThreads)
				break;
			num_threads -= 32;
		}
	}

    if (xLength / num_threads > blockLimit)
        num_blocks *= 2;

	dim3 launchDims = dim3(num_blocks, num_threads, memory_limit);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("Preliminary scalar launch params: gridSize: [%i], blockSize: [%i], base shmem: [%i], blocksPerMP: [%i], problemLength: [%i], effectiveBlockLimit: [%i]\n", num_blocks, num_threads, memory_limit, targetBlocksPerMP, xLength, effective_block_limit);


	return launchDims;
}

/**
 * This method returns kernel launch params with TAD-based memory access
 *
 * @param deviceId
 * @param dXShapeInfo
 * @param tadShapeInfo
 * @param funcAttr
 * @param dimensionLength
 * @param elementSize
 * @param reductionSize
 * @return
 */
dim3 getReduceLaunchParams(int deviceId, Nd4jLong *dXShapeInfo, Nd4jLong *tadShapeInfo, hipFuncAttributes funcAttr, int dimensionLength, int elementSize, int reductionSize) {

	Nd4jLong tadLength = 0;
	Nd4jLong numTads = 0;
	if (tadShapeInfo != nullptr) {
		tadLength = shape::length(tadShapeInfo);
		numTads = shape::length(dXShapeInfo) / tadLength;

		if (tadLength == 1) {
			if (nd4j::Environment::getInstance()->isDebugAndVerbose())
				printf("A xLength: [%i], zLength: [%i]\n", shape::length(dXShapeInfo), shape::length(tadShapeInfo));
		}
	} else{
		// we have special case - reduction along all dimensions
		tadLength = nd4j::math::nd4j_min<int>(shape::length(dXShapeInfo), 768);
		numTads = shape::length(dXShapeInfo) / tadLength;
	}

	auto xRank = shape::rank(dXShapeInfo);
	int zRank = tadShapeInfo == nullptr ? 0 : shape::rank(tadShapeInfo);

	dim3 launchDims = getBetterDimensions(deviceId, numTads, tadLength, xRank, funcAttr, dimensionLength, elementSize, reductionSize);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose()) { //|| launchDims.dX == 1
		printf("Reduce LaunchParams: xLength: [%i], numTads: [%i], tadLength: [%i], launchDims.dX: [%i], launchDims.dY: [%i], launchDims.dZ: [%i]\n", shape::length(dXShapeInfo), numTads, tadLength, launchDims.x, launchDims.y, launchDims.z);
	}

	return launchDims;
}

/**
 * Returns optimal launch parameters
 * given the extra pointers passed in.
 * The extra pointer should be
 * the host pointer for the shape information
 * associated with the data.
 * From there it is used to obtain the length
 * from which we can derive the optimal launch parameters.
 *
 */
template <typename T>
dim3 getOptimalLaunchParameters(const Nd4jLong *hXShapeInfo, hipFuncAttributes attributes, hipDeviceProp_t properties) {
	
	auto n = shape::length(hXShapeInfo);

	dim3 launchDims = getOptimalDimensions<T>(n,attributes, properties);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("Params: gridSize: [%i], blockSize: [%i], shMem: [%i], problemLength: [%i], totalThreads:[%i]\n", launchDims.x, launchDims.y, launchDims.z, n, (launchDims.x * launchDims.y));

	return launchDims;
}

nd4j::buffer::Buffer<Nd4jLong> * createScalarBuffer(hipStream_t stream) {
	Nd4jLong *scalarShapeInfo = shape::createScalarShapeInfo();
	nd4j::buffer::Buffer<Nd4jLong> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2), stream);
	nd4j::buffer::copyDataToGpu(&buff, stream);
	return buff;
}


class ScalarShapeInformation {
private:
	nd4j::buffer::Buffer<Nd4jLong> *scalarDimension;
	nd4j::buffer::Buffer<Nd4jLong> *scalarShapeInfo;
//	std::thread::id threadId;

public:
	ScalarShapeInformation(hipStream_t stream) {
		auto scalarDimensionBuff = reinterpret_cast<Nd4jLong *>(malloc(sizeof(Nd4jLong)));

		CHECK_ALLOC(scalarDimensionBuff, "Failed to allocate ShapeInfoBuffer");	

		scalarDimensionBuff[0] = MAX_DIMENSION;
		scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1, stream);
		scalarShapeInfo = createScalarBuffer(stream);
//		threadId = std::this_thread::get_id();

	}
	~ScalarShapeInformation() {
		nd4j::buffer::freeBuffer(&scalarShapeInfo);
		nd4j::buffer::freeBuffer(&scalarDimension);
	}


	Nd4jLong *getShapeInfoHostPointer() {
		return scalarShapeInfo->data;
	}

	Nd4jLong * getShapeInfoGpuPointer() {
		return scalarShapeInfo->gData;
	}

	Nd4jLong * getDimensionHostPointer() {
		return scalarDimension->data;
	}

	Nd4jLong  * getDimensionGpuPointer() {
		return scalarDimension->gData;
	}

};





template <typename T>
class ScalarInfo {
	nd4j::buffer::Buffer<T> *scalarData;
	ScalarShapeInformation *shapeInfo;
	T finalResult;
	hipStream_t streamRef;
public:
	ScalarInfo(hipStream_t stream) {
		T *scalarResult = reinterpret_cast<T*>(malloc(sizeof(T)));

		CHECK_ALLOC(scalarResult, "Failed to allocate new scalar buffer");

		shapeInfo = new ScalarShapeInformation(stream);
		scalarData = nd4j::buffer::createBuffer(scalarResult,1, stream);
		streamRef = stream;
		nd4j::buffer::copyDataToGpu(&scalarData, stream);
	}

	T getFinalResultFromDevice() {
		nd4j::buffer::copyDataFromGpu(&scalarData, streamRef);
		return scalarData->data[0];
	}

	/**
	 * Get the device shape information
	 * representing a scalar
	 */
	 Nd4jLong *getDeviceShapeInfo() {
		return shapeInfo->getShapeInfoGpuPointer();
	}

	/**
	 * Get the dZ pointers
	 */
	 T *getDevicePointer() {
		 return scalarData->gData;
	 }

	 /**
	  * Get the infinite dimension device pointer
	  */
	  Nd4jLong *getDimensionDevicePointer() {
		 return shapeInfo->getDimensionGpuPointer();
	 }

	 ~ScalarInfo() {
		 nd4j::buffer::freeBuffer(&scalarData);
		 delete shapeInfo;
	 }
};

void NativeOps::execPairwiseTransform(
        Nd4jPointer *extraPointers,
        int opNum,
        void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *hY, Nd4jLong *hYShapeInfo,
        void *dY, Nd4jLong *dYShapeInfo,
        void *hZ, Nd4jLong *hZShapeInfo,
        void *dZ, Nd4jLong *dZShapeInfo,
        void *extraParams) {

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    dim3 launchDims(256, 1024, 8192);

	if (yType != xType && yType != nd4j::DataType::BOOL && !this->isExperimentalEnabled())
		throw nd4j::datatype_exception::build("NativeOps::execPairwiseTransform both operands must have same data type", xType, yType);

    if (xType != zType && yType != zType)
        throw std::runtime_error("NativeOps::execPairwiseTransform requires Z operand to have either X or Y type");

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::pairwise_transforms::PairWiseTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dY, dYShapeInfo, hYShapeInfo, dZ, dZShapeInfo, hZShapeInfo, extraParams), LIBND4J_TYPES, LIBND4J_TYPES)
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::pairwise_transforms::PairWiseTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dY, dYShapeInfo, hYShapeInfo, dZ, dZShapeInfo, hZShapeInfo, extraParams), LIBND4J_TYPES)
#endif

    DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execPairwiseTransformBool(
        Nd4jPointer *extraPointers,
        int opNum,
        void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *hY, Nd4jLong *hYShapeInfo,
        void *dY, Nd4jLong *dYShapeInfo,
        void *hZ, Nd4jLong *hZShapeInfo,
        void *dZ, Nd4jLong *dZShapeInfo,
        void *extraParams) {    

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isB(zType))
		throw nd4j::datatype_exception::build("NativeOps::execPairwiseTransformBool wrong Z operand data type", nd4j::DataType::BOOL, zType);

    if (yType != xType)
        throw nd4j::datatype_exception::build("NativeOps::execPairwiseTransformBool both operands must have same data type", xType, yType);

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    dim3 launchDims(256, 1024, 16384);


    BUILD_DOUBLE_SELECTOR(xType, zType, functions::pairwise_transforms::PairWiseBoolTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES, BOOL_TYPES)
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execSummaryStatsScalar(Nd4jPointer *extraPointers,
                                       int opNum,
                                       void *hX, Nd4jLong *hXShapeInfo,
                                       void *dX, Nd4jLong *dXShapeInfo,
                                       void *extraParams,
                                       void *hZ, Nd4jLong *hZShapeInfo,
                                       void *dZ, Nd4jLong *dZShapeInfo,
                                       bool biasCorrected) {
	
	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    dim3 launchDims = dim3(256, 256, 32768);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, nullptr, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);
}

void   NativeOps::execBroadcastBool(
        Nd4jPointer *extraPointers,
        int opNum,
        void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *hY, Nd4jLong *hYShapeInfo,
        void *dY, Nd4jLong *dYShapeInfo,
        void *hZ, Nd4jLong *hZShapeInfo,
        void *dZ, Nd4jLong *dZShapeInfo,
		void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
	auto dTADShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
	auto dTADOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

	if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOps::execBroadcastBool requires Z operand to have BOOL type");

    if (yType != xType)
        throw std::runtime_error("NativeOps::execBroadcastBool requires both X & Y operands to have same type");

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);	
	
	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F3 opNum:[%i]\n", opNum);

	dim3 launchDims(256, 256, 16384);

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::broadcast::BroadcastBool, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, dTADShapeInfo, dTADOffsets, dTADShapeInfoZ, dTADOffsetsZ), LIBND4J_TYPES, BOOL_TYPES)

	DEBUG_KERNEL(stream, opNum);
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param dY
 * @param dYShapeInfo
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcast(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
		void *dX, Nd4jLong *dXShapeInfo,
		void *hY, Nd4jLong *hYShapeInfo,
		void *dY, Nd4jLong *dYShapeInfo,
		void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo,
		void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {
/*
    hipEvent_t start;
    hipEventCreateWithFlags(&start, hipEventDisableTiming);
    timespec tsX;
    timespec tsY;
    clock_gettime(CLOCK_REALTIME, &tsX);
*/
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
	auto dTADShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
	auto dTADOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F3 opNum:[%i]\n", opNum);

	dim3 launchDims(256, 256, 16384);

#ifdef __ND4J_EXPERIMENTAL__
	BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, dTADShapeInfo, dTADOffsets, dTADShapeInfoZ, dTADOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, dTADShapeInfo, dTADOffsets, dTADShapeInfoZ, dTADOffsetsZ), LIBND4J_TYPES);
#endif

	DEBUG_KERNEL(stream, opNum);
}


/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
void NativeOps::execReduceFloat(Nd4jPointer *extraPointers,
							int opNum,
							void *hX, Nd4jLong *hXShapeInfo,
							void *dX, Nd4jLong *dXShapeInfo,
							void *extraParams,
							void *hZ, Nd4jLong *hZShapeInfo,
							void *dZ, Nd4jLong *dZShapeInfo) {

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("FF7 opNum:[%i]\n", opNum);

	auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw std::runtime_error("NativeOps::execReduceFloat requires Z operand to have floating point type");

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceFloatFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 1, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, FLOAT_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceFloat(...) failed");
}

void   NativeOps::execReduceSame(Nd4jPointer *extraPointers,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("SF8 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != xType)
        throw datatype_exception::build("NativeOps::execReduceSame requires both X & Z operands to have same type", xType, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_SINGLE_SELECTOR(xType, functions::reduce::ReduceSameFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 1, reductionPointer, dTADShapeInfo), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceSame(...) failed");
}

void NativeOps::execReduceSame(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("SF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    auto xRank = shape::rank(hXShapeInfo);

    if (zType != xType)
        throw datatype_exception::build("NativeOps::execReduceSame requires both X & Z operands to have same type", xType, zType);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_SINGLE_SELECTOR(xType, functions::reduce::ReduceSameFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, reductionPointer, dTADShapeInfo, dTADOffsets), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceSame(...) failed");
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceLong(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("LF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::INT64)
        throw datatype_exception::build("NativeOps::execReduceLong wrong Z data type", nd4j::DataType::INT64, zType);

    auto xRank = shape::rank(hXShapeInfo);
    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, reductionPointer, dTADShapeInfo, dTADOffsets), LIBND4J_TYPES, LONG_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceLong(...) failed");

}

////////////////////////////////////////////////////////////////////////
void   NativeOps::execReduceLong(Nd4jPointer *extraPointers,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("LF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::INT64)
        throw datatype_exception::build("NativeOps::execReduceLong wrong Z data type", nd4j::DataType::INT64, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 1, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, LONG_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceLong(...) failed");
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduceBool(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
    auto dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("BF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::BOOL)
        throw std::runtime_error("NativeOps::execReduceBool requires Z operand to have BOOL type");

    auto xRank = shape::rank(hXShapeInfo);
    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, reductionPointer, dTADShapeInfo, dTADOffsets), LIBND4J_TYPES, BOOL_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceBool(...) failed");
}

////////////////////////////////////////////////////////////////////////
void   NativeOps::execReduceBool(Nd4jPointer *extraPointers,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("BF7 opNum:[%i]\n", opNum);

    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::BOOL)
        throw std::runtime_error("NativeOps::execReduceBool requires Z operand to have BOOL type");

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 1, reductionPointer, dTADShapeInfo), LIBND4J_TYPES, BOOL_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execReduceBool(...) failed");
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduce(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *extraParams,
        void *hZ, Nd4jLong *hZShapeInfo,
        void *dZ, Nd4jLong *dZShapeInfo,
		void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	
	Nd4jLong *hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	Nd4jLong *dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	Nd4jLong *dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
	
	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F2 opNum:[%i]\n", opNum);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	void *reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
	auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    if (zType != nd4j::DataType::INT64)
        throw datatype_exception::build("NativeOps::execIndexReduce requires Z operand to have INT64 type", zType);

	auto dz = reinterpret_cast<Nd4jLong*>(dZ);
	BUILD_SINGLE_SELECTOR(xType, functions::indexreduce::IndexReduce,  ::executeIndexReduce(launchDims, stream, opNum, dX, dXShapeInfo, shape::rank(hXShapeInfo), extraParams, dz, dZShapeInfo, shape::rank(hZShapeInfo), dimension, dimensionLength, 1, allocationPointer, reductionPointer, dTADShapeInfo, dTADOffsets), LIBND4J_TYPES);
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
void   NativeOps::execReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *extraParams,
        void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo,
		void *hDimension, Nd4jLong *hDimensionShape,
		void *dDimension, Nd4jLong *dDimensionShape) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);
	auto dTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
	auto dTADOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);

	
	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F8 opNum:[%i]\n", opNum);

	void *reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto xRank = shape::rank(hXShapeInfo);
    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceFloatFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX,dXShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, reductionPointer, dTADShapeInfo, dTADOffsets), LIBND4J_TYPES, FLOAT_TYPES);
}

/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 */
void NativeOps::execIndexReduceScalar(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
        void *dX, Nd4jLong *dXShapeInfo,
        void *extraParams,
        void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo){

	if (nd4j::Environment::getInstance()->isDebug())
		printf("F1 opNum:[%i]\n", opNum);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);	

	// void *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	void *reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose() && launchDims.x == 1)
		printf("AF1 opNum:[%i]\n", opNum);
	
	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    // FIXME: we want Z to be one of integer types
	//if (!DataTypeUtils::isZ(zType))
	//    throw nd4j::datatype_exception("NativeOps::execIndexReduceScalar requires Z operand to have one of integer types")
	if (zType != nd4j::DataType::INT64)
        throw nd4j::datatype_exception::build("NativeOps::exeIndexReduceScalar requires Z operand to have INT64 data type", zType);

    auto dz = reinterpret_cast<Nd4jLong*>(dZ);

    BUILD_SINGLE_SELECTOR(xType, functions::indexreduce::IndexReduce, ::executeIndexReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, shape::rank(hXShapeInfo), extraParams, dz, nullptr, 0, nullptr, 0, 1, allocationPointer, reductionPointer, nullptr, nullptr), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execIndexReduceScalar(...) failed");
}

void NativeOps::execTransformSame(Nd4jPointer *extraPointers,int opNum,
                                   void *hX, Nd4jLong *hXShapeInfo,
                                   void *dX, Nd4jLong *dXShapeInfo,
                                   void *hZ, Nd4jLong *hZShapeInfo,
                                   void *dZ, Nd4jLong *dZShapeInfo,
                                   void *extraParams) {
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    dim3 launchDims(512, 512, 16384);

    auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (xType != zType)
        throw std::runtime_error("NativeOps::execTransformSame requires X & Z to have same type");

    //nd4j_printf("Going to execute transformSame; opNum: %i\n", opNum);

    BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformSame, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "execTransformSame(...) failed");
}

void NativeOps::execTransformBool(Nd4jPointer *extraPointers,int opNum,
								  void *hX, Nd4jLong *hXShapeInfo,
								  void *dX, Nd4jLong *dXShapeInfo,
								  void *hZ, Nd4jLong *hZShapeInfo,
								  void *dZ, Nd4jLong *dZShapeInfo,
								  void *extraParams) {
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	dim3 launchDims(512, 512, 16384);

	auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOps::execTransformBool requires Z to have same boolean type");

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformBool, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, BOOL_TYPES);
}

void NativeOps::execTransformAny(Nd4jPointer *extraPointers,int opNum,
								  void *hX, Nd4jLong *hXShapeInfo,
								  void *dX, Nd4jLong *dXShapeInfo,
								  void *hZ, Nd4jLong *hZShapeInfo,
								  void *dZ, Nd4jLong *dZShapeInfo,
								  void *extraParams) {
	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
	auto zType = ArrayOptions::dataType(hZShapeInfo);

	switch (opNum) {
        case transform::IsMax: {
                bool scalarCheat = false;
                if (extraParams == nullptr) {
                    scalarCheat = true;
                }

                auto special = reinterpret_cast<double *>(extraPointers[17]);

                if (scalarCheat) {
                    auto scalarShape = ShapeBuilders::createScalarShapeInfo(nd4j::DataType::INT64);
                    /**
                    * In case of vector-input for IsMax, it just turns into IndexReduce call + further filler call
                    */
                    execIndexReduceScalar(extraPointers, indexreduce::IndexMax, nullptr, hXShapeInfo, dX, dXShapeInfo, extraParams, nullptr, scalarShape, special, nullptr);
                    Nd4jLong maxIdx = -119;
                    checkCudaErrors(hipStreamSynchronize(*stream));
                    hipMemcpyAsync(&maxIdx, special, sizeof(Nd4jLong), hipMemcpyDeviceToHost, *stream);
                    checkCudaErrors(hipStreamSynchronize(*stream));
                    int targetIdx = 0;

                    if (shape::order(hXShapeInfo) == 'c' || shape::order(hXShapeInfo) == 'f' && maxIdx * shape::stride(hXShapeInfo)[shape::rank(hXShapeInfo) - 1] >= shape::length(hXShapeInfo))
                        targetIdx = maxIdx;
                    else
                        targetIdx = maxIdx * shape::stride(hXShapeInfo)[shape::rank(hXShapeInfo) - 1];

                    dim3 launchDims(1, 512, 1024);
                    BUILD_SINGLE_SELECTOR(zType, fillIsMaxGeneric, (launchDims, stream, dZ, shape::length(hZShapeInfo), targetIdx), LIBND4J_TYPES);

                    nd4j::DebugHelper::checkErrorCode(stream, "Legacy IsMax(...) failed");

                    delete[] scalarShape;
                } else {
                    auto hostYShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[7]);
                    auto hostTShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[19]);
                    auto tadMaxShapeInfo = reinterpret_cast<Nd4jLong *> (extraPointers[10]);
                    auto tadMaxOffsets = reinterpret_cast<Nd4jLong *> (extraPointers[11]);
                    int *dimension = reinterpret_cast<int *> (extraPointers[15]);
                    int dimensionLength = getDeviceId(extraPointers[18]);

                    auto cshape = ShapeBuilders::createVectorShapeInfo(nd4j::DataType::INT32, dimensionLength);

                    // we call for IMax on specified dimension
                    execIndexReduce(extraPointers, indexreduce::IndexMax, nullptr, hXShapeInfo, dX, dXShapeInfo, extraParams, nullptr, hostTShapeInfo, special, hostYShapeInfo, nullptr, cshape, dimension, nullptr);

                    DEBUG_KERNEL(stream, opNum);

                    dim3 launchDims(256, 256, 16384);

                    // at this point, all IMax indexes are gathered, and we execute filler
                    BUILD_SINGLE_SELECTOR(zType, fillDimensionalIsMaxGeneric, (launchDims, stream, special, dZ, dZShapeInfo, tadMaxShapeInfo, dimension, dimensionLength, tadMaxOffsets), LIBND4J_TYPES);

                    nd4j::DebugHelper::checkErrorCode(stream, "Legacy IsMax(...) failed");

                    delete[] cshape;
                }
            }
            break;
        default: {
            dim3 launchDims(512, 512, 16384);

            BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformAny, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, LIBND4J_TYPES);
        }
	}
}


void NativeOps::execTransformStrict(Nd4jPointer *extraPointers,int opNum,
                                  void *hX, Nd4jLong *hXShapeInfo,
                                  void *dX, Nd4jLong *dXShapeInfo,
                                  void *hZ, Nd4jLong *hZShapeInfo,
                                  void *dZ, Nd4jLong *dZShapeInfo,
                                  void *extraParams) {
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    dim3 launchDims(512, 512, 16384);

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (xType != zType || !DataTypeUtils::isR(xType))
        throw datatype_exception::build("NativeOps::execTransformStrict requires X & Z to have same floating point type", xType, zType);

    switch (opNum) {
        case transform::SoftMax:
        case transform::SoftMaxDerivative:
        case transform::LogSoftMax: {
                if (shape::isVector(hXShapeInfo)) {
                    int length = shape::length(hXShapeInfo);
                    int block = nd4j::math::nd4j_min<int>(length, 256);

                    launchDims.x = 1;
                    launchDims.y = block;
                    launchDims.z += (block * sizeof(double) * 4);

                    BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformStrict, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), FLOAT_TYPES);
                } else {
                    auto shape = shape::shapeOf(hXShapeInfo);
                    int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
                    float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

                    // special pointer for special buffer for special ops
                    auto specialPointer = reinterpret_cast<double *>(extraPointers[6]);
                    auto dimension = reinterpret_cast<int *>(specialPointer);
                    auto maxDimension = dimension + 1;
                    auto maxShapeBuffer = reinterpret_cast<Nd4jLong *>(maxDimension + 1);
                    auto special = reinterpret_cast<double *> (maxShapeBuffer + (MAX_RANK * 2 + 4));

                    Nd4jPointer tempPointers[16];
                    tempPointers[0] = extraPointers[0];
                    tempPointers[1] = extraPointers[1];
                    tempPointers[2] = extraPointers[2];
                    tempPointers[3] = extraPointers[3];
                    tempPointers[4] = extraPointers[4];
                    tempPointers[5] = extraPointers[5];
                    tempPointers[6] = extraPointers[6];
                    tempPointers[7] = extraPointers[7];
                    tempPointers[8] = extraPointers[8];
                    tempPointers[9] = extraPointers[9];
                    tempPointers[10] = extraPointers[10];
                    tempPointers[11] = extraPointers[11];
                    tempPointers[12] = extraPointers[12];
                    tempPointers[13] = extraPointers[13];
                    tempPointers[14] = extraPointers[14];
                    tempPointers[15] = extraPointers[15];

                    Nd4jLong maxShape[2] = {shape::shapeOf(hXShapeInfo)[0], 1};
                    auto hostMaxShapeBuffer = shape::shapeBuffer(2, xType, maxShape);

                    auto cshape = ShapeBuilders::createVectorShapeInfo(nd4j::DataType::INT32, 1);

                    tempPointers[7] = (Nd4jPointer) hostMaxShapeBuffer;
                    tempPointers[8] = (Nd4jPointer) hostMaxShapeBuffer;

                    prepareShapeBuffer<<<1, 1, 128, *stream>>>(dimension, maxDimension, maxShapeBuffer, shape[0], xType);

                    DEBUG_KERNEL(stream, opNum);

                    //shape::printShapeInfo(maxShapeBuffer);
                    tempPointers[9] = extraPointers[12];
                    tempPointers[10] = extraPointers[13];
                    tempPointers[11] = extraPointers[14];

                    // max 3
                    execReduceSame(tempPointers, reduce::Max, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, nullptr, hostMaxShapeBuffer, special, maxShapeBuffer,
                                   nullptr, cshape, maxDimension, nullptr);

                    DEBUG_KERNEL(stream, opNum);

                    tempPointers[8] = extraPointers[8];
                    tempPointers[9] = extraPointers[9];
                    tempPointers[10] = extraPointers[10];
                    tempPointers[11] = extraPointers[11];
                    tempPointers[12] = extraPointers[10];
                    tempPointers[13] = extraPointers[11];

                    // sub 1
                    execBroadcast(tempPointers, broadcast::Subtract, hX, hXShapeInfo, dX, dXShapeInfo, nullptr, hostMaxShapeBuffer, special, maxShapeBuffer, nullptr, hZShapeInfo, dZ, dZShapeInfo, nullptr, cshape, dimension, nullptr);

                    DEBUG_KERNEL(stream, opNum);

                    // exp 3
                    execTransformStrict(extraPointers, transform::Exp, hZ, hZShapeInfo, dZ, dZShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams);

                    DEBUG_KERNEL(stream, opNum);

                    tempPointers[8] = tempPointers[7];
                    tempPointers[9] = extraPointers[12];
                    tempPointers[10] = extraPointers[13];
                    tempPointers[11] = extraPointers[14];

                    //sum 1
                    execReduceSame(tempPointers, reduce::Sum, hZ, hZShapeInfo, dZ, dZShapeInfo, extraParams, nullptr, hostMaxShapeBuffer, special, maxShapeBuffer,
                                   nullptr, cshape, maxDimension, nullptr);

                    tempPointers[8] = extraPointers[8];
                    tempPointers[9] = extraPointers[9];
                    tempPointers[10] = extraPointers[10];
                    tempPointers[11] = extraPointers[11];
                    tempPointers[12] = extraPointers[10];
                    tempPointers[13] = extraPointers[11];

                    // divide 3
                    execBroadcast(tempPointers, broadcast::Divide, hZ, hZShapeInfo, dZ, dZShapeInfo, nullptr, hostMaxShapeBuffer, special, maxShapeBuffer, nullptr, hZShapeInfo, dZ, dZShapeInfo,
                                  nullptr, cshape, dimension, nullptr);

                    DEBUG_KERNEL(stream, opNum);

                    // log 3
                    if (opNum == transform::LogSoftMax)
                        execTransformStrict(extraPointers, transform::Log, nullptr, hZShapeInfo, dZ, dZShapeInfo, nullptr, hZShapeInfo, dZ, dZShapeInfo, extraParams);
                    else if (opNum == transform::SoftMaxDerivative)
                        execTransformStrict(extraPointers, transform::SpecialDerivative, nullptr, hZShapeInfo, dZ, dZShapeInfo, nullptr, hZShapeInfo, dZ, dZShapeInfo, extraParams);

                    nd4j::DebugHelper::checkErrorCode(stream, "SoftMax(...) failed");

                    delete hostMaxShapeBuffer;
                    delete[] cshape;
                }
            }
            break;
        default: {
            BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformStrict, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), FLOAT_TYPES);
        }
    }
}

void NativeOps::execTransformFloat(Nd4jPointer *extraPointers,int opNum,
                                    void *hX, Nd4jLong *hXShapeInfo,
                                    void *dX, Nd4jLong *dXShapeInfo,
                                    void *hZ, Nd4jLong *hZShapeInfo,
                                    void *dZ, Nd4jLong *dZShapeInfo,
                                    void *extraParams) {
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw datatype_exception::build("NativeOps::execTransformFloat requires Z to have floating point type", zType);

    if (opNum == transform::Histogram) {
        dim3 launchDims(256, 256, 32768);

        Nd4jPointer maskedAllocPointer;
        auto length = shape::length(hZShapeInfo);
        hipMalloc(reinterpret_cast<void **>(&maskedAllocPointer), length * launchDims.x * DataTypeUtils::sizeOf(nd4j::DataType::INT64));
        auto imaskedAllocPointer = reinterpret_cast<int *>(maskedAllocPointer);

        BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformFloat, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, imaskedAllocPointer, reductionPointer, nullptr, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

        checkCudaErrors(hipStreamSynchronize(*stream));
        hipFree(maskedAllocPointer);
    } else {
        dim3 launchDims(512, 512, 16384);
        BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformFloat, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, FLOAT_TYPES);
    }
}


/**
 * Append an input array
 * to the end of a flat array
 * in a particular order
 * @param offset the offset of the array to start at
 * @param order the order
 * @param dZ the dZ array
 * @param dZShapeInfo the shape info for te array
 * @param input the input for the array
 * @param inputShapeInfo the shape information for that array
 */
void NativeOps::flatten(Nd4jPointer *extraPointers,
						int offset,
						char order,
						void *hZ, Nd4jLong *hZShapeInfo,
						void *dZ, Nd4jLong *dZShapeInfo,
						void *hInput, Nd4jLong *hInputShapeInfo,
						void *dInput, Nd4jLong *dInputShapeInfo) {
	
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hYShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[7]);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F22 opNum:[7]\n");

	// int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getBasicLaunchParams(getDeviceId(extraPointers[2]), shape::length(hYShapeInfo), 2, funcAttributes[30]);

	if (nd4j::Environment::getInstance()->isVerbose() && launchDims.x == 1)
		printf("AF222 opNum:[7]\n");
	
	auto type = nd4j::ArrayOptions::dataType(hInputShapeInfo);    
    BUILD_SINGLE_SELECTOR(type, flattenKernelGeneric, (launchDims, stream, extraPointers, offset, order, dZ, dZShapeInfo, dInput, dInputShapeInfo), LIBND4J_TYPES);

	DEBUG_KERNEL(stream, -1);
}



void NativeOps::checkP2P() {
	int curDevice = 0;

	hipGetDevice(&curDevice);

	int devCnt = 0;
	hipGetDeviceCount(&devCnt);

	if (curDevice < 0 && curDevice > devCnt)
		curDevice = 0;

	bool tempSupport = true;

	if (devCnt > 1) {
		for (int dX = 0; dX < devCnt; dX++) {

			for (int dY = 0; dY < devCnt; dY++) {
				if (dX == dY)
					continue;

				int canAccess = 0;
				hipSetDevice(dX);

				hipDeviceCanAccessPeer(&canAccess, dX , dY);

				if (!canAccess) {
                    tempSupport = false;
                    break;
                }
			}
		}

		supportedP2P = tempSupport;

		hipSetDevice(curDevice);
	} else {
		// if we have only 1 device - we say that we support P2P, since all data will be on 1 device
		supportedP2P = true;
	}
}

void NativeOps::enableP2P(bool enable) {
    if (enable == allowedP2P)
        return;

    int curDevice = 0;

    hipGetDevice(&curDevice);

    int devCnt = 0;
    hipGetDeviceCount(&devCnt);

	if (curDevice < 0 && curDevice > devCnt)
		curDevice = 0;

    if (devCnt > 1) {
        for (int dX = 0; dX < devCnt; dX++) {

            for (int dY = 0; dY < devCnt; dY++) {
                if (dX == dY)
                    continue;

                int canAccess = 0;
                hipSetDevice(dX);

                hipDeviceCanAccessPeer(&canAccess, dX , dY);

                if (canAccess) {
                    if (enable) {
                        hipDeviceEnablePeerAccess(dY, 0);
                    } else {
                        hipDeviceDisablePeerAccess(dY);
                    }
                } else {
					if (nd4j::Environment::getInstance()->isVerbose()) printf("Peer access [%i] -> [%i] isn't possible\n", dX, dY);
				}
            }
        }

        hipSetDevice(curDevice);
    }

    allowedP2P = enable;

    hipSetDevice(curDevice);
}

bool NativeOps::isP2PAvailable() {
	return supportedP2P;
}


void NativeOps::initializeDevicesAndFunctions() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	deviceProperties = new hipDeviceProp_t[devCnt];
	for (int i = 0; i < devCnt; i++) {
		hipSetDevice(i);
		hipGetDeviceProperties(&deviceProperties[i], i);

		hipDeviceSetLimit(hipLimitStackSize, 4096);
	}

	hipSetDevice(0);

	checkP2P();

	// enabling p2p gpu access if it's supported
	if (supportedP2P && devCnt > 1)
    	enableP2P(allowedP2P);	
}

void NativeOps::initializeFunctions(Nd4jPointer *functions) {
    nd4j::BlasHelper::getInstance()->initializeDeviceFunctions(functions);
	/*
	this->hipblasSgemv = (CublasSgemv)functions[0];
    this->hipblasDgemv = (CublasDgemv)functions[1];
    this->hipblasHgemm = (CublasHgemm)functions[2];
    this->hipblasSgemm = (CublasSgemm)functions[3];
    this->hipblasDgemm = (CublasDgemm)functions[4];
    this->cublasSgemmEx = (CublasSgemmEx)functions[5];
    this->hipblasHgemmBatched = (CublasHgemmBatched)functions[6];
    this->hipblasSgemmBatched = (CublasSgemmBatched)functions[7];
    this->hipblasDgemmBatched = (CublasDgemmBatched)functions[8];
	*/
}


/**
 * This method acquires memory chunk of requested size on host side
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocHost(Nd4jLong memorySize, int flags) {
	Nd4jPointer pointer;
	// hipHostMallocMapped |hipHostMallocPortable
	hipError_t res = hipHostAlloc(reinterpret_cast<void **>(&pointer), memorySize, hipHostMallocDefault);
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method acquires memory chunk of requested size on specified device
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param ptrToDeviceId pointer to deviceId. For cuda that's just and int, for OpenCL that's pointer to device_id, etc
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocDevice(Nd4jLong memorySize, Nd4jPointer ptrToDeviceId, int flags) {
	Nd4jPointer pointer;
	hipError_t res = hipMalloc(reinterpret_cast<void **>(&pointer), memorySize);
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method releases previously allocated host memory space
 *
 * @param pointer pointer that'll be freed
 */
int NativeOps::freeHost(Nd4jPointer pointer) {
	hipError_t res = hipHostFree(reinterpret_cast<void *>(pointer));
	if (res != 0)
		pointer = 0L;
	return 1L;
}

/**
 * This method releases previously allocated memory space on device
 *
 * @param pointer pointer that'll be freed
 * @param ptrToDeviceId pointer to deviceId.
 */
int NativeOps::freeDevice(Nd4jPointer pointer, Nd4jPointer ptrToDeviceId) {
	hipError_t res = hipFree(reinterpret_cast<void *>(pointer));
	if (res != 0)
		pointer = 0L;
	return 1L;
}


Nd4jPointer NativeOps::createContext() {
	return 0L;
}

Nd4jPointer NativeOps::createStream() {
	Nd4jPointer nativeStream = (Nd4jPointer) malloc(sizeof(hipStream_t));

	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");

	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipStreamCreate(...) failed");

	return nativeStream;
}

Nd4jPointer NativeOps::createEvent() {
	Nd4jPointer nativeEvent= (Nd4jPointer) malloc(sizeof(hipEvent_t));

	CHECK_ALLOC(nativeEvent, "Failed to allocate new CUDA event buffer");

	hipError_t dZ = hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&nativeEvent), hipEventDisableTiming);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipEventCreateWithFlags(...) failed");


	return nativeEvent;
}

int NativeOps::registerEvent(Nd4jPointer event, Nd4jPointer stream) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipError_t dZ = hipEventRecord(*pEvent, *pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipEventRecord(...) failed");

	return 1;
}

int NativeOps::setDevice(Nd4jPointer ptrToDeviceId) {
	int deviceId = getDeviceId(ptrToDeviceId);
	hipError_t dZ = hipSetDevice(deviceId);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipSetDevice(...) failed");

	return 1;
}

Nd4jLong NativeOps::getDeviceFreeMemory(Nd4jPointer ptrToDeviceId) {
	int device = getDeviceId(ptrToDeviceId);
	int orig = -1;

	hipGetDevice(&orig);

	if (device >= 0 && device != orig) {
		hipSetDevice(device);
	}

	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	if (device >= 0 && device != orig) {
		hipSetDevice(orig);
	}

	return (Nd4jLong) memFree;
}

Nd4jLong NativeOps::getDeviceTotalMemory(Nd4jPointer ptrToDeviceId) {
	int device = getDeviceId(ptrToDeviceId);
	int orig = -1;

	hipGetDevice(&orig);

	if (device >= 0 && device != orig) {
		hipSetDevice(device);
	}
	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	if (device >= 0 && device != orig) {
		hipSetDevice(orig);
	}

	return (Nd4jLong) memTotal;
}

int NativeOps::memcpy(Nd4jPointer dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {

	return memcpyAsync(dst, src, size, flags, reserved);
}

int NativeOps::memcpyAsync(Nd4jPointer dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipMemcpyKind 	kind;

	DEBUG_KERNEL(pStream, 0);

	switch (flags) {
		case 0: {
				kind = hipMemcpyHostToHost;
			}
			break;
		case 1: {
				kind = hipMemcpyHostToDevice;
			}
			break;
		case 2: {
				kind = hipMemcpyDeviceToHost;
			}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
		default: {

			printf("UNDEFINED MEMCPY!\n");
			break;
		}
	}

	hipError_t dZ = hipMemcpyAsync(reinterpret_cast<void *>(dst), const_cast<const void *>(reinterpret_cast<void *>(src)), static_cast<size_t>(size), kind, *pStream);
	if (dZ != 0) {
        checkCudaErrors(dZ);
		printf("Failed on [%lu] -> [%lu], size: [%i], direction: [%i], dZ: [%i]\n", src, dst, size, flags, static_cast<int>(dZ));
        fflush(stdout);
        fflush(stderr);
        throw std::runtime_error("hipMemcpyAsync(...) failed");
		//return 0L;
	}

	return 1;
}

int NativeOps::memset(Nd4jPointer dst, int value, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipError_t dZ = hipMemset(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size));
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipMemset(...) failed");

	return 1;
}

int NativeOps::memsetAsync(Nd4jPointer dst, int value, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipError_t dZ = hipMemsetAsync(reinterpret_cast<void *>(dst), value, static_cast<size_t>(size), *pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("hipMemsetAsync(...) failed");

	return 1;
}

int NativeOps::destroyEvent(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipError_t dZ = hipEventDestroy(*pEvent);
	checkCudaErrors(dZ);
	if (dZ != 0)
		throw std::runtime_error("cudaEvenDestroy(...) failed");

	return 1;
}

int NativeOps::streamSynchronize(Nd4jPointer stream) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipError_t dZ = hipStreamSynchronize(*pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
        throw std::runtime_error("hipStreamSynchronize(...) failed");

	return 1L;
}

int NativeOps::eventSynchronize(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);

	hipError_t dZ = hipEventSynchronize(*pEvent);
	checkCudaErrors(dZ);
	if (dZ != 0)
        throw std::runtime_error("hipEventSynchronize(...) failed");

	return 1L;
}

int NativeOps::getAvailableDevices() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	return devCnt;
}

void NativeOps::enableDebugMode(bool reallyEnable) {
	nd4j::Environment::getInstance()->setDebug(reallyEnable);
}

void NativeOps::setGridLimit(int gridSize) {
	if (gridSize > 8192)
		gridSize = 8192;
	if (gridSize < 1)
		gridSize = 1;
	blockLimit = gridSize;
}

int NativeOps::ompGetMaxThreads() {
	return maxThreads;
}

int NativeOps::ompGetNumThreads() {
	return maxThreads;
}

void NativeOps::setOmpNumThreads(int threads) {
	if (threads > 1024)
		threads = 1024;
	if (threads < 32)
		threads = 32;
	maxThreads = threads;
}

void NativeOps::enableVerboseMode(bool reallyEnable) {
	nd4j::Environment::getInstance()->setVerbose(reallyEnable);
}

int NativeOps::getDeviceMajor(Nd4jPointer ptrToDeviceId) {
	int device = getDeviceId(ptrToDeviceId);
	return deviceProperties[device].major;
}

int NativeOps::getDeviceMinor(Nd4jPointer ptrToDeviceId) {
	int device = getDeviceId(ptrToDeviceId);
	return deviceProperties[device].minor;
}


const char * NativeOps::getDeviceName(Nd4jPointer ptrToDeviceId) {
    int device = getDeviceId(ptrToDeviceId);

    return deviceProperties[device].name;
}

/**
  * Concatneate multi array of the same shape together
  * along a particular dimension
  */
 void NativeOps::concat(
		Nd4jPointer *extraPointers,
        int dimension,
        int numArrays,
        Nd4jPointer *data, Nd4jPointer *inputShapeInfo,
		Nd4jPointer *ddata, Nd4jPointer *dinputShapeInfo,
		void *hZ, Nd4jLong *hZShapeInfo,
        void *dZ, Nd4jLong *dZShapeInfo,
		Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto hXShapeInfo = hZShapeInfo;
	auto hShapePointers = reinterpret_cast<Nd4jLong **>(inputShapeInfo);
	// numArrays will be used as number of TADs, so each block process 1 input

	int smem = 8192;
	bool isVstack = false;
	bool isScalar = true;
	bool isHstack = false;	

	for (int i = 0; i < numArrays; i++) {
		if (!shape::isScalar(hShapePointers[i])) {
			isScalar = false;
			break;
		}
	}

	if (!isScalar && dimension == 0 && shape::rank(hZShapeInfo) == 2 && shape::order(hZShapeInfo) == 'c' ) {
		isVstack = true;
        for (int i = 0; i < numArrays; i++) {
			if (!shape::isVector(hShapePointers[i]) || shape::elementWiseStride(hShapePointers[i]) <= 0 ||
				shape::order(hShapePointers[i]) != 'c') {
				isVstack = false;
				break;
			}
		}
	}

    // let's try to fit N-dimensional vstack
    if (!isVstack && !isScalar && dimension == 0 && shape::order(hXShapeInfo) == 'c') {
		auto length0 = shape::length(hShapePointers[0]);
        isVstack = true;
        for (int i = 0; i < numArrays; i++) {
            if (shape::elementWiseStride(hShapePointers[i]) <= 0 || shape::order(hShapePointers[i]) != 'c' || length0 != shape::length(hShapePointers[i])) {
                isVstack = false;
                break;
            }
        }
    }

	if (!isScalar && !isVstack && dimension == 1 && shape::isVector(hZShapeInfo)) {
		isHstack = true;
		for (int i = 0; i < numArrays; i++) {
			if (!shape::isVector(hShapePointers[i]) || shape::elementWiseStride(hShapePointers[i]) <= 0) {
				isHstack = false;
				break;
			}
		}
	}

	if (isScalar) {
		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
			printf("Going scalar concat\n");	

		dim3 launchDims(128, 128, 16384);
		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
		BUILD_SINGLE_SELECTOR(zType, concatKernelScalarGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), dZ), LIBND4J_TYPES);

	} else if (isVstack) {
		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
			printf("Going VStack concat\n");

		dim3 launchDims(128, 512, 16384);
		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
		BUILD_SINGLE_SELECTOR(zType, concatKernelVStackGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), reinterpret_cast<Nd4jPointer *>(dinputShapeInfo[0]), dZ, dZShapeInfo), LIBND4J_TYPES);

	} else if (isHstack) {
		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
			printf("Going HStack concat\n");
		
		dim3 launchDims(128, 128, 16384);
		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
		BUILD_SINGLE_SELECTOR(zType, concatKernelHStackGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), reinterpret_cast<Nd4jPointer *>(dinputShapeInfo[0]), dZ, dZShapeInfo), LIBND4J_TYPES);
	} else {
		if (nd4j::Environment::getInstance()->isDebugAndVerbose())
			printf("Going generic concat\n");

        auto devZTadShape = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
		auto devZOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
		
		dim3 launchDims(128, 128, 8192);
		auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
		BUILD_SINGLE_SELECTOR(zType, concatKernelGeneric, (launchDims, stream, numArrays, reinterpret_cast<Nd4jPointer *>(ddata[0]), reinterpret_cast<Nd4jPointer *>(dinputShapeInfo[0]), dZ, dZShapeInfo,  reinterpret_cast<Nd4jPointer *>(tadPointers[0]), reinterpret_cast<Nd4jPointer *>(offsetPointers[0]), devZTadShape, devZOffsets), LIBND4J_TYPES);
	}
	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("sharedMemory requested for concatFloat: [%i], registers: [%i]\n", smem, funcAttributes[31].numRegs);

    hipError_t res = hipStreamSynchronize(*stream);
    checkCudaErrors(res);
    nd4j::DebugHelper::checkErrorCode(stream, "Legacy ConcatFloat(...) failed");
}



void NativeOps::specialConcat(
        Nd4jPointer *extraPointers,
        int dimension,
        int numArrays,
        Nd4jPointer *data,
        Nd4jPointer *inputShapeInfo,
        void *dZ,
        Nd4jLong *dZShapeInfo, Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {
    nd4j::SpecialMethods<float>::concatCpuGeneric(
            dimension,
            numArrays,
            data,
            inputShapeInfo,
            dZ,
            dZShapeInfo);

}


/**
 * This method saves
 */
void NativeOps::tadOnlyShapeInfo(Nd4jLong *dXShapeInfo, int *dimension, int dimensionLength, Nd4jLong *target, Nd4jLong *offsets) {
    //nd4j_printf("START ------->\n","");
    //nd4j_printf("Shape pointer: [%p]\n", dXShapeInfo);
	//nd4j_printf("Dimension pointer: [%p]\n", dimension);
    //nd4j_printf("shape rank: [%i]; dimLength: [%i]\n", shape::rank(dXShapeInfo), dimensionLength);
    //shape::printShapeInfoLinear(dXShapeInfo);
    //fflush(stdout);
    //shape::printArray<int>(reinterpret_cast<void*>(dimension), dimensionLength, "dimensions");
    //fflush(stdout);
    //nd4j_printf("END ------->\n","");

	shape::TAD tad;
	tad.init(dXShapeInfo, dimension, dimensionLength);

	//nd4j_printf("Creating TAD shape...\n","");
	tad.createTadOnlyShapeInfo();
	//nd4j_printf("Creating TAD offsets...\n","");
	tad.createOffsets();

	//nd4j_printf("memcpy TAD shape...\n","");
	std::memcpy(reinterpret_cast<void *>(target), tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo));
	//nd4j_printf("memcpy TAD offsets...\n","");
	std::memcpy(reinterpret_cast<void *>(offsets), tad.tadOffsets, tad.numTads * sizeof(Nd4jLong));
	//nd4j_printf("memcpy finished...\n","");
}

int NativeOps::memcpyConstantAsync(Nd4jLong dst, Nd4jPointer src, Nd4jLong size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipMemcpyKind 	kind;

	DEBUG_KERNEL(pStream, -1);

	switch (flags) {
		case 0: {
			kind = hipMemcpyHostToHost;
		}
			break;
		case 1: {
			kind = hipMemcpyHostToDevice;
		}
			break;
		case 2: {
			kind = hipMemcpyDeviceToHost;
		}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
	}
	//hipError_t dZ = hipMemcpyAsync((void *) dst, (const void *) src, (size_t) size, kind, *pStream);
	hipError_t dZ = hipMemcpyToSymbolAsync(HIP_SYMBOL(deviceConstantMemory), const_cast<const void *>(src), size, dst, kind, *pStream);
	checkCudaErrors(dZ);
	if (dZ != 0)
        throw std::runtime_error("hipMemcpyToSymbolAsync(...) failed");

	return 1;
}

Nd4jPointer NativeOps::getConstantSpace() {
	Nd4jPointer dConstAddr;
	hipError_t dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

	if (dZ != 0)
        throw std::runtime_error("hipGetSymbolAddress(...) failed");

	return dConstAddr;
}

void NativeOps::pullRows(Nd4jPointer *extraPointers,
						 void *x, Nd4jLong *xShapeInfo,
						 void *dX, Nd4jLong *dXShapeInfo,
						 void *z, Nd4jLong *zShapeInfo,
						 void *dZ, Nd4jLong *dZShapeInfo,
						 Nd4jLong n,
						 Nd4jLong *indexes,
						 Nd4jLong *tadShapeInfo,
						 Nd4jLong *tadOffsets,
						 Nd4jLong *zTadShapeInfo,
						 Nd4jLong *zTadOffsets) {

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);	
	dim3 launchDims(64, 256, 1024);
	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, pullRowsKernelGeneric, (launchDims, stream, dX, dZ, n, indexes, tadShapeInfo, tadOffsets,  zTadShapeInfo,  zTadOffsets), LIBND4J_TYPES);
	
	DEBUG_KERNEL(stream, -1);
}


void NativeOps::average(Nd4jPointer *extras,
						Nd4jPointer *x, Nd4jLong *xShapeInfo,
						Nd4jPointer *dx, Nd4jLong *dXShapeInfo,
						void *z, Nd4jLong *zShapeInfo,
						void *dz, Nd4jLong *dzShapeInfo,
						int n,
						Nd4jLong length,
						bool propagate) {

	hipStream_t * stream = reinterpret_cast<hipStream_t *>(&extras[1]);
	int mode = getDeviceId(extras[3]);

	auto dX = reinterpret_cast<void **>(dx);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("averageFloat called\n");

	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
	// launching on gpu
	if (mode == 0) {
		dim3 launchDims(256, 256, 4096);
		// averagingKernelFloat<<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dX, dz, n, length, propagate);		
    	BUILD_SINGLE_SELECTOR(xType, averagingKernelGeneric, (launchDims, stream, dX, dz, n, length, propagate), LIBND4J_TYPES);		    	
        nd4j::DebugHelper::checkErrorCode(stream, "AverageFloat(...) failed");
	} else {
		// launching on host memory
        BUILD_SINGLE_SELECTOR(xType, nd4j::SpecialMethods, ::averageGeneric(x, z, zShapeInfo, n, length, propagate), LIBND4J_TYPES);
	}
}

void NativeOps::accumulate(Nd4jPointer *extras,
						   Nd4jPointer *x, Nd4jLong *xShapeInfo,
						   Nd4jPointer *dx, Nd4jLong *dXShapeInfo,
						   void *z, Nd4jLong *zShapeInfo,
						   void *dz, Nd4jLong *dzShapeInfo,
						   int n,
						   Nd4jLong length) {
	
	auto stream = reinterpret_cast<hipStream_t *>(&extras[1]);
	int mode = getDeviceId(extras[3]);

	auto dX = reinterpret_cast<void **>(dx);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("accumulateFloat called\n");
	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);

	// launching on gpu
	if (mode == 0) {
		dim3 launchDims(n, 256, 16384);
        BUILD_SINGLE_SELECTOR(xType, accumulateKernelGeneric, (launchDims, stream, dX, dz, n,length), LIBND4J_TYPES);
        nd4j::DebugHelper::checkErrorCode(stream, "AccumulateFloat(...) failed");
	} else {
		// launching on host memory        
        BUILD_SINGLE_SELECTOR(xType, nd4j::SpecialMethods, ::accumulateGeneric(x, z, zShapeInfo, n, length), LIBND4J_TYPES);
	}
}


void NativeOps::shuffle(Nd4jPointer *extras,
						Nd4jPointer *x, Nd4jPointer *xShapeInfo,
						Nd4jPointer *dx, Nd4jPointer *dXShapeInfo,
						Nd4jPointer *z, Nd4jPointer *zShapeInfo,
						Nd4jPointer *dz, Nd4jPointer *dZShapeInfo,
						int N,
						int *shuffleMap,
						Nd4jPointer *tadShapeInfo,
						Nd4jPointer *tadOffsets) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extras[1]);

    auto dX = reinterpret_cast<void **>(dx);
    auto dZ = reinterpret_cast<void **>(dz);
    auto xShape = reinterpret_cast<Nd4jLong **>(xShapeInfo);
    auto dxShape = reinterpret_cast<Nd4jLong **>(dXShapeInfo);
    auto tadOnlyShapeInfo = reinterpret_cast<Nd4jLong **>(tadShapeInfo);
    auto tadOffset = reinterpret_cast<Nd4jLong **>(tadOffsets);

    auto xType = nd4j::ArrayOptions::dataType(xShape[0]);
    dim3 launchDims(N, 256, 8192);
    BUILD_SINGLE_SELECTOR(xType, shuffleKernelGeneric, (launchDims, stream, dX, dxShape, dZ, N, shuffleMap,  tadOnlyShapeInfo, tadOffset), LIBND4J_TYPES);

	DEBUG_KERNEL(stream, 0);
}

/*
void NativeOps::execMetaPredicateShape(Nd4jPointer *extras, 
	                                  const int opTypeA, 
	                                  const int opNumA, 
	                                  const int opTypeB, 
	                                  const int opNumB, 
	                                  Nd4jLong N, 
	                                  void *hX, Nd4jLong *hXShapeInfo,
                                      void *dX, Nd4jLong *dXShapeInfo,
                                      void *hY, Nd4jLong *hYShapeInfo,
                                      void *dY, Nd4jLong *dYShapeInfo,
                                      void *hZ, Nd4jLong *hZShapeInfo,
                                      void *dZ, Nd4jLong *dZShapeInfo,
	                                  void *extraA, 
	                                  void *extraB, 
	                                  double scalarA, 
	                                  double scalarB) {
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extras[1]);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, functions::grid::GRIDShaped, ::execMetaPredicateShaped(stream, extras, opTypeA, opNumA, opTypeB, opNumB, N, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraA, extraB, scalarA, scalarB), LIBND4J_TYPES);
    // functions::grid::GRIDShaped<float>::execMetaPredicateShaped(stream, extras, opTypeA, opNumA, opTypeB, opNumB, N, dX, dXShapeInfo, dy, dYShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB);

	DEBUG_KERNEL(stream, opNumA);
}
*/

bool NativeOps::isExperimentalEnabled() {
    return nd4j::Environment::getInstance()->isExperimentalBuild();
}

void NativeOps::setOmpMinThreads(int threads) {
    minThreads = nd4j::math::nd4j_max<int>(32, threads);
    minThreads = nd4j::math::nd4j_min<int>(maxThreads, minThreads);
}

int NativeOps::getDevice() {
    int curDevice = -1;

    hipGetDevice(&curDevice);

    return curDevice;
}

void NativeOps::setElementThreshold(int num) {
    // this is no-op for CUDA
}

void NativeOps::setTADThreshold(int num) {
    // this is no-op for CUDA
}

void NativeOps::execSummaryStats(Nd4jPointer *extraPointers,
                                 int opNum,
                                 void *hX, Nd4jLong *hXShapeInfo,
                                 void *dX, Nd4jLong *dXShapeInfo,
                                 void *extraParams,
                                 void *hZ, Nd4jLong *hZShapeInfo,
                                 void *dZ, Nd4jLong *dZShapeInfo,
                                 bool biasCorrected) {
    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    dim3 launchDims = dim3(256, 256, 32768);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOps::execReduce3 requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduce(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, nullptr, biasCorrected, nullptr), LIBND4J_TYPES, FLOAT_TYPES);
}

void NativeOps::execSummaryStats(Nd4jPointer *extraPointers,
                                 int opNum,
                                 void *hX, Nd4jLong *hXShapeInfo,
                                 void *dX, Nd4jLong *dXShapeInfo,
                                 void *extraParams,
                                 void *hZ, Nd4jLong *hZShapeInfo,
                                 void *dZ, Nd4jLong *dZShapeInfo,
								 void *hDimension, Nd4jLong *hDimensionShape,
								 void *dDimension, Nd4jLong *dDimensionShape,
                                 bool biasCorrected,
								 Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    dim3 launchDims = dim3(256, 256, 32768);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOps::execReduce3 requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduce(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);
}

void NativeOps::execReduce3(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
                            Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                            Nd4jLong *yTadOnlyShapeInfo, Nd4jLong *yTadOffsets) {

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    dim3 launchDims(256, 256, 32768);

    if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOps::execReduce3 requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOps::execReduce3 requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::exec(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, nullptr, 1, 1, allocationPointer, tadOnlyShapeInfo, tadOffsets, yTadOnlyShapeInfo, yTadOffsets), LIBND4J_TYPES, FLOAT_TYPES)

    DEBUG_KERNEL(stream, opNum);
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduce3(Nd4jPointer *extraPointers,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
							void *hDimension, Nd4jLong *hDimensionShape,
							void *dDimension, Nd4jLong *dDimensionShape,
                            Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                            Nd4jLong *yTadOnlyShapeInfo, Nd4jLong *yTadOffsets) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOps::execReduce3 requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOps::execReduce3 requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::exec(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, 1, allocationPointer, tadOnlyShapeInfo, tadOffsets, yTadOnlyShapeInfo, yTadOffsets), LIBND4J_TYPES, FLOAT_TYPES)
}

////////////////////////////////////////////////////////////////////////
void NativeOps::execReduce3Scalar(Nd4jPointer *extraPointers,int opNum,
                                  void *hX, Nd4jLong *hXShapeInfo,
                                  void *dX, Nd4jLong *dXShapeInfo,
                                  void *extraParams,
                                  void *hY, Nd4jLong *hYShapeInfo,
                                  void *dY, Nd4jLong *dYShapeInfo,
                                  void *hZ, Nd4jLong *hZShapeInfo,
                                  void *dZ, Nd4jLong *dZShapeInfo) {

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	auto allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	auto reductionPointer = reinterpret_cast<void *>(extraPointers[4]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOps::execReduce3Scalar requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOps::execReduce3Scalar requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execScalar(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, allocationPointer, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);
}



void NativeOps::execScalarBool(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
		void *dX, Nd4jLong *dXShapeInfo,
		void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo,
		void *hScalar, Nd4jLong *hScalarShapeInfo,
		void *dScalar, Nd4jLong *dScalarShapeInfo,
		void *extraParams) {
	
	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = dim3(256, 512, 8192);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (xType != yType )
		throw std::runtime_error("NativeOps::execScalarBool requires X & Y to have same type");

	if (!DataTypeUtils::isB(zType) )
		throw std::runtime_error("NativeOps::execScalarBool requires Z operand to have BOOL type");

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::scalar::ScalarBoolTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalar, extraParams), LIBND4J_TYPES, BOOL_TYPES);

	DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execScalarBool(Nd4jPointer *extraPointers,
						   int opNum,
						   void *hX, Nd4jLong *hXShapeInfo,
						   void *dX, Nd4jLong *dXShapeInfo,
						   void *hZ, Nd4jLong *hZShapeInfo,
						   void *dZ, Nd4jLong *dZShapeInfo,
						   void *hScalars, Nd4jLong *hScalarShapeInfo,
						   void *dScalars, Nd4jLong *dScalarShapeInfo,
						   void *extraParams,
							   void *hDimension, Nd4jLong *hDimensionShape,
							   void *dDimension, Nd4jLong *dDimensionShape,
                           Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                           Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims(256, 512, 8192);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (xType != yType )
		throw nd4j::datatype_exception::build("NativeOps::execScalarBool requires X & Y to have same type", xType, yType);

	if (!DataTypeUtils::isB(zType) )
		throw nd4j::datatype_exception::build("NativeOps::execScalarBool requires Z operand to have BOOL type", nd4j::DataType::BOOL, zType);

	BUILD_DOUBLE_SELECTOR(xType, yType, functions::scalar::ScalarBoolTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, BOOL_TYPES);

	DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execScalar(
		Nd4jPointer *extraPointers,
		int opNum,
		void *hX, Nd4jLong *hXShapeInfo,
		void *dX, Nd4jLong *dXShapeInfo,
		void *hZ, Nd4jLong *hZShapeInfo,
		void *dZ, Nd4jLong *dZShapeInfo,
		void *hScalar, Nd4jLong *hScalarShapeInfo,
		void *dScalar, Nd4jLong *dScalarShapeInfo,
		void *extraParams) {
	
	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims(256, 512, 8192);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
	if (yType != xType && yType != nd4j::DataType::BOOL && !this->isExperimentalEnabled())
		throw nd4j::datatype_exception::build("NativeOps::execScalar both operands must have same data type", xType, yType);

	if (!Environment::getInstance()->isExperimentalBuild() && Environment::getInstance()->isDebug()) {
        auto sX = DataTypeUtils::asString(xType);
        auto sY = DataTypeUtils::asString(yType);
        auto sZ = DataTypeUtils::asString(zType);

        nd4j_printf("Running execScalar with dtypes: [%s], [%s], [%s]\n", sX.c_str(), sY.c_str(), sZ.c_str());
    }


#ifdef __ND4J_EXPERIMENTAL__
	BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dZ, dZShapeInfo, hZShapeInfo, dScalar, extraParams), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dZ, dZShapeInfo, hZShapeInfo, dScalar, extraParams), LIBND4J_TYPES);
#endif

	DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execScalar(Nd4jPointer *extraPointers,
					 int opNum,
					 void *hX, Nd4jLong *hXShapeInfo,
                     void *dX, Nd4jLong *dXShapeInfo,
                     void *hZ, Nd4jLong *hZShapeInfo,
                     void *dZ, Nd4jLong *dZShapeInfo,
                     void *hScalars, Nd4jLong *hScalarShapeInfo,
                     void *dScalars, Nd4jLong *dScalarShapeInfo,
					 void *extraParams,
						   void *hDimension, Nd4jLong *hDimensionShape,
						   void *dDimension, Nd4jLong *dDimensionShape,
                     Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                     Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (yType != xType && yType != nd4j::DataType::BOOL && !this->isExperimentalEnabled())
		throw nd4j::datatype_exception::build("NativeOps::execScalar both operands must have same data type", xType, yType);

	dim3 launchDims(256, 256, 16384);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

	DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execAggregate(Nd4jPointer *extraPointers,
								   int opNum,
                                   void **arguments,
                                   int numArguments,
                                   Nd4jLong **shapes,
                                   int numShapes,
                                   int *indexArguments,
                                   int numIndexArguments,
                                   int **intArrays,
                                   int numIntArrays,
                                   void *realArguments,
                                   int numRealArguments,
                                   nd4j::DataType dtype) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    int numBlocks = getDeviceId(extraPointers[2]);
    int numThreads = getDeviceId(extraPointers[3]);
    int shmem = getDeviceId(extraPointers[4]);

    dim3 launchDims = dim3(numBlocks, numThreads, shmem);
	
    BUILD_SINGLE_SELECTOR(dtype, functions::aggregate::AggregatedFunction, ::aggregateKernelGeneric(launchDims, stream, opNum, arguments, numArguments, shapes, numShapes, indexArguments, numIndexArguments, intArrays, numIntArrays, realArguments, numRealArguments), FLOAT_TYPES);
    nd4j::DebugHelper::checkErrorCode(stream, "execAggregateFloat(...) failed");
}

void NativeOps::execAggregateBatch(Nd4jPointer *extraPointers, 
									int numAggregates, int opNum, 
									int maxArgs, int maxShapes, 
									int maxIntArrays, int maxIntArraySize, 
									int maxIdx, int maxReals,  
									void *ptrToArguments, nd4j::DataType dtype) {
    // not implemented yet
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    int numBlocks = getDeviceId(extraPointers[2]);
    int numThreads = getDeviceId(extraPointers[3]);
    int shmem = getDeviceId(extraPointers[4]);

    dim3 launchDims = dim3(numAggregates, numThreads, shmem);

	BUILD_SINGLE_SELECTOR(dtype, functions::aggregate::AggregatedFunction, ::aggregateBatchKernelGeneric(launchDims, stream, opNum, numAggregates, maxArgs, maxShapes, maxIntArrays, maxIntArraySize, maxIdx, maxReals, ptrToArguments), FLOAT_TYPES);

	DEBUG_KERNEL(stream, opNum);
}

void NativeOps::execRandom(Nd4jPointer *extraPointers, 
						  int opNum,
                          Nd4jPointer stateHost,
                          void *hZ, Nd4jLong *hZShapeInfo,
                          void *dZ, Nd4jLong *dZShapeInfo,
                          void *extraArguments) {

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto sizeOf = sizeof(nd4j::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    dim3 launchDims = dim3(512, 512, 32768);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    // functions::random::RandomFunction<float>::executeCudaSingle(launchDims, extraPointers, opNum, stateHost, dZ, dZShapeInfo, extraArguments),
    BUILD_SINGLE_SELECTOR(zType, functions::random::RandomFunction, ::executeCudaSingle(launchDims, extraPointers, opNum, stateDevice, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    checkCudaErrors(hipMemcpyAsync(stateHost, stateDevice, sizeOf, hipMemcpyDeviceToHost, *stream));
    checkCudaErrors(hipStreamSynchronize(*stream));
    hipFree(stateDevice);
}

void NativeOps::execRandom(Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, 
						   void *hX, Nd4jLong *hXShapeInfo, 
						   void *dX, Nd4jLong *dXShapeInfo, 
						   void *hZ, Nd4jLong *hZShapeInfo, 
						   void *dZ, Nd4jLong *dZShapeInfo, 
						   void *extraArguments) {
    
    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    auto sizeOf = sizeof(nd4j::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    dim3 launchDims = dim3(512, 512, 32768);
    auto xType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    // functions::random::RandomFunction<float>::executeCudaDouble(launchDims, extraPointers, opNum, stateHost, dX, dXShapeInfo, dZ, dZShapeInfo, extraArguments);
    BUILD_SINGLE_SELECTOR(xType, functions::random::RandomFunction, ::executeCudaDouble(launchDims, extraPointers, opNum, stateDevice, dX, dXShapeInfo, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    checkCudaErrors(hipMemcpyAsync(stateHost, stateDevice, sizeOf, hipMemcpyDeviceToHost, *stream));
    checkCudaErrors(hipStreamSynchronize(*stream));
    hipFree(stateDevice);
}

void NativeOps::execRandom(Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, 
							void *hX, Nd4jLong *hXShapeInfo, 
							void *dX, Nd4jLong *dXShapeInfo, 
							void *hY, Nd4jLong *hYShapeInfo, 
							void *dY, Nd4jLong *dYShapeInfo, 
							void *hZ, Nd4jLong *hZShapeInfo, 
							void *dZ, Nd4jLong *dZShapeInfo, 
							void *extraArguments) {

    auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto sizeOf = sizeof(nd4j::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    dim3 launchDims = dim3(512, 512, 32768);
    auto xType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    // functions::random::RandomFunction<float>::executeCudaTriple(launchDims, extraPointers, opNum, stateHost, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraArguments);
    BUILD_SINGLE_SELECTOR(xType, functions::random::RandomFunction, ::executeCudaTriple(launchDims, extraPointers, opNum, stateDevice, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    checkCudaErrors(hipMemcpyAsync(stateHost, stateDevice, sizeOf, hipMemcpyDeviceToHost, *stream));
    checkCudaErrors(hipStreamSynchronize(*stream));
    hipFree(stateDevice);
}


Nd4jPointer NativeOps::initRandom(Nd4jPointer *extraPointers, long seed, long bufferSize, Nd4jPointer ptrToBuffer) {

    unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    // we don't synchronize at random initialization, it's safe to go unsync here
	// hipStreamSynchronize(*stream);

    auto ptrDev = reinterpret_cast<unsigned long long *>(ptrToBuffer);
    auto buffer = new nd4j::random::RandomBuffer(seed, bufferSize, reinterpret_cast<uint64_t *>(ptrHost), reinterpret_cast<uint64_t *>(ptrDev));
    buffer->propagateToDevice(buffer, *stream);

    nd4j::DebugHelper::checkErrorCode(stream, "initRandom(...) failed A");

	// we generate sequence in the host memory
    nd4j::random::Xoroshiro128 generator(buffer);
    generator.refreshBuffer();

	// and copy it to gpu
    hipMemcpyAsync(ptrDev, ptrHost, bufferSize * 8, hipMemcpyHostToDevice, *stream);
    nd4j::DebugHelper::checkErrorCode(stream, "initRandom(...) failed B");

    return buffer;
}


void NativeOps::destroyRandom(Nd4jPointer ptrBuffer) {
    
    nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (ptrBuffer);

    // FIXME: it's bad thing, but we can't know in advance, which stream(s) where using this generator in practice
    hipDeviceSynchronize();

    delete buffer;
}

void NativeOps::refreshBuffer(Nd4jPointer *extraPointers, long seed, Nd4jPointer ptrRandom) {
    
    nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (ptrRandom);

    unsigned long long *ptrHost = reinterpret_cast<unsigned long long *>(extraPointers[0]);
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    hipStreamSynchronize(*stream);

    uint64_t *ptrDev = buffer->getDeviceBuffer();

	// update rng state
    buffer->setSeed(seed);
    buffer->setOffset(0);
    buffer->propagateToDevice(buffer, *stream);

	// refresh buffer on host size
    nd4j::random::Xoroshiro128 generator(buffer);
    generator.refreshBuffer();

	// copy back to gpu
    hipMemcpyAsync(ptrDev, ptrHost, buffer->getSize() * 8, hipMemcpyHostToDevice, *stream);
}

void NativeOps::reSeedBuffer(Nd4jPointer *extraPointers, long seed, Nd4jPointer ptrRandom) {
    
    nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (ptrRandom);

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    hipStreamSynchronize(*stream);

	// update rng state
    buffer->reSeed(seed);
    buffer->setOffset(0);
    buffer->propagateToDevice(buffer, *stream);
}



/**
    * Return the length of a shape buffer
    * based on the pointer
    * @param buffer  the buffer pointer to check
    * @return
    */
int NativeOps::lengthForShapeBufferPointer(Nd4jPointer buffer) {
    auto shapeBuffer = reinterpret_cast<Nd4jLong *>(buffer);
    return shape::shapeInfoLength(shape::rank(shapeBuffer));
}


/**
  * The pointer to get the address for
  *
  * @param address the address to get the pointer
  * @return the pointer for the given address
  */

Nd4jPointer NativeOps::pointerForAddress(Nd4jLong address) {
	return reinterpret_cast<Nd4jPointer >(address);
}

void NativeOps::tear(Nd4jPointer *extras,
					 void *x, Nd4jLong *xShapeInfo,
					 void *dX, Nd4jLong *dXShapeInfo,
					 Nd4jPointer *targets,
					 Nd4jLong *zShapeInfo,
					 Nd4jLong *tadShapeInfo,
					 Nd4jLong *tadOffsets) {
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extras[1]);
    dim3 launchDims(512, 512, 512);   
    auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, tearKernelGeneric, (launchDims, stream, dX, dXShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "tearFloat(...) failed");
}


void prescanArrayRecursive(Nd4jPointer *extras, int *dZ, int *dX, int numElements, int level) {

    auto stream = reinterpret_cast<hipStream_t *>(&extras[1]);
    auto g_scanBlockSums = reinterpret_cast<int **>(&extras[2]);

    int blockSize = 512; // max size of the thread blocks
    int numBlocks = nd4j::math::nd4j_max<int>(1, static_cast<int>(ceil(static_cast<float>(numElements) / (2.f * blockSize))));
    int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (nd4j::isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = nd4j::floorPow2(numElements);

    int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    int numEltsLastBlock =
            numElements - (numBlocks-1) * numEltsPerBlock;
    int numThreadsLastBlock = nd4j::math::nd4j_max<int>(1, numEltsLastBlock / 2);
    int np2LastBlock = 0;
    int sharedMemLastBlock = 0;

    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);

        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = sizeof(int) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    int extraSpace = numEltsPerBlock / NUM_BANKS;
    int sharedMemSize = sizeof(int) * (numEltsPerBlock + extraSpace);

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3 grid(max(1, numBlocks - np2LastBlock), 1, 1);
    dim3 threads(numThreads, 1, 1);
    dim3 gridOnes(1, 1, 1);
    dim3 threadsOnes(numThreadsLastBlock, 1, 1);

    if (sharedMemSize < 2048)
        sharedMemSize = 2048;

    if (sharedMemLastBlock < 2048)
        sharedMemLastBlock = 2048;

    // execute the scan
    if (numBlocks > 1) {
        nd4j::prescanLauncher<true, false>(grid, threads, sharedMemSize, stream, dZ, dX, g_scanBlockSums[level], numThreads * 2, 0, 0);
        if (np2LastBlock) {
            nd4j::prescanLauncher<true, true>(gridOnes, threadsOnes, sharedMemLastBlock, stream, dZ, dX, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we
        // need to take all of the last values of the sub-blocks and scan those.
        // This will give us a new value that must be sdded to each block to
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(extras, g_scanBlockSums[level], g_scanBlockSums[level], numBlocks, level+1);

        nd4j::uniformAdd<<<grid, threads, 1024, *stream>>>(dZ, g_scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);

        if (np2LastBlock) {
            nd4j::uniformAdd<<<1, numThreadsLastBlock, 1024, *stream>>>(dZ, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }
    } else if (isPowerOfTwo(numElements)) {
        nd4j::prescanLauncher<false, false>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numThreads * 2, 0, 0);
    } else {
        nd4j::prescanLauncher<false, true>(grid, threads, sharedMemSize, stream, dZ, dX, 0, numElements, 0, 0);
    }
}


void NativeOps::encodeThresholdP1(Nd4jPointer *extras, void *dx, Nd4jLong *hXShapeInfo, Nd4jLong N, int *dz, float threshold) {
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extras[1]);

    int blockSize = 1024;
    int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);
    
    dim3 launchDims(numBlocks, blockSize, 1024);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, encoderKernelP1Generic, (launchDims, stream, dx, N, dz, threshold), LIBND4J_TYPES);        

    nd4j::DebugHelper::checkErrorCode(stream, "encodeThresholdP1Float(...) failed");
}



void NativeOps::encodeThresholdP2Int(Nd4jPointer *extraPointers, int *dx, Nd4jLong N, int *dz) {
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    //encoderKernelP2Float<<<numBlocks, blockSize , 1024 * sizeof(float), *stream>>>(dx, N, dz);    
    prescanArrayRecursive(extraPointers, dz, dx + 1, (int) N, 0);
    nd4j::DebugHelper::checkErrorCode(stream, "encodeThresholdP2Int(...) failed");
}

void NativeOps::encodeThresholdP3(Nd4jPointer *extraPointers, void *dx, Nd4jLong *hXShapeInfo, int *offsets, Nd4jLong N, int *dz){
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    int blockSize = 1024;
    int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);
    
    dim3 launchDims(numBlocks, blockSize, 4096);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, encoderKernelP3Generic, (launchDims, stream, dx, offsets, N, dz), LIBND4J_TYPES);    

    nd4j::DebugHelper::checkErrorCode(stream, "encodeThresholdP3Float(...) failed");
}

void NativeOps::decodeThreshold(Nd4jPointer *extraPointers, void *dx, Nd4jLong N, void *dz, Nd4jLong *zShapeInfo){
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

    // we probably want to have smaller blocks here, memory writes are misaligned anyway
    int blockSize = 128;
    int numBlocks = N / blockSize + (N % blockSize ? 1 : 0);
    
    dim3 launchDims(numBlocks, blockSize, 1024);
    auto zType = nd4j::ArrayOptions::dataType(zShapeInfo);
    BUILD_SINGLE_SELECTOR(zType, decoderKernelGeneric, (launchDims, stream, dx, N, dz), LIBND4J_TYPES);    

    nd4j::DebugHelper::checkErrorCode(stream, "decodeThresholdFloat(...) failed");
}


void NativeOps::execReduce3All(Nd4jPointer *extraPointers,
									int opNum,
									void *hX, Nd4jLong *hXShapeInfo,
                            		void *dX, Nd4jLong *dXShapeInfo,
                            		void *extraParamsVals,
									void *hY, Nd4jLong *hYShapeInfo,
                            		void *dY, Nd4jLong *dYShapeInfo,
                            		void *hZ, Nd4jLong *hZShapeInfo,
                            		void *dZ, Nd4jLong *dZShapeInfo,
							   		void *hDimension, Nd4jLong *hDimensionShape,
							   		void *dDimension, Nd4jLong *dDimensionShape,
									Nd4jLong *xTadShapeInfo, Nd4jLong *xOffsets,
									Nd4jLong *yTadShapeInfo, Nd4jLong *yOffsets) {
	auto dimension = reinterpret_cast<int *>(dDimension);
	int dimensionLength = static_cast<int>(shape::length(hDimensionShape));

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    auto hTADShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[9]);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("D119 opNum:[%i]\n", opNum);

    int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
    double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

    dim3 launchDims(shape::length(hZShapeInfo), 256, 32768);

    if (nd4j::Environment::getInstance()->isVerbose() && launchDims.x == 1)
        printf("AD119 opNum:[%i]\n", opNum);
    
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (yType != xType && yType != nd4j::DataType::BOOL && !this->isExperimentalEnabled())
		throw nd4j::datatype_exception::build("NativeOps::execReduce3All both operands must have same data type", xType, yType);

    if (yType != xType)
        throw nd4j::datatype_exception::build("NativeOps::execReduce3All both operands must have same data type", xType, yType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execAll(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParamsVals, dZ, dZShapeInfo, dimension, dimensionLength, 1, allocationPointer, xTadShapeInfo, xOffsets, yTadShapeInfo, yOffsets), LIBND4J_TYPES, FLOAT_TYPES);

	DEBUG_KERNEL(stream, opNum);
}


void NativeOps::sort(Nd4jPointer *extraPointers,
					 void *x, Nd4jLong *xShapeInfo,
					 void *dX, Nd4jLong *dXShapeInfo,
					 bool descending) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[     1]);

    auto xLength = shape::length(xShapeInfo);
    auto xEWS = shape::elementWiseStride(xShapeInfo);
    auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);


    // check if xLength is a power of 2, and use bitonic sort, if that's the case
    if ((xLength != 0) && ((xLength & (xLength - 1)) == 0) && (xLength <= 1024 * 1024 * 10)) {
        int numThreads = nd4j::math::nd4j_min<int>(512, xLength);
        int numBlocks = xLength / numThreads;
        if (xLength % numThreads > 0 || numBlocks == 0)
            numBlocks++;

        dim3 launchDims(numBlocks, numThreads, 32768);

        for (int k = 2; k <= xLength; k = 2*k) {
            for (int j = k >> 1; j > 0; j = j >> 1) {
				BUILD_SINGLE_SELECTOR(xType, bitonicSortStepGeneric, (launchDims, stream, dX, dXShapeInfo, j, k, xLength, descending), LIBND4J_TYPES);
			}
        }
    } else {
    	int numThreads = nd4j::math::nd4j_min<int>(512, xLength);
    	int numBlocks = xLength / numThreads;
    	if (xLength % numThreads > 0 || numBlocks == 0)
    		numBlocks++;

    	numBlocks = nd4j::math::nd4j_min<int>(512, numBlocks);
    	dim3 launchDims(numBlocks, numThreads, 32768);

    	int max = 2, dg = 0;
    	while (max < xLength) {
    		max <<= 1;
    		dg++;
    	}
    	max <<= 1;

    	for (int window = 2; window < max; window<<=1) {
    		int n = window;
    		int rev = 0;
    		do{
    			int half = n >> 1;
    			BUILD_SINGLE_SELECTOR(xType, bitonicArbitraryStepGeneric, (launchDims, stream, dX, dXShapeInfo, n, xLength, rev, descending), LIBND4J_TYPES);
    			n>>=1;
    			rev = 1;
    		} while(n > 1);
    	}
    }

    nd4j::DebugHelper::checkErrorCode(stream, "sort(...) failed");
}


void NativeOps::sortTad(Nd4jPointer *extraPointers,
						void *x, Nd4jLong *xShapeInfo,
						void *dX, Nd4jLong *dXShapeInfo,
						int *dimension,
						int dimensionLength,
						Nd4jLong *tadShapeInfo,
						Nd4jLong *tadOffsets,
						bool descending) {
    // to be implemented
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    dim3 launchDims(512, 512, 32768);
	auto xType = nd4j::ArrayOptions::dataType(xShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, oesTadGeneric, (launchDims, stream, dX, dXShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, descending), LIBND4J_TYPES);                     
    
    nd4j::DebugHelper::checkErrorCode(stream, "sortTadFloat(...) failed");
}

void NativeOps::sortCooIndices(Nd4jPointer *extraPointers, Nd4jLong *indices, void *values, Nd4jLong length, int rank) {
	throw std::runtime_error("sortCooIndices:: Not implemented yet");
}


Nd4jLong NativeOps::encodeBitmap(Nd4jPointer *extraPointers, 
								void *dx, Nd4jLong *hXShapeInfo,
								Nd4jLong N, 
								int *dz, 
								float threshold) {
    
    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    int *resultPointer = reinterpret_cast<int *>(extraPointers[2]);
    int *reductionPointer = reinterpret_cast<int *>(extraPointers[3]);
        
    dim3 launchDims(512, 512, 32768);
    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, cudaEncodeBitmapGeneric, (launchDims, stream, dx, N, dz, resultPointer, reductionPointer, threshold), LIBND4J_TYPES);     

    nd4j::DebugHelper::checkErrorCode(stream, "encodeBitmapFloat(...) failed");

    Nd4jLong dZ = (Nd4jLong) resultPointer[0];
    resultPointer[0] = 0;

    return dZ;
}


void NativeOps::decodeBitmap(Nd4jPointer *extraPointers, 
							void *dx,
							Nd4jLong N, 
							void *dz, Nd4jLong *zShapeInfo) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);        
    dim3 launchDims(512, 512, 16384);
    auto xType = nd4j::ArrayOptions::dataType(zShapeInfo);
    BUILD_SINGLE_SELECTOR(xType, cudaDecodeBitmapGeneric, (launchDims, stream, dx, N, dz), LIBND4J_TYPES);

    nd4j::DebugHelper::checkErrorCode(stream, "decodeBitmapFloat(...) failed");
}

Nd4jLong* NativeOps::mmapFile(Nd4jPointer *extraPointers, const char *fileName, Nd4jLong length) {
	return nullptr;
}

void NativeOps::munmapFile(Nd4jPointer *extraPointers, Nd4jLong* ptrMap, Nd4jLong length) {

}


nd4j::graph::ResultWrapper* NativeOps::executeFlatGraph(Nd4jPointer *extraPointers, Nd4jPointer flatBufferPointer) {
	return nullptr;
}


const char* NativeOps::getAllCustomOps() {
	return nd4j::ops::OpRegistrator::getInstance()->getAllCustomOperations();
}


nd4j::ShapeList* _calculateOutputShapes(Nd4jPointer* extraPointers, nd4j::ops::DeclarableOp* op, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool *bArgs, int numBArgs) {
    nd4j::graph::VariableSpace varSpace;
    Context block(2, &varSpace);
    nd4j::ShapeList inShapes;

    for (int e = 0; e < numIArgs; e++)
        block.getIArguments()->push_back(iArgs[e]);

    for (int e = 0; e < numTArgs; e++)
        block.getTArguments()->push_back(tArgs[e]);

	for (int e = 0; e < numBArgs; e++)
		block.getBArguments()->push_back(bArgs[e]);

	for (int e = 0; e < numInputShapes; e++) {
		auto shape_ = reinterpret_cast<Nd4jLong *>(inputShapes[e]);

		// we shouldn't copy buffer if that's empty array
		void *buffer_ = nd4j::ArrayOptions::arrayType(shape_) == ArrayType::EMPTY ? nullptr : inputBuffers[e];

		auto array = new nd4j::NDArray(buffer_, shape_);
		array->triggerAllocationFlag(false, false);

		// block should contain references to proper variable
		varSpace.putVariable(1, e, array);
		block.pickInput(1, e);

		inShapes.push_back(shape_);
	}

    auto shapeList = op->calculateOutputShape(&inShapes, block);

    if (varSpace.workspace() != nullptr)
        shapeList->detach();

    return shapeList;
}

nd4j::ShapeList* NativeOps::calculateOutputShapes(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool *bArgs, int numBArgs) {
    auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);

    return _calculateOutputShapes(extraPointers, op, inputBuffers, inputShapes, numInputShapes, tArgs, numTArgs, iArgs, numIArgs, bArgs, numBArgs);
}

nd4j::ShapeList* _calculateOutputShapes(Nd4jPointer* extraPointers, nd4j::ops::DeclarableOp* op, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs) {
    Context block(1);
	nd4j::ShapeList inShapes;

	for (int e = 0; e < numIArgs; e++)
		block.getIArguments()->push_back(iArgs[e]);

	for (int e = 0; e < numTArgs; e++)
		block.getTArguments()->push_back(tArgs[e]);

	for (int e = 0; e < numInputShapes; e++)
		inShapes.push_back(reinterpret_cast<Nd4jLong *>(inputShapes[e]));

	auto shapeList = op->calculateOutputShape(&inShapes, block);

	return shapeList;
}

nd4j::ShapeList* NativeOps::calculateOutputShapes(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputShapes, int numInputShapes, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs) {
	auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);

	return _calculateOutputShapes(extraPointers, op, inputShapes, numInputShapes, tArgs, numTArgs, iArgs, numIArgs);
}


static FORCEINLINE Nd4jStatus realExec(nd4j::ops::DeclarableOp* op, Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputs, Nd4jPointer* outputBuffers, Nd4jPointer* outputShapes, int numOutputs, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool* bArgs, int numBArgs, bool isInplace) {
	if (op == nullptr)
		nd4j_printf("Can't find requested operation: [%lld]\n", hash);

	// we're using the same fake nodeId everywhere here

	std::vector<nd4j::NDArray*> inputs(numInputs);
	std::vector<nd4j::NDArray*> outputs(numOutputs);
	std::vector<double> ttArgs(numTArgs);
	std::vector<bool> bbArgs(0);
	std::vector<Nd4jLong> iiArgs(numIArgs);

	// filling block now with inputs
	for (int e = 0; e < numInputs; e++) {
		auto shape = reinterpret_cast<Nd4jLong *>(inputShapes[e]);
		void *buffer = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[e];

		inputs[e] = new nd4j::NDArray(buffer, shape);
	}

	// if not inplace - transferring output arrays

	if (!isInplace)
		for (int e = 0; e < numOutputs; e++) {
			// we want to keep original output shape intact
			auto shape = shape::copyShape(reinterpret_cast<Nd4jLong *>(outputShapes[e]));
			void *buffer = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : outputBuffers[e];

			// FIXME: revisit this.
			bool canNullify = true;
			for (int i = 0; i < numInputs; i++) {
				void *ibuffer = nd4j::ArrayOptions::arrayType(shape) == ArrayType::EMPTY ? nullptr : inputBuffers[i];
				if (ibuffer == buffer) {
					canNullify = false;
					break;
				}
			}

			if (canNullify)
				memset((uint8_t *) buffer, '\0', shape::length(shape) * DataTypeUtils::sizeOfElement(ArrayOptions::dataType(shape)));

			auto array = new nd4j::NDArray(buffer, shape);
			outputs[e] = array;

			// and we want to release shape copy once we're done
			array->triggerAllocationFlag(false, true);
		}

	for (int e = 0; e < numIArgs; e++)
		iiArgs[e] = iArgs[e];


	for (int e = 0; e < numTArgs; e++)
		ttArgs[e] = tArgs[e];


	// hypothetically at this point we have everything filled
	auto dZ = op->execute(inputs, outputs, ttArgs, iiArgs, bbArgs, isInplace);
	//auto dZ = op->execute(inputs, ttArgs, iiArgs, isInplace);


	if (!isInplace)
		for (int e = 0; e < numOutputs; e++) {
			//shape::printShapeInfoLinear("JVM output shape", (int *) outputShapes[e]);
			//shape::printShapeInfoLinear("C++ output shape", (int *) outputs[e]->shapeInfo());
			//outputs[e]->printIndexedBuffer("C++ raw output");
			//outputs[e]->printBuffer("C++ indexed output");

			if (outputs[e]->ordering() != shape::order(reinterpret_cast<Nd4jLong *>(outputShapes[e])))
				outputs[e]->streamline(shape::order(reinterpret_cast<Nd4jLong *>(outputShapes[e])));
		}

/*
    if (!isInplace) {
        if (dZ->size() != numOutputs) {
            return ND4J_STATUS_BAD_OUTPUT;
        }

        for (int e = 0; e < numOutputs; e++) {
            auto buffer = (T *) outputBuffers[e];
            auto shape = (int *) outputShapes[e];
            nd4j::NDArray<T> tmp(buffer, shape);

            if (tmp.lengthOf() != dZ->at(e)->lengthOf()) {
                nd4j_printf("Provided output array for [%s] has length of %i, but actual dZ has length of %i\n", op->getOpName()->c_str(), tmp.lengthOf(), dZ->at(e)->lengthOf());
                return ND4J_STATUS_BAD_OUTPUT;
            }

            tmp.assign(dZ->at(e));
        }
    } else {
        // if op is inplace, our ResultSet holds pointers
        dZ->purge();
    }


    delete dZ;

*/

	for (auto v: inputs)
		delete v;

	for (auto v: outputs)
		delete v;

	return Status::OK();
}


int NativeOps::execCustomOp(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer* inputBuffers, Nd4jPointer* inputShapes, int numInputs, Nd4jPointer* outputBuffers, Nd4jPointer* outputShapes, int numOutputs, double* tArgs, int numTArgs, Nd4jLong *iArgs, int numIArgs, bool* bArgs, int numBArgs, bool isInplace) {
	auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);

	return realExec(op, extraPointers, hash, inputBuffers, inputShapes, numInputs, outputBuffers, outputShapes, numOutputs, tArgs, numTArgs, iArgs, numIArgs, bArgs, numBArgs, isInplace);
}

int NativeOps::execCustomOp(Nd4jPointer* extraPointers, Nd4jLong hash, Nd4jPointer opContext) {
    auto op = nd4j::ops::OpRegistrator::getInstance()->getOperation(hash);
    auto context = reinterpret_cast<Context*>(opContext);

    return op->execute(context);
}

int NativeOps::registerGraph(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer flatBufferPointer) {
	
	auto graph = nd4j::graph::GraphExecutioner::importFromFlatPointer(flatBufferPointer);

	nd4j::graph::GraphHolder::getInstance()->registerGraph(graphId, graph);

	return ND4J_STATUS_OK;
}


static VariablesSet* executeStoredGraphT(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int* inputIndices, int numInputs) {
	auto graph = nd4j::graph::GraphHolder::getInstance()->pullGraph(graphId);
	auto varSpace = graph->getVariableSpace()->clone();

	std::vector<nd4j::NDArray*> handles;

	for (int e = 0; e < numInputs; e++) {
		auto idx = inputIndices[e];

		// we'll delete this array later, together with cloned VariableSpace
		auto array = new nd4j::NDArray(inputBuffers[e], reinterpret_cast<Nd4jLong *>(inputShapes[e]));
		handles.emplace_back(array);

		if (varSpace->hasVariable(idx)) {
			auto var = varSpace->getVariable(idx);
			if (var->hasNDArray())
				delete var->getNDArray();

			var->setNDArray(array);
		} else
			varSpace->putVariable(idx, array);
	}

	auto dZ = nd4j::graph::GraphExecutioner::execute(graph, varSpace);
	auto varSet = new nd4j::graph::VariablesSet(dZ);

	if (dZ == ND4J_STATUS_OK) {
		// pull back results, and provide them
		auto outputs = graph->fetchOutputs();
		for (int e = 0; e < outputs->size(); e++) {
			// we're only getting variable ID/Index from original grap. values will be taken from cloned workspace
			std::pair<int, int> varId(outputs->at(e)->id(), outputs->at(e)->index());

			auto var = varSpace->getVariable(varId);

			varSet->push_back(var->clone());
		}

		delete outputs;
	}

	delete varSpace;

	return varSet;
}

VariablesSet* NativeOps::executeStoredGraph(Nd4jPointer *extraPointers, Nd4jLong graphId, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int* inputIndices, int numInputs) {
	return executeStoredGraphT(extraPointers, graphId, inputBuffers, inputShapes, inputIndices, numInputs);
}

int NativeOps::unregisterGraph(Nd4jPointer *extraPointers, Nd4jLong graphId) {

	nd4j::graph::GraphHolder::getInstance()->dropGraphAny(graphId);

	return ND4J_STATUS_OK;
}

void NativeOps::deletePointerArray(Nd4jPointer pointer) {
    Nd4jPointer *ptr = reinterpret_cast<Nd4jPointer *>(pointer);
    delete[] ptr;
}

void NativeOps::deleteIntArray(Nd4jPointer pointer) {
	auto ptr = reinterpret_cast<int *>(pointer);
	delete[] ptr;
}

void NativeOps::deleteLongArray(Nd4jPointer pointer) {
	auto ptr = reinterpret_cast<Nd4jLong *>(pointer);
	delete[] ptr;
}

template <typename T>
static void deleteVariablesSetT(Nd4jPointer pointer) {
	nd4j::graph::VariablesSet* ptr = reinterpret_cast<nd4j::graph::VariablesSet*>(pointer);
	delete ptr;
}

void NativeOps::deleteVariablesSet(Nd4jPointer pointer) {
	deleteVariablesSetT<double>(pointer);
}

void NativeOps::deleteShapeList(Nd4jPointer shapeList) {
    nd4j::ShapeList* list = reinterpret_cast<nd4j::ShapeList*>(shapeList);

    list->destroy();
    delete list;
}

const char* NativeOps::getAllOperations() {
    return nd4j::OpTracker::getInstance()->exportOperations();
}

Nd4jPointer NativeOps::getGraphState(Nd4jLong id) {
    return (Nd4jPointer) new nd4j::graph::GraphState(id);
}


void NativeOps::deleteGraphState(Nd4jPointer state) {
    auto stateP = reinterpret_cast<nd4j::graph::GraphState*>(state);
    delete stateP;
}


Nd4jStatus execCustomOpWithScope(Nd4jPointer *extraPointers, nd4j::graph::GraphState *state, Nd4jLong opHash, Nd4jLong *scopes, int numScopes, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int numInputs, Nd4jPointer *outputBuffers, Nd4jPointer *outputShapes, int numOutputs) {
    /**
     * That's basically exec, with VariableSpace provided in GraphState:
     * depending on operation (i.e. while of if), different logic executors could be used
     */

    auto graph = state->graph();
    auto varSpace = state->variableSpace();

    // Node is dynamically created, and has nothing beyond it: only inputs and outputs
    // this node has id of 0, and inputs are
    Node node(OpType_LOGIC, opHash, 0);

    // mapping inputs
    for (int e = 0; e < numInputs; e++) {
        auto buffer = inputBuffers[e];
        auto shapeInfo = reinterpret_cast<Nd4jLong *>(inputShapes[e]);

        auto array = new nd4j::NDArray(buffer, shapeInfo, varSpace->workspace());

        // now we just put array to VarSpace
        varSpace->putVariable(0, e, array);
        node.pickInput(0, e);
    }

    // mapping scopes
    for (int e = 0; e < numScopes; e++) {
        // we should check scope existence in GraphState/Graph
        int scopeId = (int) scopes[e];
        if (!state->hasScope(scopeId)) {
            // nd4j_printf("execCustomOpWithScope: referenced scope [%i] doesn't exist\n", scopeId);
            return Status::THROW();
        }
        node.pickInput(scopeId, 0);
    }

    auto dZ = LogicExecutor::processNode(graph, &node);
    if (dZ != Status::OK())
        return dZ;

    // mapping outputs

    for (int e = 0; e < numOutputs; e++) {
        auto buffer = outputBuffers[e];
        auto shapeInfo = reinterpret_cast<Nd4jLong *>(outputShapes[e]);

        NDArray array(buffer, shapeInfo, varSpace->workspace());

        // now we just put array to VarSpace to the same ID
        //varSpace->putVariable(0, e, array);

        auto t = varSpace->getVariable(0, e)->getNDArray();
        array.assign(t);
    }

    // removing input variables
    for (int e = 0; e < numInputs; e++) {
        varSpace->dropVariable(0, e);
    }

    // after some bla-bla-bla we should have Graph and Node for current op
    return Status::OK();
}

           
Nd4jStatus NativeOps::execCustomOpWithScope(Nd4jPointer *extraPointers, Nd4jPointer state, Nd4jLong opHash, Nd4jLong *scopes, int numScopes, Nd4jPointer *inputBuffers, Nd4jPointer *inputShapes, int numInputs, Nd4jPointer *outputBuffers, Nd4jPointer *outputShapes, int numOutputs) {
    
    return execCustomOpWithScope(extraPointers, reinterpret_cast<nd4j::graph::GraphState*>(state), opHash, scopes, numScopes, inputBuffers, inputShapes, numInputs, outputBuffers, outputShapes, numOutputs);
}

void NativeOps::deleteResultWrapper(Nd4jPointer ptr) {
	// just 0 room for compiler s@!t
	auto p = reinterpret_cast<nd4j::graph::ResultWrapper *>(ptr);
	delete p;
}

int NativeOps::estimateThreshold(Nd4jPointer *extraPointers, Nd4jPointer dX, Nd4jLong *dXShapeInfo, int N, float threshold) {
	throw std::runtime_error("estimateThreshold: Not implemented yet");
}

/*
 * TypeDef:
 *     void convertTypes(Nd4jPointer *extras, int srcType, Nd4jPointer dX, long N, int dstType, Nd4jPointer dZ);
 */
void NativeOps::convertTypes(Nd4jPointer *extras, int srcType, Nd4jPointer dX, Nd4jLong N, int dstType, Nd4jPointer dZ) {
 	auto dx = reinterpret_cast<void *>(dX);
	auto dz = reinterpret_cast<void *>(dZ);

    if (srcType == ND4J_FLOAT8) {
        if (dstType == ND4J_FLOAT8) {
            // convertKernel<double, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::int8>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::uint8>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::int16>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, nd4j::uint16>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {

        } else if (dstType == ND4J_FLOAT32) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::float8, double>(extras, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_INT8) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<nd4j::int8, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            //convertKernel<nd4j::int8, nd4j::int8>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<int8_t, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<int8_t, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<int8_t, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<int8_t, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO: eventually we might want to add it
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<int8_t, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<int8_t, double>(extras, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_UINT8) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<uint8_t, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO: still might want to add
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<uint8_t, double>(extras, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_FLOAT16) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<float16, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<float16, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<float16, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<float16, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<float16, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<float16, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO: .... ^^^
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<float16, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<float16, double>(extras, dx, N, dz);
        } else if (dstType == ND4J_THRESHOLD) {
            //nd4j::convertToThreshold<float16>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_INT16) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<int16_t, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<int16_t, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<int16_t, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<int16_t, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<int16_t, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<int16_t, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {
            // TODO...
        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<int16_t, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<int16_t, double>(extras, dx, N, dz);
        } else {
            printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_FLOAT24) {

    } else if (srcType == ND4J_FLOAT32) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<float, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<float, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<float, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<float, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<float, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<float, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {

        } else if (dstType == ND4J_DOUBLE) {
            nd4j::TypeCast::convertGenericCuda<float, double>(extras, dx, N, dz);
        } else if (dstType == ND4J_THRESHOLD) {
            //nd4j::convertToThreshold<float>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_DOUBLE) {
        if (dstType == ND4J_FLOAT8) {
            //nd4j::TypeCast::convertGenericCuda<double, nd4j::float8>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT8) {
            nd4j::TypeCast::convertGenericCuda<double, int8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT8) {
            nd4j::TypeCast::convertGenericCuda<double, uint8_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT16) {
            nd4j::TypeCast::convertGenericCuda<double, float16>(extras, dx, N, dz);
        } else if (dstType == ND4J_INT16) {
            nd4j::TypeCast::convertGenericCuda<double, int16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_UINT16) {
            nd4j::TypeCast::convertGenericCuda<double, uint16_t>(extras, dx, N, dz);
        } else if (dstType == ND4J_FLOAT24) {

        } else if (dstType == ND4J_FLOAT32) {
            nd4j::TypeCast::convertGenericCuda<double, float>(extras, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            //
        } else if (dstType == ND4J_THRESHOLD) {
            //nd4j::convertToThreshold<double>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else if (srcType == ND4J_THRESHOLD) {
        if (dstType == ND4J_FLOAT16) {
            //nd4j::convertFromThreshold<float16>(nullptr, dx, N, dz);
        } else if (dstType == ND4J_FLOAT32) {
            //nd4j::convertFromThreshold<float>(nullptr, dx, N, dz);
        } else if (dstType == ND4J_DOUBLE) {
            //nd4j::convertFromThreshold<double>(nullptr, dx, N, dz);
        } else {
            nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
        }
    } else {
        nd4j_printf("Unsupported types conversion: [%i] -> [%i]\n", srcType, dstType);
    }
}

Nd4jPointer NativeOps::createUtf8String(Nd4jPointer *extraPointers, const char *string, int length) {
    auto u = new nd4j::utf8string(string, length);
    return reinterpret_cast<Nd4jPointer>(u);
}

void NativeOps::deleteUtf8String(Nd4jPointer *extraPointers, Nd4jPointer ptr) {
    delete(reinterpret_cast<nd4j::utf8string*>(ptr));
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void scatterUpdateCuda(const int opCode, const int numOfSubArrs, 
										      void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
										      void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
										      const int* indexes) {
        
    __shared__ T *x, *y;
    __shared__ Nd4jLong arrLenX, arrLenY;

    for (int e = 0; e < numOfSubArrs; e++ ) {
        
        const auto xIndex = indexes[e];
        const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

        if (!isOwner)
            continue;

        if (threadIdx.x == 0) {
            x = reinterpret_cast<T*>(vx) + xOffsets[xIndex];
            y = reinterpret_cast<T*>(vy) + yOffsets[e];
            arrLenX = shape::length(xShapeInfo);
            arrLenY = shape::length(yShapeInfo);
        }

        __syncthreads();

        if (arrLenX != arrLenY)
            return;

        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

            const auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLenX);
            const auto yOffset = shape::getIndexOffset(i, yShapeInfo, arrLenY);

            switch (opCode) {
                case 0:
                    x[xOffset] += y[yOffset];
                    break;
                case 1:
                    x[xOffset] -= y[yOffset];
                    break;
                case 2:
                    x[xOffset] *= y[yOffset];
                    break;
                case 3:
                    x[xOffset] /= y[yOffset];
                    break;
                case 4:
                    x[xOffset] = y[yOffset] - x[xOffset];
                    break;
                case 5:
                    x[xOffset] = y[yOffset] / x[xOffset];
                    break;
                case 6:
                    x[xOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        __syncthreads();
    }
}

template<typename T>
__host__ static void scatterUpdateCudaLauncher(const hipStream_t* stream, const int opCode, const int numOfSubArrs, void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets, void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets, const int* indexes) {

    scatterUpdateCuda<T><<<512, 256, MAX_NUM_THREADS, *stream>>>(opCode, numOfSubArrs, vx, xShapeInfo, xOffsets, vy, yShapeInfo, yOffsets, indexes);
}


//////////////////////////////////////////////////////////////////////////
void NativeOps::scatterUpdate(Nd4jPointer *extraPointers, int opCode, int numOfSubArrs,
                      			void* hX, Nd4jLong* hXShapeInfo, Nd4jLong* hXOffsets,
                      			void* dX, Nd4jLong* dXShapeInfo, Nd4jLong* dXOffsets,
                      			void* hY, Nd4jLong* hYShapeInfo, Nd4jLong* hYOffsets,
                      			void* dY, Nd4jLong* dYShapeInfo, Nd4jLong* dYOffsets,
                      			int* hIindexes, int* dIndexes) {

	auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
		
	nd4j::DataType type = ArrayOptions::dataType(hXShapeInfo);

    BUILD_SINGLE_SELECTOR(type, scatterUpdateCudaLauncher, (stream, opCode, numOfSubArrs, dX, dXShapeInfo, dXOffsets, dY, dYShapeInfo, dYOffsets, dIndexes), LIBND4J_TYPES);
}

