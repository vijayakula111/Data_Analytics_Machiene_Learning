#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//  @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018

#ifndef PAIRWISE_BOOL_CU
#define PAIRWISE_BOOL_CU


#include "../pairwise_bool.h"


using namespace simdOps;

////////////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpType>
__global__ void pairwiseSimpleShaped(void* x, Nd4jLong *xShapeInfo, 
									void *y, Nd4jLong *yShapeInfo, 
									void *z, Nd4jLong *zShapeInfo, 
									void *params, 
									int *allocationBuffer) {
        
	functions::pairwise_transforms::PairWiseBoolTransform<X,Z>::template transformCuda<OpType>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, params, allocationBuffer, nullptr);
}



namespace functions           {
namespace pairwise_transforms {

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
void _CUDA_H PairWiseBoolTransform<X,Z>::intermediateShaped(dim3& launchDims, hipStream_t *stream, 
														void *vx, Nd4jLong *xShapeInfo, 
														void *vy, Nd4jLong *yShapeInfo, 
														void *vz, Nd4jLong *zShapeInfo, 
														void *vextraParams, 
														int *allocPointer){

	pairwiseSimpleShaped<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams, allocPointer);
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void PairWiseBoolTransform<X,Z>::transformCuda(Nd4jLong len,
														void *vx, void *vy,
														Nd4jLong xEws, Nd4jLong yEws,
														void *vparams,
														void *vz, Nd4jLong zEws,
														int *allocPointer, 
														Nd4jLong *tadOnlyShapeInfo) {
	auto x = reinterpret_cast<X*>(vx);
	auto y = reinterpret_cast<X*>(vy);
	auto z = reinterpret_cast<Z*>(vz);
	auto params = reinterpret_cast<X*>(vparams);

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(xEws == yEws && yEws == zEws && xEws == 1) {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) 
				z[i] = OpType::op(x[i], y[i], params);
	}
	else {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) 
			z[i * zEws] = OpType::op(x[i * xEws], y[i * yEws], params);			
	}
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void PairWiseBoolTransform<X,Z>::transformCuda(void *vx, Nd4jLong *xShapeInfo, 
														void *vy, Nd4jLong *yShapeInfo, 
														void *vz, Nd4jLong *zShapeInfo, 
														void *vextraParams, 
														int *allocPointer, 
														Nd4jLong *tadOnlyShapeInfo) {

	auto x = reinterpret_cast<X*>(vx);
	auto y = reinterpret_cast<X*>(vy);
	auto z = reinterpret_cast<Z*>(vz);
	auto extraParams = reinterpret_cast<X*>(vextraParams);

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int xEWS;
	__shared__ int yEWS;
	__shared__ int zEWS;

	__shared__ char xOrder;
	__shared__ char yOrder;
	__shared__ char zOrder;

	__shared__ bool xRow;
	__shared__ bool yRow;
	__shared__ bool zRow;

	if (threadIdx.x == 0) {
		
		xEWS = shape::elementWiseStride(xShapeInfo);
		yEWS = shape::elementWiseStride(yShapeInfo);
    	zEWS = shape::elementWiseStride(zShapeInfo);
		xOrder = shape::order(xShapeInfo);
		yOrder = shape::order(yShapeInfo);
		zOrder = shape::order(zShapeInfo);
		xRow = shape::isRowVector(xShapeInfo);
		yRow = shape::isRowVector(yShapeInfo);
		zRow = shape::isRowVector(zShapeInfo);
	}
	
	__syncthreads();

	Nd4jLong len = shape::length(xShapeInfo);
	if((xEWS >= 1 && yEWS == xEWS && zEWS == xEWS &&  xOrder == yOrder && zOrder == xOrder) || (xEWS >= 1 && yEWS == xEWS && zEWS == xEWS && xRow && yRow && zRow)) {
		// TODO: this is wrong, and should be moved to host side
		transformCuda<OpType>(len, x, y, xEWS, yEWS, extraParams, z, zEWS, allocPointer, tadOnlyShapeInfo);
    } 
    else {

    	if (vx == vz) {
			
			for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {

				auto xOffset = shape::getIndexOffset(i, xShapeInfo, len);
				auto yOffset = shape::getIndexOffset(i, yShapeInfo, len);
				
				z[xOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
	    	}
		} 
		else {
			
			for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
	    		
	    		auto xOffset = shape::getIndexOffset(i, xShapeInfo, len);
				auto yOffset = shape::getIndexOffset(i, yShapeInfo, len);
				auto zOffset = shape::getIndexOffset(i, zShapeInfo, len);

				z[zOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
    		}
    	}
    }
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
void PairWiseBoolTransform<X,Y>::executeCudaShaped(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *vextraParams) {
    auto xType = nd4j::DataTypeUtils::fromT<X>();
    auto yType = nd4j::DataTypeUtils::fromT<Y>();    

	DISPATCH_BY_OPNUM_TT(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams, nullptr), PAIRWISE_BOOL_OPS);
}
      
    BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT PairWiseBoolTransform, , LIBND4J_TYPES, BOOL_TYPES);
}
}

#endif // PAIRWISE_BOOL_CU