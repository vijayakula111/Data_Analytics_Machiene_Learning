#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by raver on 4/9/2018.
//

#include <Environment.h>
#include "../indexreduce.h"
#include <op_boilerplate.h>
#include <helpers/DebugHelper.h>
#include <types/types.h>

#include "../legacy_ops.h"

using namespace simdOps;


template <typename T>
static __global__ void simpleIndexReduceGeneric(const int op,
                                           void *dx,
                                           Nd4jLong *xShapeInfo, int xRank,
                                           void *extraParams,
                                           Nd4jLong *result,
                                           Nd4jLong *resultShapeInfo, int zRank,
                                           int *dimension,
                                           int dimensionLength,
                                           int postProcessOrNot, int *allocationBuffer, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {

     functions::indexreduce::IndexReduce<T>::transform(op,dx,xShapeInfo,extraParams,result,resultShapeInfo,dimension,dimensionLength,postProcessOrNot,allocationBuffer,reductionBuffer,tadOnlyShapeInfo,tadOffsets);
}

namespace functions {
    namespace indexreduce {

        template <typename T>
        _CUDA_H void IndexReduce<T>::executeIndexReduceScalar(dim3 launchDims, hipStream_t *stream, const int opNum, void *dx, Nd4jLong *xShapeInfo, int xRank, void *extraParams, Nd4jLong *result, Nd4jLong *resultShapeInfo, int zRank, int *dimension, int dimensionLength, int postProcessOrNot, int *allocationBuffer, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {

            simpleIndexReduceGeneric<T><<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			 opNum,
			 dx,
			 xShapeInfo, xRank,
			 extraParams,
			 result,
			 nullptr, 0,
			 nullptr,
			 1,
			 1, allocationBuffer, reductionBuffer, tadOnlyShapeInfo, tadOffsets);

            checkCudaErrors(hipStreamSynchronize(*stream));
            nd4j::DebugHelper::checkErrorCode(stream, "execIndexReduceScalarFloat(...) failed");
        }

        template <typename T>
        _CUDA_H void IndexReduce<T>::executeIndexReduce(dim3 launchDims, hipStream_t *stream, const int opNum, void *dx, Nd4jLong *xShapeInfo, int xRank, void *extraParams, Nd4jLong *result, Nd4jLong *resultShapeInfo, int zRank, int *dimension, int dimensionLength, int postProcessOrNot, int *allocationBuffer, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {

            simpleIndexReduceGeneric<T><<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			 opNum,
			 dx,
			 xShapeInfo, xRank,
			 extraParams,
			 result,
			 resultShapeInfo, zRank,
			 dimension,
			 dimensionLength,
			 1, allocationBuffer, reductionBuffer, tadOnlyShapeInfo, tadOffsets);

            DEBUG_KERNEL(stream, opNum);
        }

        // This is the un-specialized struct.  Note that we prevent instantiation of this
        // struct by putting an undefined symbol in the function body so it won't compile.
        template<typename T>
        struct SharedIndexValue {
            // Ensure that we won't compile any un-specialized types
            __device__ T * getPointer() {
                extern __device__ void error(void);
                error();
                return 0;
            }
        };

// Following are the specializations for the following types.
// int, uint, char, uchar, short, ushort, long long, ulong long, bool, float, and double
// One could also specialize it for user-defined types.

        template<>
        struct SharedIndexValue<float> {
            __device__ IndexValue<float> * getPointer() {
                extern __shared__ IndexValue<float> s_int2[];
                return s_int2;
            }
        };
// Following are the specializations for the following types.
// int, uint, char, uchar, short, ushort, long long, ulong long, bool, float, and double
// One could also specialize it for user-defined types.

        template<>
        struct SharedIndexValue<double> {
            __device__ IndexValue<double> * getPointer() {
                extern __shared__ IndexValue<double> s_int6[];
                return s_int6;
            }
        };

        template <typename T>
        template <typename OpType>
        __device__ void IndexReduce<T>::aggregatePartials(IndexValue<T> **sPartialsRef, Nd4jLong tid, Nd4jLong numElements, void *vextraParams) {
            // start the shared memory loop on the next power of 2 less
            // than the block size.  If block size is not a power of 2,
            // accumulate the intermediate sums in the remainder range.
            auto extraParams = static_cast<T*>(vextraParams);
            IndexValue<T> *sPartials = *sPartialsRef;
            Nd4jLong floorPow2 = blockDim.x;

            if (floorPow2 & (floorPow2 - 1)) {
                while ( floorPow2 & (floorPow2 - 1) ) {
                    floorPow2 &= floorPow2 - 1;
                }

                if (tid >= floorPow2) {
                    IndexValue<T> prev = sPartials[tid - floorPow2];
                    IndexValue<T> curr = sPartials[tid];
                    sPartials[tid - floorPow2] = OpType::update(prev,curr,extraParams);
                }
                __syncthreads();
            }

            for (int activeThreads = floorPow2 >> 1;activeThreads; activeThreads >>= 1) {
                if (tid < activeThreads && tid + activeThreads < numElements) {
                    IndexValue<T> curr = sPartials[tid];
                    IndexValue<T> next = sPartials[tid + activeThreads];
                    sPartials[tid] = OpType::update(curr,next,extraParams);
                }
                __syncthreads();
            }
        }

        template <typename X>
        __device__ void IndexReduce<X>::transform(
                const int opNum,
                void *x,
                Nd4jLong *xShapeInfo,
                void *extraParams,
                Nd4jLong *result,
                Nd4jLong *resultShapeInfo,
                int *dimension,
                int dimensionLength,
                int postProcessOrNot,
                int *allocationBuffer,
                void *reductionBuffer,
                Nd4jLong *tadShapeInfo,
                Nd4jLong *tadOffset) {
             DISPATCH_BY_OPNUM_T(transform, PARAMS(x, xShapeInfo, extraParams, result, resultShapeInfo, dimension, dimensionLength, postProcessOrNot, allocationBuffer, reductionBuffer, tadShapeInfo, tadOffset), INDEX_REDUCE_OPS);
        }


        template <typename T>
        template <typename OpType>
        __device__ void IndexReduce<T>::transform(
                void *vdx,
                Nd4jLong *xShapeInfo,
                void *vextraParams,
                Nd4jLong *result,
                Nd4jLong *resultShapeInfo,
                int *dimension,
                int dimensionLength,
                int postProcessOrNot,
                int *allocationBuffer,
                void *vreductionBuffer,
                Nd4jLong *tadOnlyShapeInfo,
                Nd4jLong *tadOffsets){
            /**int
             * Gpu information for the problem
             */
            auto dx = static_cast<T*>(vdx);
            auto extraParams = static_cast<T*>(vextraParams);
            auto reductionBuffer = static_cast<T*>(vreductionBuffer);

            int tid = blockIdx.x * blockDim.x + threadIdx.x;
            __shared__ volatile int resultScalar;

            //shared memory space for storing intermediate results
            __shared__ IndexValue<T>* sPartials;
            if(threadIdx.x == 0) {
                extern __shared__ unsigned char shmem[];
                sPartials = reinterpret_cast<IndexValue<T>*>(shmem);
            }
            __syncthreads();

            sPartials[threadIdx.x] = OpType::startingIndexValue(dx);

            //length for the tad
            __shared__ volatile Nd4jLong xLength;

            __shared__ volatile Nd4jLong resultLength;



            //only compute the tad indexes once
            IndexValue <T> reduction = OpType::startingIndexValue(dx);

            if (threadIdx.x == 0) {
                if (resultShapeInfo != nullptr)
                    resultLength = shape::length(resultShapeInfo);
                else resultLength = 1;

                if (dimensionLength == 1) {
                    if (dimension == nullptr || dimension[0] == MAX_DIMENSION)
                        resultScalar = 1;
                    else
                        resultScalar = 0;
                }
                else
                    resultScalar = 0;

                if (resultLength == 1)
                    resultScalar = 1;

                //	xElementWiseStride = shape::elementWiseStride(xShapeInfo);

                xLength = shape::length(xShapeInfo);
            }
            __syncthreads();

            if (!resultScalar) {

                __shared__ Nd4jLong tadLength;
                __shared__ int tadEWS;
                __shared__ int numTads;

                if (threadIdx.x == 0) {
                    tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
                    tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                    numTads = shape::length(xShapeInfo) / tadLength;
                }
                __syncthreads();

                if (dimensionLength > 1 || tadEWS < 1) {

                    for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
                        
                        auto tadOffsetForBlock = tadOffsets[r];
                        sPartials[threadIdx.x] = OpType::startingIndexValue(dx);

                        for(int i = threadIdx.x;i < tadLength; i += blockDim.x) {                            
                            auto xOffset = tadOffsetForBlock + shape::getIndexOffset(i, tadOnlyShapeInfo, tadLength);
                            IndexValue<T> comp {dx[xOffset], i};
                            sPartials[threadIdx.x] = OpType::update(sPartials[threadIdx.x], comp, extraParams);
                        }

                        __syncthreads();
                        aggregatePartials<OpType>(&sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, tadLength),extraParams);

                        __syncthreads();
                        if (threadIdx.x == 0) {
                            result[r] = sPartials[threadIdx.x].index;
                        }
                    }
                } else {

                    for(int i = blockIdx.x; i < numTads; i+= gridDim.x) {
                        Nd4jLong tadOffsetForBlock = tadOffsets[i];

                        sPartials[threadIdx.x] = OpType::startingIndexValue(dx);

                        for (int x = threadIdx.x; x < tadLength; x+= blockDim.x) {
                            IndexValue<T> comp {dx[tadOffsetForBlock + x * tadEWS], x};
                            sPartials[threadIdx.x] =  OpType::update(sPartials[threadIdx.x], comp, extraParams);
                        }

                        __syncthreads();
                        aggregatePartials<OpType>(&sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, tadLength),extraParams);

                        __syncthreads();
                        if (threadIdx.x == 0) {
                            result[i] = sPartials[threadIdx.x].index; //postProcess(sPartials[0],tadLength ,extraParams);
                        }
                    }
                }
            } else {
                auto n = shape::length(xShapeInfo);
                auto xElementWiseStride = shape::elementWiseStride(xShapeInfo);

                if(xElementWiseStride >= 1) {
                    for(Nd4jLong i = tid;i < n; i += (blockDim.x * gridDim.x)) {
                        IndexValue <T> indexVal = {dx[i * xElementWiseStride], i};
                        reduction = OpType::update(reduction, indexVal, extraParams);
                    }
                } else {
                                        
                    for(Nd4jLong i = tid;i < n; i += blockDim.x * gridDim.x) {                                                
                        auto offset = shape::getIndexOffset(i, xShapeInfo, n);
                        IndexValue <T> indexVal = {dx[offset], i};
                        reduction = OpType::update(reduction, indexVal, extraParams);
                    }
                }


                sPartials[threadIdx.x] = reduction;
                __syncthreads();

                aggregatePartials<OpType>(&sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, (int) n),extraParams);
                __syncthreads();

                if (gridDim.x > 1) {
                    __shared__ bool amLast;
                    unsigned int *tc = (unsigned int *) reductionBuffer;
                    tid = threadIdx.x;
                    if (threadIdx.x == 0) {
                        auto pBuffer = reinterpret_cast<IndexValue<T> *>(reductionBuffer);
                        pBuffer[blockIdx.x] = {sPartials[0].value, sPartials[0].index};
                    }
                    __threadfence();
                    __syncthreads();

                    if (tid==0) {
                        unsigned int ticket = atomicInc(&tc[16384], gridDim.x);
                        amLast = (ticket == gridDim.x-1);
                    }

                    __syncthreads();

                    if (amLast) {
                        tc[16384] = 0;
                        IndexValue<T> *pBuffer = (IndexValue<T> *) reductionBuffer;

                        sPartials[threadIdx.x] = OpType::startingIndexValue(dx);

                        for (Nd4jLong i = threadIdx.x; i < gridDim.x; i += blockDim.x) {
                            sPartials[threadIdx.x] = OpType::update(sPartials[threadIdx.x], pBuffer[i], extraParams);
                        }

                        __syncthreads();
                        aggregatePartials<OpType>(&sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(gridDim.x, blockDim.x),extraParams);

                        __syncthreads();
                        if (tid == 0) {
                            result[0] = sPartials[0].index;
                        }
                    }
                } else {
                    if (tid == 0) {
                        auto tc = reinterpret_cast<unsigned int *>(reductionBuffer);
                        tc[16384] = 0;
                        result[0] = sPartials[0].index;
                    }
                }
            }
        }

        BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT IndexReduce, , LIBND4J_TYPES);
    }
}



