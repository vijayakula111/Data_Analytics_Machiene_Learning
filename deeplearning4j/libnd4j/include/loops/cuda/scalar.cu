#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#ifndef SCALAR_CU
#define SCALAR_CU

#include "loops/scalar.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <op_boilerplate.h>
#include <helpers/TAD.h>
#include <types/types.h>

using namespace simdOps;

////////////////////////////////////////////////////////////////////////////////
template <typename X, typename Y, typename Z, typename OpType>
__global__ static void scalarSimpleShaped(void* vx, void *vscalar, Nd4jLong *xShapeInfo, void *vparams, void *vz, Nd4jLong *zShapeInfo, int *allocationBuffer) {
    
    auto scalar = reinterpret_cast<Y*>(vscalar)[0];
    auto x      = reinterpret_cast<X*>(vx);
    auto params = reinterpret_cast<Z*>(vparams);
    auto z = reinterpret_cast<Z*>(vz);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ Nd4jLong length;
    if(threadIdx.x == 0) {
        length = shape::length(xShapeInfo);
    }
    __syncthreads();

    auto xEws = shape::elementWiseStride(xShapeInfo);
    auto zEws = shape::elementWiseStride(zShapeInfo);

    auto xOrder = shape::order(xShapeInfo);
    auto zOrder = shape::order(zShapeInfo);


    if (xEws >= 1 && zEws >= 1 && xOrder == zOrder) {
        for (Nd4jLong i = tid; i < length; i += totalThreads) {
            z[i * zEws] = OpType::op(x[i * xEws], scalar, params);
        }
    } else {
        for (Nd4jLong i = tid; i < length; i += totalThreads) {
            z[shape::getIndexOffset(i, zShapeInfo, length)] = OpType::op(x[shape::getIndexOffset(i, xShapeInfo, length)], scalar, params);
        }
    }
    
}

////////////////////////////////////////////////////////////////////////////////
template <typename X, typename Y, typename Z, typename OpType>
__global__ static void scalarAlongDimension(void *vx, Nd4jLong *xShapeInfo,
                                          void *vextraParams,
                                          void *vz, Nd4jLong *zShapeInfo,
                                          void *vscalars,
                                          int *dimension, int dimensionLength,
                                          Nd4jLong *tadShapeInfo,  Nd4jLong *tadOffsets,
                                          Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
        
    auto x = reinterpret_cast<X*>(vx);
    auto extraParams = reinterpret_cast<Z*>(vextraParams);
    auto z = reinterpret_cast<Z*>(vz);
    auto scalars = reinterpret_cast<Y*>(vscalars);

    if (tadShapeInfoZ == nullptr) {
        tadShapeInfoZ = tadShapeInfo;
        tadOffsetsZ = tadOffsets;
    }

    // tad preparation
    auto tadEws = shape::elementWiseStride(tadShapeInfo);
    auto zEws = shape::elementWiseStride(tadShapeInfo);
    auto tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
    auto numTads =shape::length(xShapeInfo) / tadLength;

    if(tadEws < 1 || zEws < 1) {
        printf("ScalarTransform<X,Y,Z>::transformCuda: super-bad loop visited. Shouldn't ever happen\n");
        return;
    }

    // main loop, rolling over tads
    for (int r = blockIdx.x; r < numTads; r+=gridDim.x) {
        
        Z *oZ = z + tadOffsetsZ[r];
        X *oX = x + tadOffsets[r];

        for (int f = threadIdx.x; f < tadLength; f+= blockDim.x)
            oZ[f] = OpType::op(oX[f], scalars[r], extraParams);         
    }
}


namespace functions {
namespace scalar    {

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y, typename Z>
template<typename OpType>
void _CUDA_H ScalarTransform<X,Y,Z>::intermediateShaped(dim3& launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong *hzShapeInfo, void* vscalar, void *vextraParams, int *allocPointer){
    
    auto xEws = shape::elementWiseStride(hxShapeInfo);
    auto xOrder = shape::order(hxShapeInfo);

    auto zEws = shape::elementWiseStride(hzShapeInfo);
    auto zOrder = shape::order(hzShapeInfo);

    auto length = shape::length(hxShapeInfo);

    scalarSimpleShaped<X, Y, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vscalar, xShapeInfo, vextraParams, vz, zShapeInfo, allocPointer);
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y, typename Z>
template<typename OpType>
void _CUDA_H ScalarTransform<X,Y,Z>::intermediateAlongDimension(dim3& launchDims, hipStream_t *stream, void *x, Nd4jLong *xShapeInfo, void *z, Nd4jLong *zShapeInfo, void *scalars, void *extraParams, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
    scalarAlongDimension<X, Y, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z>>>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y, typename Z>
void ScalarTransform<X,Y,Z>::executeCudaShaped(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong *hzShapeInfo, void* vscalar, void *vextraParams) {

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
	   printf("H14 opNum:[%i]\n", opNum);

    DISPATCH_BY_OPNUM_TTT(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, hxShapeInfo, vz, zShapeInfo, hzShapeInfo, vscalar, vextraParams, nullptr), SCALAR_OPS);
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y, typename Z>
void ScalarTransform<X,Y,Z>::executeCudaAlongDimension(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *vscalars, void *vextraParams, int *dimension, int dimensionLength,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
    DISPATCH_BY_OPNUM_TTT(intermediateAlongDimension, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalars, vextraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), SCALAR_OPS);
}



BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_0);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_1);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_2);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_3);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_4);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_5);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_6);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_7);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_8);
BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT ScalarTransform, , PAIRWISE_TYPES_9);

}
}



#endif // SCALAR_CU