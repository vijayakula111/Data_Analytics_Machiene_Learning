#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void concatKernelHStack(int numArrays,
                                       Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                       void *vz, Nd4jLong *zShapeInfo) {

        // we expect all data coming in as vectors, and z as 2D matrix
        // the only significant difference here is the fact that input lengths might be different
        auto z = reinterpret_cast<T *>(vz);
        auto inputShapes = (Nd4jLong **) inputShapeInfos;
        T **input = (T **) data;

        __shared__ int inputEWS;
        __shared__ int resultEWS;
        __shared__ int inputLength;

        if (threadIdx.x == 0) {
            resultEWS = shape::elementWiseStride(zShapeInfo);
        }
        __syncthreads();

        for (int r = blockIdx.x; r < numArrays; r += gridDim.x) {

            __shared__ int baseIdx;
            if (threadIdx.x == 0) {
                baseIdx = 0;
                for (int f = 0; f < r; f++) {
                    baseIdx += shape::length(inputShapes[f]);
                }
            }
            __syncthreads();


            T *inputData = (T *) input[r];

            if (threadIdx.x == 0) {
                inputEWS = shape::elementWiseStride(inputShapes[r]);
                inputLength = shape::length(inputShapes[r]);
            }
            __syncthreads();

            for (int i = threadIdx.x; i < inputLength; i += blockDim.x) {
                z[baseIdx + i * resultEWS] = inputData[i * inputEWS];
            }
            __syncthreads();
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execConcatKernelHStack(int numArrays,
                                           Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                           void *vz, Nd4jLong *zShapeInfo) {

        concatKernelHStack<T>(numArrays, data, inputShapeInfos, vz, zShapeInfo);
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void concatKernelHStackGeneric(dim3 &launchDims, hipStream_t *stream,
                                            int numArrays,
                                            Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                            void *vz, Nd4jLong *zShapeInfo) {

        execConcatKernelHStack<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                                 (numArrays, data, inputShapeInfos, vz, zShapeInfo);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT concatKernelHStackGeneric, (dim3 & launchDims, hipStream_t * stream, int numArrays, Nd4jPointer * data, Nd4jPointer * inputShapeInfos, void * vz, Nd4jLong * zShapeInfo), LIBND4J_TYPES);
}