#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void shuffleKernel(void **vdX, Nd4jLong **xShapeInfo,
                                  void **vdZ,
                                  int N,
                                  int *shuffleMap,
                                  Nd4jLong **tadOnlyShapeInfo, Nd4jLong **tadOffsets) {

        // we assume that shuffle map for each X contains pair TAD Y

        auto dX = reinterpret_cast<T **>(vdX);
        auto dZ = reinterpret_cast<T **>(vdZ);

        __shared__ int tadLength;
        __shared__ int tadEWS;
        __shared__ int numTads;

        for (int f = 0; f < N; f++) {

            T *x = (T *) dX[f];
            T *z = (T *) dZ[f];

            __syncthreads();

            if (threadIdx.x == 0) {
                tadLength = shape::length(tadOnlyShapeInfo[f]);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo[f]);
                numTads = shape::length(xShapeInfo[f]) / tadLength;
            }
            __syncthreads();

            // we roll over the pairs of TADs, thus limit is numTads / 2
            for (Nd4jLong r = blockIdx.x; r < numTads; r += blockDim.x) {

                if (shuffleMap[r] < 0)
                    continue;

                Nd4jLong oldOffset = tadOffsets[f][r];
                Nd4jLong newOffset = tadOffsets[f][shuffleMap[r]];

                T *rX = x + oldOffset;
                T *rY = x + newOffset;

                T *zX = z + oldOffset;
                T *zY = z + newOffset;

                // so we're going to change TAD[oldOffset] with TAD[newOffset]
                if (tadEWS == 1) {

                    for (Nd4jLong i = threadIdx.x; i < tadLength; i += blockDim.x) {

                        T oldX = rX[i];
                        rX[i] = rY[i];
                        zY[i] = oldX;
                    }

                } else {

                    for (Nd4jLong i = threadIdx.x; i < tadLength; i += blockDim.x) {

                        auto xOffset = shape::getIndexOffset(i, tadOnlyShapeInfo[f], tadLength);
                        auto yOffset = newOffset + xOffset;
                        xOffset += oldOffset;

                        T oldX = x[xOffset];
                        z[xOffset] = x[yOffset];
                        z[yOffset] = oldX;
                    }
                }
            }
        }
    }

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execShuffleKernel(void **vdX, Nd4jLong **xShapeInfo,
                                      void **vdZ,
                                      int N,
                                      int *shuffleMap,
                                      Nd4jLong **tadOnlyShapeInfo, Nd4jLong **tadOffsets) {

        shuffleKernel<T>(vdX, xShapeInfo, vdZ, N, shuffleMap, tadOnlyShapeInfo, tadOffsets);
    }

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void shuffleKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                       void **vdX, Nd4jLong **xShapeInfo,
                                       void **vdZ,
                                       int N,
                                       int *shuffleMap,
                                       Nd4jLong **tadOnlyShapeInfo, Nd4jLong **tadOffsets) {

        execShuffleKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vdX, xShapeInfo, vdZ, N, shuffleMap, tadOnlyShapeInfo, tadOffsets);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT shuffleKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * *vdX, Nd4jLong * *xShapeInfo, void **vdZ, int N, int * shuffleMap, Nd4jLong * *tadOnlyShapeInfo, Nd4jLong * *tadOffsets), LIBND4J_TYPES);
}