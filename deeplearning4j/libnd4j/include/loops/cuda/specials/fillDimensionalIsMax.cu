#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {


////////////////////////////////////////////////////////////////////////
    template <typename T>
    __device__ void fillDimensionalIsMax(void *vdX,
                                         void *vdZ, Nd4jLong *zShapeInfo,
                                         Nd4jLong *tadOnlyShapeInfo,
                                         int *dimension, int dimensionLength,
                                         Nd4jLong *tadOffsets) {

        auto dX = reinterpret_cast<Nd4jLong *>(vdX);
        auto dZ = reinterpret_cast<T *>(vdZ);

        __shared__ int tadLength;
        __shared__ int tadEWS;
        __shared__ int numTads;

        if (threadIdx.x == 0) {
            tadLength = shape::tadLength(zShapeInfo, dimension, dimensionLength);
            tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
            numTads = shape::length(zShapeInfo) / tadLength;
        }
        __syncthreads();

        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            auto tadOffsetForBlock = tadOffsets[r];

            int highestElement = (int) dX[r];

            if (dimensionLength > 1 || tadEWS < 1) {

                for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {

                    auto xOffset = tadOffsetForBlock + shape::getIndexOffset(e, tadOnlyShapeInfo, tadLength);
                    dZ[xOffset] = (e == highestElement ? (T) 1 : (T) 0);
                }
            } else {
                for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                    // so, we just set dZ[e] for each TAD. Sure, e should be replaced with
                    auto idx = tadOffsetForBlock + (e * tadEWS);
                    dZ[idx] = (e == highestElement ? (T) 1 : (T) 0);
                }
            }
        }
    }


////////////////////////////////////////////////////////////////////////
    template <typename T>
    __global__ void execfillDimensionalIsMax(void *dX,
                                             void *dZ, Nd4jLong *zShapeInfo,
                                             Nd4jLong *tadOnlyShapeInfo,
                                             int *dimension, int dimensionLength,
                                             Nd4jLong *tadOffsets) {

        fillDimensionalIsMax<T>(dX, dZ, zShapeInfo, tadOnlyShapeInfo, dimension, dimensionLength, tadOffsets);
    }

////////////////////////////////////////////////////////////////////////
    template <typename T>
    __host__ void fillDimensionalIsMaxGeneric(dim3 &launchDims, hipStream_t *stream,
                                              void *dX,
                                              void *dZ, Nd4jLong *zShapeInfo,
                                              Nd4jLong *tadOnlyShapeInfo,
                                              int *dimension, int dimensionLength,
                                              Nd4jLong *tadOffsets) {

        execfillDimensionalIsMax<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dX, dZ, zShapeInfo, tadOnlyShapeInfo, dimension, dimensionLength, tadOffsets);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT fillDimensionalIsMaxGeneric, (dim3& launchDims, hipStream_t *stream, void *dX, void *dZ, Nd4jLong *zShapeInfo, Nd4jLong *tadOnlyShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOffsets), LIBND4J_TYPES);
}