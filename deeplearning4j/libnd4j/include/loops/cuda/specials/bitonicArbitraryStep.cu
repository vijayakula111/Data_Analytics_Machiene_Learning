#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 28.11.2018
//

#include <ops/specials_cuda.h>


//////////////////////////////////////////////////////////////////////////
template<typename T>
__device__
void bitonicArbitraryStepKernel(void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending) {

    auto x = static_cast<T*>(vx);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int half = window>>1;

    __shared__ T *shmem;
    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shrd[];
        shmem = (T *) shrd;
    }
    __syncthreads();

    //for (int i = 0; i < length; i+= window)
    /*
        if window == 4;
        iterations will be: 0; 4; 8; 12; 16; 20
        if gridDim = 3;
        on first iteration we'll have: 0; 4; 8;
        on second iteration we'll have: 0 + (3 * 4) = 12;  4 + (3 * 4) = 16; 8 + (3 * 4) = 20
    */
    int firstPosition;
    int firstStep;
    int secondPosition;
    int secondStep;

    int WARP_SIZE = 32;
    int numWarps = (gridDim.x * blockDim.x) / 32;
    int warpId = tid / WARP_SIZE;
    int warpIdx = tid % WARP_SIZE;

    if (half >= 128) {
        firstPosition = blockIdx.x * window;
        firstStep = gridDim.x * window;

        secondPosition = threadIdx.x;
        secondStep = blockDim.x;
    } else if (half >= 32) {
        firstPosition = warpId * window;
        firstStep = numWarps * window;

        secondPosition = warpIdx;
        secondStep = WARP_SIZE;
    } else {
        firstPosition = tid * window;
        firstStep = blockDim.x * gridDim.x * window;

        secondPosition = 0;
        secondStep = 1;
    }


    for (int i = firstPosition; i < length; i += firstStep) {
        for (int j = secondPosition; j < half; j += secondStep) {
            int it = (reverse) ? i + j + half : i + window - j - 1;
            int ij = i+j;
            if (it < length && ij < length ) {
                int posIT = getDevicePosition(xShapeInfo,it);
                int posIJ = getDevicePosition(xShapeInfo, ij);

                shmem[threadIdx.x] = x[posIJ];
                shmem[threadIdx.x + blockDim.x] = x[posIT];

                if(!descending == (shmem[threadIdx.x] > shmem[threadIdx.x + blockDim.x])) {
                    x[posIJ] = shmem[threadIdx.x + blockDim.x];
                    x[posIT] = shmem[threadIdx.x];
                }
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void execBitonicArbitraryStepKernel(void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending) {

    bitonicArbitraryStepKernel<T>(vx, xShapeInfo, window, length, reverse, descending);
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__host__ void bitonicArbitraryStepGeneric(dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending) {

    execBitonicArbitraryStepKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, window, length, reverse, descending);
}
BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT bitonicArbitraryStepGeneric, (dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending), LIBND4J_TYPES);
