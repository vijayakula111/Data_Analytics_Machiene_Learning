#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

///////////////////////////////////////////////////////////////////////
/**
 * This kernel accumulates X arrays, and stores z into Z
 *
 * @tparam T
 * @param x
 * @param z
 * @param n
 * @param length
 */
    template<typename T>
    __device__ void accumulateKernel(void **vx, void *vz, int n, const Nd4jLong length) {

        auto x = reinterpret_cast<T **>(vx);
        auto z = reinterpret_cast<T *>(vz);

        __shared__
        T *shmem;

        if (threadIdx.x == 0) {
            extern __shared__ unsigned char sharedmem[];
            shmem = (T *) sharedmem;
        }
        __syncthreads();

        for (int r = blockDim.x * blockIdx.x; r < length; r += blockDim.x * gridDim.x) {
            shmem[threadIdx.x] = 0.0f;

            Nd4jLong baseIdx = r;

            // aggregation step, we roll over all arrays
            for (int ar = 0; ar < n; ar++) {
                T *cdata = (T *) x[ar];
                cdata += baseIdx;

                if (baseIdx + threadIdx.x < length)
                    shmem[threadIdx.x] += cdata[threadIdx.x];
            }

            T *wdata = z + baseIdx;

            // saving accumulated values
            if (baseIdx + threadIdx.x < length)
                wdata[threadIdx.x] = shmem[threadIdx.x];
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execAccumulateKernel(void **vx, void *vz, int n, const Nd4jLong length) {

        accumulateKernel<T>(vx, vz, n, length);
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void
    accumulateKernelGeneric(dim3 &launchDims, hipStream_t *stream, void **vx, void *vz, int n, const Nd4jLong length) {

        execAccumulateKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> > (vx, vz, n, length);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT accumulateKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * *vx, void * vz, int n, const Nd4jLong length), LIBND4J_TYPES);
}