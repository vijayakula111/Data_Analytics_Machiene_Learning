#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void convertHalfs(half *dx, Nd4jLong n, void *dz) {

        auto z = reinterpret_cast<T *>(dz);
        int tid = threadIdx.x + blockIdx.x * gridDim.x;

        for (Nd4jLong i = tid; i < n; i += blockDim.x * gridDim.x)
            z[i] = static_cast<T>(__half2float(dx[i]));
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execConvertHalfs(half *dx, Nd4jLong n, void *dz) {

        convertHalfs<T>(dx, n, dz);
    }


///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void convertHalfsToGeneric(dim3 &launchDims, hipStream_t *stream, half *dx, Nd4jLong n, void *dz) {

        execConvertHalfs<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> > (dx, n, dz);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT convertHalfsToGeneric, (dim3 & launchDims, hipStream_t * stream, half * dx, Nd4jLong n, void * dz), LIBND4J_TYPES);
}