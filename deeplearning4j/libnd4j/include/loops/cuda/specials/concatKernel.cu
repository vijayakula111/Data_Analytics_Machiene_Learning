#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {
///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void concatKernel(int numArrays,
                                 Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                 void *vz, Nd4jLong *resultShapeInfo,
                                 Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers,
                                 Nd4jLong *zTadShape, Nd4jLong *zOffsets) {

        int tid = threadIdx.x + blockIdx.x * blockDim.x;

        int zRank = shape::rank(resultShapeInfo);

        auto result = reinterpret_cast<T*>(vz);
        auto dataT = reinterpret_cast<T **>(data);
        auto shapeInfoPointers = reinterpret_cast<Nd4jLong **>(inputShapeInfos);
        auto tadShapes = reinterpret_cast<Nd4jLong **>(tadPointers);
        auto tadOffsets = reinterpret_cast<Nd4jLong **>(offsetPointers);

        //if (threadIdx.x == 0 && blockIdx.x == 0) {
        //    shape::printShapeInfoLinear("zTadShape", zTadShape);
        //}

        //__shared__ int tDim[1];
        __shared__ int baseIdx;

        __shared__ int yLength;
        __shared__ char yOrder;
        __shared__ int yEWS;

        char zOrder = shape::order(resultShapeInfo);

        int zEWS = shape::elementWiseStride(resultShapeInfo);
        int tadEWS = shape::elementWiseStride(zTadShape);
        int zLength = shape::length(resultShapeInfo);

        __shared__ int arrOffset;
        __shared__ int numTads;


        if (shape::isVector(resultShapeInfo)) {
            //if (threadIdx.x == 0 && blockIdx.x == 0)
            //	printf("Vector here\n");

            if (zEWS >= 1) {
                for (int r = blockIdx.x; r < numArrays; r += gridDim.x) {
                    if(shape::isVector(shapeInfoPointers[r]) || shape::order(shapeInfoPointers[r]) == shape::order(resultShapeInfo)) {
                        yLength = shape::length(shapeInfoPointers[r]);
                        yEWS = shape::elementWiseStride(shapeInfoPointers[r]);
                        // FIXME: this is bad
                        __shared__ int baseIdx;
                        if (threadIdx.x == 0) {
                            baseIdx = 0;
                            for (int f = 0; f < r; f++) {
                                baseIdx += shape::length(shapeInfoPointers[f]);
                            }
                        }
                        __syncthreads();
                        for (int i = threadIdx.x; i < yLength && baseIdx + i < zLength; i += blockDim.x) {
                            result[baseIdx + i * zEWS] = dataT[r][i * yEWS];
                        }
                        __syncthreads();
                    } else {
                        if (tid == 0)
                            printf("Non-matched order for vector\n");
                    }
                }
            } else {
                if (tid == 0)
                    printf("Vector Non-1 zEWS\n");
            }
            return;
        }


        bool _vec = shape::isVector(resultShapeInfo);


        // TODO: to be pulled into separate kernel. matrix concatenation
        for (int r = 0; r < numArrays; r ++) {

            auto currentShape = shapeInfoPointers[r];
            auto currentData = dataT[r];
            auto currentTad = tadShapes[r];
            auto currentOffsets = tadOffsets[r];


            if (threadIdx.x == 0) {
                yLength = shape::length(currentTad);
                yOrder = shape::order(currentTad);
                yEWS = shape::elementWiseStride(currentTad);
                numTads = shape::length(currentShape) / yLength;

                arrOffset = 0;
                for (int f = 0; f < r; f++) {
                    arrOffset +=  shape::length(tadShapes[f]);
                }

                //if (threadIdx.x == 0 && blockIdx.x == 0) {
                //    shape::printShapeInfoLinear("currentTad", currentTad);
                //}
            }
            __syncthreads();

            if (yLength == 1 && _vec) {
                //if (threadIdx.x == 0 && blockIdx.x == 0)
                //	printf("Branch 0\n");

                // edge case, each thread will handle it's own tad then
                for (int j = tid; j < numTads; j += blockDim.x * gridDim.x) {
                    Nd4jLong inputOffset = currentOffsets[j];
                    Nd4jLong resultOffset = zOffsets[j];

                    T *dataTAD = currentData + inputOffset;
                    T *resultTAD = result + resultOffset;

                    Nd4jLong sub[MAX_RANK];

                    if (shape::order(zTadShape) == 'f') {
                        shape::ind2sub(shape::rank(zTadShape),shape::shapeOf(zTadShape),arrOffset, sub);
                    } else {
                        shape::ind2subC(shape::rank(zTadShape),shape::shapeOf(zTadShape),arrOffset, sub);
                    }
                    Nd4jLong baseOffset = shape::getOffset(0,shape::shapeOf(zTadShape),shape::stride(zTadShape), sub, shape::rank(zTadShape));

                    resultTAD += baseOffset;

                    auto yRank = shape::rank(currentTad);
                    auto tadRank = shape::rank(zTadShape);

                    shape::ind2subC(yRank, shape::shapeOf(currentTad), 0,sub);

                    auto yOffset = shape::getOffset(0, shape::shapeOf(currentTad), shape::stride(currentTad), sub, yRank);
                    resultOffset = shape::getOffset(0, shape::shapeOf(zTadShape), shape::stride(zTadShape), sub, tadRank);

                    resultTAD[resultOffset] =  dataTAD[yOffset];
                }
            } else {
                //if (threadIdx.x == 0 && blockIdx.x == 0)
                //	printf("Branch 1\n");

                for (int j = blockIdx.x; j < numTads; j += gridDim.x) {
                    auto inputOffset = currentOffsets[j];
                    auto resultOffset = zOffsets[j];

                    auto dataTAD = currentData + inputOffset;
                    auto resultTAD = result + resultOffset;

                    Nd4jLong sub[MAX_RANK];

                    shape::ind2subC(shape::rank(zTadShape),shape::shapeOf(zTadShape),arrOffset, sub);
                    Nd4jLong baseOffset = shape::getOffset(0,shape::shapeOf(zTadShape),shape::stride(zTadShape), sub, shape::rank(zTadShape));

                    resultTAD += baseOffset;

                    if (zOrder == yOrder && yEWS > 0  && tadEWS > 0) {
                        //if (threadIdx.x == 0 && blockIdx.x == 0)
                        //    printf("Branch A\n");

                        for (int i = threadIdx.x; i < yLength; i += blockDim.x) {
                            resultTAD[i * tadEWS] = dataTAD[i * yEWS];
                        }
                    } else {
                        if(tadEWS > 0 && shape::order(resultShapeInfo) == shape::order(currentTad)) {
                            //if (threadIdx.x == 0 && blockIdx.x == 0)
                            //    printf("Branch B\n");

                            if (threadIdx.x == 0) {
                                baseIdx = 0;
                                for (int f = 0; f < r; f++) {
                                    baseIdx += shape::length(shapeInfoPointers[f]);
                                }
                                //printf("R: %i; baseIdx: %i;\n", baseIdx);
                            }
                            __syncthreads();

                            if (numTads == 1) {
                                for(int k = threadIdx.x; k < yLength; k+= blockDim.x) {
                                    resultTAD[baseIdx + k * tadEWS] = dataTAD[k];
                                }
                            } else {
                                Nd4jLong yIdx[MAX_RANK];
                                auto yRank = shape::rank(currentTad);

                                for (int i = threadIdx.x; i < yLength; i+= blockDim.x) {
                                    shape::ind2subC(yRank, shape::shapeOf(currentTad), i, yIdx);
                                    auto yOffset = shape::getOffset(0, shape::shapeOf(currentTad), shape::stride(currentTad), yIdx, yRank);

                                    resultTAD[baseIdx + i * tadEWS] =  dataTAD[yOffset];
                                }
                            }
                            __syncthreads();
                        } else {
                            //if (threadIdx.x == 0 && blockIdx.x  == 0)
                            //    printf("Branch C; yLength: %i;\n", yLength);

                            Nd4jLong zIdx[MAX_RANK];
                            Nd4jLong yIdx[MAX_RANK];
                            auto yRank = shape::rank(currentTad);
                            auto tadRank = shape::rank(zTadShape);

                            for (int i = threadIdx.x; i < yLength; i+= blockDim.x) {
                                shape::ind2subC(yRank, shape::shapeOf(currentTad), i,yIdx);
                                shape::ind2subC(tadRank, shape::shapeOf(zTadShape), i,zIdx);

                                auto yOffset = shape::getOffset(0, shape::shapeOf(currentTad), shape::stride(currentTad), yIdx, yRank);
                                auto resultOffset = shape::getOffset(0, shape::shapeOf(zTadShape), shape::stride(zTadShape), zIdx, tadRank);

                                resultTAD[resultOffset] =  dataTAD[yOffset];
                            }
                        }
                    }
                    __syncthreads();
                }
            }
            __syncthreads();
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execConcatKernel(int numArrays,
                                     Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                     void *vz, Nd4jLong *zShapeInfo,
                                     Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers,
                                     Nd4jLong *zTadShape,
                                     Nd4jLong *zOffsets) {

        concatKernel<T>(numArrays, data, inputShapeInfos, vz, zShapeInfo, tadPointers, offsetPointers, zTadShape,
                        zOffsets);
    }


///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void concatKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                      int numArrays,
                                      Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                      void *vz, Nd4jLong *zShapeInfo,
                                      Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers,
                                      Nd4jLong *zTadShape,
                                      Nd4jLong *zOffsets) {


        execConcatKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                           (numArrays, data, inputShapeInfos, vz, zShapeInfo, tadPointers, offsetPointers, zTadShape, zOffsets);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT concatKernelGeneric, (dim3 & launchDims, hipStream_t * stream, int numArrays, Nd4jPointer * data, Nd4jPointer * inputShapeInfos, void * vz, Nd4jLong *zShapeInfo, Nd4jPointer * tadPointers, Nd4jPointer * offsetPointers, Nd4jLong * zTadShape, Nd4jLong * zOffsets), LIBND4J_TYPES);
}