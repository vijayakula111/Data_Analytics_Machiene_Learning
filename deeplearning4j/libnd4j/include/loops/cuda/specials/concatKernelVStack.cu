#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void concatKernelVStack(int numArrays,
                                       Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                       void *vz, Nd4jLong *zShapeInfo) {

        /*
         this is special case for concat: we group bunch of vectors into 2D matrix
         also: we expect each inputShapeInfo to have EWS, be a vector, and have equal size
         */
        auto z = static_cast<T *>(vz);

        auto inputShapes = (Nd4jLong **) inputShapeInfos;
        T **input = (T **) data;

        __shared__ int inputEWS;
        __shared__ int resultEWS;
        __shared__ int inputLength;

        if (threadIdx.x == 0) {
            inputLength = shape::length(inputShapes[0]);
            inputEWS = shape::elementWiseStride(inputShapes[0]);
            resultEWS = shape::elementWiseStride(zShapeInfo);
        }
        __syncthreads();

        for (int r = blockIdx.x; r < numArrays; r += gridDim.x) {

            int zOffset = r * inputLength * resultEWS;
            T *inputData = (T *) input[r];

            for (int i = threadIdx.x; i < inputLength; i += blockDim.x) {
                z[zOffset + i * resultEWS] = inputData[i * inputEWS];
            }
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execConcatKernelVStack(int numArrays,
                                           Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                           void *vz, Nd4jLong *zShapeInfo) {

        concatKernelVStack<T>(numArrays, data, inputShapeInfos, vz, zShapeInfo);
    }


///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void concatKernelVStackGeneric(dim3 &launchDims, hipStream_t *stream,
                                            int numArrays,
                                            Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                            void *vz, Nd4jLong *zShapeInfo) {

        execConcatKernelVStack<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                                 (numArrays, data, inputShapeInfos, vz, zShapeInfo);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT concatKernelVStackGeneric, (dim3 & launchDims, hipStream_t * stream, int numArrays, Nd4jPointer * data, Nd4jPointer * inputShapeInfos, void * vz, Nd4jLong *zShapeInfo), LIBND4J_TYPES);
}