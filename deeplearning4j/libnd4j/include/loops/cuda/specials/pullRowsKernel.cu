#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void pullRowsKernel(void *vx,
                                   void *vz,
                                   Nd4jLong len,
                                   Nd4jLong *indexes,
                                   Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                   Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets) {

        auto x = reinterpret_cast<T *>(vx);
        auto z = reinterpret_cast<T *>(vz);
        auto xEWS = shape::elementWiseStride(tadShapeInfo);
        auto zEWS = shape::elementWiseStride(zTadShapeInfo);
        auto tadLength = shape::length(tadShapeInfo);

        if (xEWS >= 1 && zEWS >= 1) {
            for (int idx = blockIdx.x; idx < len; idx += gridDim.x) {
                T *rX = x + tadOffsets[indexes[idx]];
                T *rZ = z + zTadOffsets[idx];

                for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
                    rZ[i * zEWS] = rX[i * xEWS];
                }
            }
        } else {
            for (int idx = blockIdx.x; idx < len; idx += gridDim.x) {
                T *rX = x + tadOffsets[indexes[idx]];
                T *rZ = z + zTadOffsets[idx];

                for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
                    auto xOffset = shape::getIndexOffset(i, tadShapeInfo, tadLength);
                    auto zOffset = shape::getIndexOffset(i, zTadShapeInfo, tadLength);
                    rZ[zOffset] = rX[xOffset];
                }
            }
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execPullRowsKernel(void *vx,
                                       void *vz,
                                       Nd4jLong len,
                                       Nd4jLong *indexes,
                                       Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                       Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets) {

        pullRowsKernel<T>(vx, vz, len, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets);
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void pullRowsKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                        void *vx,
                                        void *vz,
                                        Nd4jLong len,
                                        Nd4jLong *indexes,
                                        Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                        Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets) {

        execPullRowsKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                             (vx, vz, len, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT pullRowsKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * vx, void * vz, Nd4jLong len, Nd4jLong * indexes, Nd4jLong * tadShapeInfo, Nd4jLong * tadOffsets, Nd4jLong *zTadShapeInfo, Nd4jLong * zTadOffsets), LIBND4J_TYPES);
}

