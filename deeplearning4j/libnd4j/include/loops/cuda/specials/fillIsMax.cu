#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
    template <typename T>
    __device__ void fillIsMax(void *vdZ, Nd4jLong length, long idx) {
        auto dz = reinterpret_cast<T*>(vdZ);
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x)
            dz[i] = (i == idx ? (T) 1 : (T) 0);
    }

////////////////////////////////////////////////////////////////////////
    template <typename T>
    __global__ void execFillIsMax(void *dx, Nd4jLong length, long idx) {
        fillIsMax<T>(dx, length, idx);
    }

////////////////////////////////////////////////////////////////////////
    template <typename T>
    __host__ void fillIsMaxGeneric(dim3 &launchDims, hipStream_t *stream, void *dx, Nd4jLong length, long idx) {
        execFillIsMax<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dx, length, idx);
    }


    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT fillIsMaxGeneric, (dim3& launchDims, hipStream_t *stream, void* dz, Nd4jLong length, long idx), LIBND4J_TYPES);
}