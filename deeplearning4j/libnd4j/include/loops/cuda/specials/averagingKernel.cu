#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void averagingKernel(void **vdx, void *vdz, int n, Nd4jLong length, bool propagate) {

        auto dx = reinterpret_cast<T **>(vdx);
        auto dz = reinterpret_cast<T *>(vdz);

        __shared__
        T *shmem;

        if (threadIdx.x == 0) {
            extern __shared__ unsigned char sharedmem[];
            shmem = (T *) sharedmem;
        }
        __syncthreads();


        // each block cycles over it's own part of arrays
        for (int r = blockDim.x * blockIdx.x; r < length; r += blockDim.x * gridDim.x) {
            shmem[threadIdx.x] = (T) 0.0f;

            Nd4jLong baseIdx = r;

            // aggregation step, we roll over all arrays
            for (int ar = 0; ar < n; ar++) {
                T *cdata = (T *) dx[ar];
                cdata += baseIdx;

                if (baseIdx + threadIdx.x < length)
                    shmem[threadIdx.x] += cdata[threadIdx.x];
            }


            // average data in shared memory
            if (baseIdx + threadIdx.x < length)
                shmem[threadIdx.x] /= n;

            // div step & write out step
            if (dz != nullptr) {
                T *wdata = dz + baseIdx;

                if (baseIdx + threadIdx.x < length) {
                    wdata[threadIdx.x] = shmem[threadIdx.x];
                }
            }

            // propagate averaged data to all arrays
            if (propagate)
                for (int ar = 0; ar < n; ar++) {
                    T *cdata = (T *) dx[ar];
                    cdata += baseIdx;

                    if (baseIdx + threadIdx.x < length)
                        cdata[threadIdx.x] = shmem[threadIdx.x];
                }
        }
    }


///////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execAveragingKernel(void **vdx, void *vdz, int n, Nd4jLong length, bool propagate) {

        averagingKernel<T>(vdx, vdz, n, length, propagate);
    }


///////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void
    averagingKernelGeneric(dim3 &launchDims, hipStream_t *stream, void **vdx, void *vdz, int n, Nd4jLong length,
                           bool propagate) {

        execAveragingKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                              (vdx, vdz, n, length, propagate);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT averagingKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * *vdx, void * vdz, int n, Nd4jLong length, bool propagate), LIBND4J_TYPES);
}