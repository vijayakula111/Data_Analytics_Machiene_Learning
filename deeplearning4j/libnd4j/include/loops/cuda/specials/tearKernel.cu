#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __device__ void
    tearKernel(void *vx, Nd4jLong *xShapeInfo, Nd4jPointer *targets, Nd4jLong *zShapeInfo, Nd4jLong *tadShapeInfo,
               Nd4jLong *tadOffsets) {

        auto x = static_cast<T *>(vx);

        __shared__
        Nd4jLong tadLength;
        __shared__ int tadEWS;
        __shared__ int zEWS;
        __shared__ int tadRank;
        __shared__
        Nd4jLong numTads;
        __shared__ int zRank;
        __shared__
        Nd4jLong *tadShape;
        __shared__
        Nd4jLong *tadStride;
        __shared__
        Nd4jLong *zShape;
        __shared__
        Nd4jLong *zStride;

        if (threadIdx.x == 0) {
            tadLength = shape::length(tadShapeInfo);
            tadEWS = shape::elementWiseStride(tadShapeInfo);
            zEWS = shape::elementWiseStride(zShapeInfo);
            numTads = shape::length(xShapeInfo) / tadLength;
        }
        __syncthreads();

        for (Nd4jLong r = blockIdx.x; r < numTads; r += gridDim.x) {
            T *z = (T *) targets[r];
            T *s = x + tadOffsets[r];

            if (zEWS > 0 && tadEWS > 0) {
                for (Nd4jLong i = threadIdx.x; i < tadLength; i += blockDim.x)
                    z[i * zEWS] = s[i * tadEWS];
            } else {

                for (Nd4jLong j = 0; j < tadLength; j++) {
                    auto xOffset = shape::getIndexOffset(j, tadShapeInfo, tadLength);
                    auto zOffset = shape::getIndexOffset(j, zShapeInfo, tadLength);

                    z[zOffset] = s[xOffset];
                }
            }
        }
    }


////////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void
    execTearKernel(void *vx, Nd4jLong *xShapeInfo, Nd4jPointer *targets, Nd4jLong *zShapeInfo, Nd4jLong *tadShapeInfo,
                   Nd4jLong *tadOffsets) {

        tearKernel<T>(vx, xShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets);
    }

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void tearKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                    void *vx, Nd4jLong *xShapeInfo,
                                    Nd4jPointer *targets, Nd4jLong *zShapeInfo,
                                    Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        execTearKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                         (vx, xShapeInfo, targets, zShapeInfo, tadShapeInfo, tadOffsets);
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT tearKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * vx, Nd4jLong * xShapeInfo, Nd4jPointer *targets, Nd4jLong * zShapeInfo, Nd4jLong * tadShapeInfo, Nd4jLong * tadOffsets), LIBND4J_TYPES);
}