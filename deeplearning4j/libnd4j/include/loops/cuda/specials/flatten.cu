#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 27.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ void flattenKernel(
                    Nd4jPointer *extraPointers,
                    int dOffset,
                    char order,
                    void *vz, Nd4jLong *zShapeInfo,
                    void *vy, Nd4jLong *yShapeInfo) {

    auto z = reinterpret_cast<T*>(vz);
    auto y = reinterpret_cast<T*>(vy);
    
    __shared__ Nd4jLong lenY, yOrder, zEWS, yEWS;

    if (threadIdx.x == 0) {                
        
        yEWS = shape::elementWiseStride(yShapeInfo);
        zEWS = shape::elementWiseStride(zShapeInfo);
        lenY = shape::length(yShapeInfo);
    }
    __syncthreads();

    Nd4jLong tid = blockIdx.x * blockDim.x + threadIdx.x;        
        
    if (zEWS >= 1 && yEWS >= 1 && yOrder == order) {
 
        for (int i = tid; i < lenY; i += gridDim.x * blockDim.x)
            z[i * zEWS + dOffset] = y[i * yEWS];
    } 
    else {
        
        for(auto i = tid; i < lenY; i += gridDim.x * blockDim.x)
            z[i * zEWS + dOffset] = y[shape::getIndexOrderOffset(i, yShapeInfo, lenY, order)];
    } 
}

////////////////////////////////////////////////////////////////////////
template <typename T>
__host__ void flattenKernelGeneric(dim3& launchDims, hipStream_t *stream, 
                            Nd4jPointer *extraPointers,
                            int dOffset,
                            char order,
                            void *vz, Nd4jLong *zShapeInfo,
                            void *vy, Nd4jLong *yShapeInfo) {

    flattenKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(extraPointers, dOffset, order, vz, zShapeInfo, vy, yShapeInfo);
}

BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT flattenKernelGeneric, (dim3& launchDims, hipStream_t *stream, Nd4jPointer *extraPointers, int dOffset, char order, void *vz, Nd4jLong *zShapeInfo, void *vy, Nd4jLong *yShapeInfo), LIBND4J_TYPES);


}