#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 28.11.2018
//

#include <ops/specials_cuda.h>

//////////////////////////////////////////////////////////////////////////
template<typename T>
__device__ void bitonicSortStepKernel(void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending) {

    auto x = static_cast<T*>(vx);

    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= length)
        return;

    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        int posI = getDevicePosition(xShapeInfo, i);
        int posIXJ = getDevicePosition(xShapeInfo, ixj);

        if ((i&k)==0) {
            /* Sort ascending */
            if (!descending == (x[posI]>x[posIXJ])) {
                /* exchange(i,ixj); */
                T temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;
            }
        } else if ((i&k)!=0) {
            /* Sort descending */
            if (!descending == (x[posI]<x[posIXJ])) {
                /* exchange(i,ixj); */
                T temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void execBitonicSortStepKernel(void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending) {

    bitonicSortStepKernel<T>(vx, xShapeInfo, j, k, length, descending);
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__host__ void bitonicSortStepGeneric(dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending) {

    execBitonicSortStepKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, j, k, length, descending);
}
BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT bitonicSortStepGeneric, (dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending), LIBND4J_TYPES);
