#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <Environment.h>
#include <loops/transform_bool.h>
#include <types/types.h>
#include <op_boilerplate.h>

#include <loops/legacy_ops.h>
#include <helpers/DebugHelper.h>

using namespace simdOps;


template <typename X, typename Z, typename OpType>
__global__ void transformBoolSimple(void *dy, Nd4jLong *xShapeInfo, int xRank,
								void *params,
								void *result, Nd4jLong *zShapeInfo, int zRank,
								int *allocationPointer,
								void *reductionPointer,
								Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	functions::transform::TransformBool<X,Z>::template transformCuda<OpType>(dy,xShapeInfo,params,result,zShapeInfo,allocationPointer,reductionPointer,tadShapeInfo, tadOffsets);
}


namespace functions {
    namespace transform {

        template<typename X, typename Y>
        _CUDA_H void TransformBool<X,Y>::executeTransformShaped(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			DISPATCH_BY_OPNUM_TT(intermediateShaped, PARAMS(launchDims, stream, x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), TRANSFORM_BOOL_OPS);

            DEBUG_KERNEL(stream, opNum);
        }


        template<typename X, typename Z>
        template <typename OpType>
        __device__ void TransformBool<X,Z>::transformCuda(
						void *vdy,
						Nd4jLong *shapeInfo,
						void *vparams,
						void *vresult,
						Nd4jLong *zShapeInfo,
						int *allocationPointer, void *vreductionPointer, 
						Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        	auto dy = static_cast<X*>(vdy);
		    auto result = static_cast<Z*>(vresult);
		    auto params = static_cast<X*>(vparams);
		    auto reductionPointer = static_cast<Z*>(vreductionPointer);

		    if(OpType::requiresSpecial) {
			    OpType::execSpecialCuda(dy,shapeInfo,result,zShapeInfo,params, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
			    return;
		    } else {

		        auto xOrder = shape::order(shapeInfo);
		        auto zOrder = shape::order(zShapeInfo);

		        auto xEws = shape::elementWiseStride(shapeInfo);
    		    auto zEws = shape::elementWiseStride(zShapeInfo);
	    	    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

                __shared__ Nd4jLong length;
		        if(threadIdx.x == 0)
			        length = shape::length(shapeInfo);
		        __syncthreads();

				int totalThreads = gridDim.x * blockDim.x;

		        if(xEws >= 1 && zEws >= 1 && xOrder == zOrder) {
					if(xEws == 1 && zEws == 1) {
						/* equal, positive, non-unit increments. */
						for (Nd4jLong i = tid; i < length; i += totalThreads) {
							result[i] = OpType::op(dy[i], params);
						}
					}
					else {
						for (Nd4jLong i = tid; i < length; i += totalThreads) {
							result[i * zEws] = OpType::op(dy[i * xEws], params);
						}
					}
		        }
		        else {
			
		    	    for (Nd4jLong i = tid; i < length; i+= totalThreads) {
						auto xOffset2 = shape::getIndexOffset(i, shapeInfo,  length);
						auto zOffset2 = shape::getIndexOffset(i, zShapeInfo, length);						
	    			    result[zOffset2] = OpType::op(dy[xOffset2], params);
		    	    }
		        }
	        }
	    };


		template<typename X, typename Z>
		template <typename OpType>
		_CUDA_H void TransformBool<X,Z>::intermediateShaped(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			transformBoolSimple<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
		}

        BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT TransformBool, , LIBND4J_TYPES, BOOL_TYPES);
    }
}
