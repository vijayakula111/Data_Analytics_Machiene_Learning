#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <loops/random.h>
#include <dll.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>
#include <helpers/DebugHelper.h>

using namespace randomOps;

template <typename T, typename OpClass>
static inline __device__ void randomSingleGeneric(
        Nd4jPointer state,
        void *z,
        Nd4jLong *zShapeBuffer,
        void *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            z,
            zShapeBuffer,
            extraArguments);
}

template <typename T, typename OpClass>
static inline __device__ void randomDoubleGeneric(
        Nd4jPointer state,
        void *x,
        Nd4jLong *xShapeBuffer,
        void *z,
        Nd4jLong *zShapeBuffer,
        void *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            x,
            xShapeBuffer,
            z,
            zShapeBuffer,
            extraArguments);
}


template <typename T, typename OpClass>
static inline __device__ void randomTripleGeneric(
        Nd4jPointer state,
        void *x,
        Nd4jLong *xShapeBuffer,
        void *y,
        Nd4jLong *yShapeBuffer,
        void *z,
        Nd4jLong *zShapeBuffer,
        void *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            x,
            xShapeBuffer,
            y,
            yShapeBuffer,
            z,
            zShapeBuffer,
            extraArguments);
}


#ifndef __CLION_IDE__
// here we generate kernels for target operations
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, float, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, double, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, float16, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, bfloat16, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, float, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, double, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, float16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, bfloat16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, float, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, double, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, float16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, bfloat16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

#endif

namespace functions {
    namespace random {
            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

                auto x = reinterpret_cast<T*>(vx);
                auto y = reinterpret_cast<T*>(vy);
                auto z = reinterpret_cast<T*>(vz);
                auto extraArguments = reinterpret_cast<T*>(vextraArguments);
                
                if (OpClass::requiresSpecial) {
                    OpClass::specialOpCuda(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments);
                    return;
                } else {

                __shared__ Nd4jLong length;
                __shared__ int xEWS;
                __shared__ int yEWS;
                __shared__ int zEWS;

                __shared__ nd4j::graph::RandomGenerator *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                nd4j::graph::RandomGenerator *devBuffer;
                if (threadIdx.x == 0) {
                    length = shape::length(zShapeBuffer);
                    xEWS = shape::elementWiseStride(xShapeBuffer);
                    yEWS = shape::elementWiseStride(yShapeBuffer);
                    zEWS = shape::elementWiseStride(zShapeBuffer);

                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::graph::RandomGenerator *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::graph::RandomGenerator *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::graph::RandomGenerator); e+= blockDim.x) {
                    cB[e] = dB[e];
                }
                __syncthreads();


                int tid = blockIdx.x * blockDim.x + threadIdx.x;

                if (xEWS >= 1 && yEWS >= 1 && zEWS >= 1) {
                    for (Nd4jLong e = tid; e < length; e += blockDim.x * gridDim.x) {
                        z[e * zEWS] = OpClass::op(x[e * xEWS], y[e * yEWS], e, length, buffer, extraArguments);
                    }
                } else {
                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {
                        
                        auto xOffset2 = shape::getIndexOffset(i, xShapeBuffer, length);
                        auto yOffset2 = shape::getIndexOffset(i, yShapeBuffer, length);
                        auto zOffset2 = shape::getIndexOffset(i, zShapeBuffer, length);                        

                        z[zOffset2] = OpClass::op(x[xOffset2], y[yOffset2], i, length, buffer, extraArguments);
                    }
                }

                __syncthreads();
                if (threadIdx.x == 0 && blockIdx.x == 0)
                    devBuffer->rewindH(length);
                }
            };


            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

                auto x = reinterpret_cast<T*>(vx);
                auto z = reinterpret_cast<T*>(vz);
                auto extraArguments = reinterpret_cast<T*>(vextraArguments);

                __shared__ Nd4jLong length;
                __shared__ int xEWS;
                __shared__ int zEWS;

                __shared__ nd4j::graph::RandomGenerator *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::graph::RandomGenerator *devBuffer;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::graph::RandomGenerator *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::graph::RandomGenerator *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);

                    length = shape::length(zShapeBuffer);
                    xEWS = shape::elementWiseStride(xShapeBuffer);
                    zEWS = shape::elementWiseStride(zShapeBuffer);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::graph::RandomGenerator); e+= blockDim.x) {
                    cB[e] = dB[e];
                }
                __syncthreads();


                if (xEWS >= 1 && zEWS >= 1) {
                    for (Nd4jLong e = blockIdx.x * blockDim.x + threadIdx.x; e < length; e += blockDim.x * gridDim.x) {
                        z[e * zEWS] = OpClass::op(x[e * xEWS], e, length, buffer, extraArguments);
                    }
                } else {
                    
                    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x) {
                        
                        auto xOffset2 = shape::getIndexOffset(i, xShapeBuffer, length);
                        auto zOffset2 = shape::getIndexOffset(i, zShapeBuffer, length);

                        z[zOffset2] = OpClass::op(x[xOffset2], i, length, buffer, extraArguments);
                    }
                }

                __syncthreads();

                if (threadIdx.x == 0 && blockIdx.x == 0)
                    devBuffer->rewindH(length);
            }


            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

                auto z = reinterpret_cast<T*>(vz);
                auto extraArguments = reinterpret_cast<T*>(vextraArguments);

                Nd4jLong length = shape::length(zShapeBuffer);
                int ews = shape::elementWiseStride(zShapeBuffer);

                __shared__ nd4j::graph::RandomGenerator *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::graph::RandomGenerator *devBuffer;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::graph::RandomGenerator *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::graph::RandomGenerator *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::graph::RandomGenerator); e+= blockDim.x) {
                    cB[e] = dB[e];
                }
                __syncthreads();

                int tid = blockIdx.x * blockDim.x + threadIdx.x;

                if (ews >= 1) {
                    for (Nd4jLong x = tid; x < length; x += blockDim.x * gridDim.x) {
                        z[x * ews] = OpClass::op(x, length, buffer, extraArguments);
                    }
                } else {
                    
                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {                        
                        auto zOffset2 = shape::getIndexOffset(i, zShapeBuffer, length);
                        z[zOffset2] = OpClass::op(i, length, buffer,  extraArguments);
                    }
                }

                __syncthreads();

                if (threadIdx.x == 0 && blockIdx.x == 0)
                    devBuffer->rewindH(length);
            }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto z = reinterpret_cast<float*>(vz);
            auto extraArguments = reinterpret_cast<float*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, float, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            
            auto z = reinterpret_cast<float16*>(vz);
            auto extraArguments = reinterpret_cast<float16*>(vextraArguments);
            
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, float16, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<bfloat16>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto z = reinterpret_cast<bfloat16*>(vz);
            auto extraArguments = reinterpret_cast<bfloat16*>(vextraArguments);

            auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, bfloat16, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<double>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            
            auto z = reinterpret_cast<double*>(vz);
            auto extraArguments = reinterpret_cast<double*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, double, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            
            auto x = reinterpret_cast<float*>(vx);
            auto z = reinterpret_cast<float*>(vz);
            auto extraArguments = reinterpret_cast<float*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, float, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }


        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            
            auto x = reinterpret_cast<float16*>(vx);
            auto z = reinterpret_cast<float16*>(vz);
            auto extraArguments = reinterpret_cast<float16*>(vextraArguments);
            
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, float16, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<bfloat16>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<bfloat16*>(vx);
            auto z = reinterpret_cast<bfloat16*>(vz);
            auto extraArguments = reinterpret_cast<bfloat16*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, bfloat16, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<double>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            
            auto x = reinterpret_cast<double*>(vx);
            auto z = reinterpret_cast<double*>(vz);
            auto extraArguments = reinterpret_cast<double*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, double, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            

            auto x = reinterpret_cast<float*>(vx);
            auto y = reinterpret_cast<float*>(vy);
            auto z = reinterpret_cast<float*>(vz);
            auto extraArguments = reinterpret_cast<float*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, float, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {
            
            auto x = reinterpret_cast<float16*>(vx);
            auto y = reinterpret_cast<float16*>(vy);
            auto z = reinterpret_cast<float16*>(vz);
            auto extraArguments = reinterpret_cast<float16*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, float16, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<bfloat16>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<bfloat16*>(vx);
            auto y = reinterpret_cast<bfloat16*>(vy);
            auto z = reinterpret_cast<bfloat16*>(vz);
            auto extraArguments = reinterpret_cast<bfloat16*>(vextraArguments);

            auto stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, bfloat16, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }



        template <>
        _CUDA_H void RandomFunction<double>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<double*>(vx);
            auto y = reinterpret_cast<double*>(vy);
            auto z = reinterpret_cast<double*>(vz);
            auto extraArguments = reinterpret_cast<double*>(vextraArguments);

            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, double, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT RandomFunction, , FLOAT_TYPES);
    }
}
