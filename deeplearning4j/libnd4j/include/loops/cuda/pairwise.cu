#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//  @author raver119@gmail.com
//  @author Yurii Shyrma (iuriish@yahoo.com)

#ifndef PAIRWISE_CU
#define PAIRWISE_CU


#include "../pairwise_transform.h"


using namespace simdOps;

////////////////////////////////////////////////////////////////////////////////
template <typename X, typename Y, typename Z, typename OpType>
__global__ static void pairwiseSimpleShaped(void* vx, Nd4jLong *xShapeInfo, 
											void *vy, Nd4jLong *yShapeInfo, 
											void *vz, Nd4jLong *zShapeInfo, 
											void *vextraParams, 
											int *allocationBuffer) {
	
	auto x = reinterpret_cast<X*>(vx);
	auto y = reinterpret_cast<Y*>(vy);
	auto z = reinterpret_cast<Z*>(vz);
	auto extraParams = reinterpret_cast<Z*>(vextraParams);

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	Nd4jLong len = shape::length(xShapeInfo);

	auto xEws = shape::elementWiseStride(xShapeInfo);
	auto yEws = shape::elementWiseStride(yShapeInfo);
	auto zEws = shape::elementWiseStride(zShapeInfo);

	auto xOrder = shape::order(xShapeInfo);
	auto yOrder = shape::order(yShapeInfo);
	auto zOrder = shape::order(zShapeInfo);


	if (xEws >= 1 && yEws >= 1 && zEws >= 1 && xOrder == yOrder && xOrder == zOrder) {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
			z[i * zEws] = OpType::op(x[i * xEws], y[i * yEws], extraParams);
		}
	} else if (vx == vz) {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
			auto xOffset = shape::getIndexOffset(i, xShapeInfo, len);
			auto yOffset = shape::getIndexOffset(i, yShapeInfo, len);
				
			z[xOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
		}
	} else {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
			auto xOffset = shape::getIndexOffset(i, xShapeInfo, len);
			auto yOffset = shape::getIndexOffset(i, yShapeInfo, len);
			auto zOffset = shape::getIndexOffset(i, zShapeInfo, len);

			z[zOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
		}
	}
}

namespace functions           {
namespace pairwise_transforms {

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y, typename Z>
template<typename OpType>
void _CUDA_H PairWiseTransform<X,Y,Z>::intermediateShaped(dim3& launchDims, hipStream_t *stream, 
														void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo,
														void *vy, Nd4jLong *yShapeInfo, Nd4jLong *hyShapeInfo,
														void *vz, Nd4jLong *zShapeInfo, Nd4jLong *hzShapeInfo,
														void *vextraParams,
														int *allocPointer){

	pairwiseSimpleShaped<X, Y, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams, allocPointer);
}

////////////////////////////////////////////////////////////////////////////////
template<typename X, typename Y, typename Z>
void PairWiseTransform<X,Y,Z>::executeCudaShaped(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo, void *vy, Nd4jLong *yShapeInfo, Nd4jLong *hyShapeInfo, void *vz, Nd4jLong *zShapeInfo, Nd4jLong *hzShapeInfo, void *vextraParams) {
	DISPATCH_BY_OPNUM_TTT(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, hxShapeInfo, vy, yShapeInfo, hyShapeInfo, vz, zShapeInfo, hzShapeInfo, vextraParams, nullptr), PAIRWISE_TRANSFORM_OPS);
}


    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_0);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_1);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_2);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_3);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_4);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_5);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_6);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_7);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_8);
    BUILD_PAIRWISE_TEMPLATE(template class ND4J_EXPORT PairWiseTransform, , PAIRWISE_TYPES_9);
}
}

#endif // PAIRWISE_CU