#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018
// @author raver119@gmail.com
//

#include "../scalar_bool.h"
#include <op_boilerplate.h>
#include <types/types.h>

#include "../legacy_ops.h"

using namespace simdOps;

////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpType>
__global__ void scalarAlongDimension(void *x, Nd4jLong *xShapeInfo,
                                    void *extraParams,
                                    void *z, Nd4jLong *zShapeInfo,
                                    void *scalars,
                                    int *dimension, int dimensionLength,
                                    Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                    Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
        
    functions::scalar::ScalarBoolTransform<X,Z>::template transformCuda<OpType>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}


////////////////////////////////////////////////////////////////////////
template <typename X, typename Z, typename OpType>
__global__ void scalarSimpleShaped(void* x, void *y, Nd4jLong *xShapeInfo, void *params, void *z, Nd4jLong *zShapeInfo, int *allocationBuffer) {

    functions::scalar::ScalarBoolTransform<X,Z>::template transformCuda<OpType>(y, x, xShapeInfo, params, z, zShapeInfo, allocationBuffer);
}





// *********************************************************************//
// *********************************************************************//
namespace functions {
namespace scalar    {

////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void  ScalarBoolTransform<X, Z>::transformCuda(void* vscalar, 
                                                        void *vy, Nd4jLong *yShapeInfo, 
                                                        void *vparams, 
                                                        void *vz, Nd4jLong *zShapeInfo, 
                                                        int *allocationBuffer) {
    auto scalar = reinterpret_cast<X*>(vscalar)[0];
    auto y      = reinterpret_cast<X*>(vy);
    auto params = reinterpret_cast<X*>(vparams);
    auto z      = reinterpret_cast<Z*>(vz);

    auto yRank   = shape::rank(yShapeInfo);
    auto yEWS    = shape::elementWiseStride(yShapeInfo);
    auto yShape  = shape::shapeOf(yShapeInfo);
    auto yStride = shape::stride(yShapeInfo);        
    
    auto zRank   = shape::rank(zShapeInfo);
    auto zEWS    = shape::elementWiseStride(zShapeInfo);
    auto zShape  = shape::shapeOf(zShapeInfo);
    auto zStride = shape::stride(zShapeInfo);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int len;
    if(threadIdx.x == 0)
        len = shape::length(yShapeInfo);
    __syncthreads();

    if(yEWS >= 1 && zEWS >= 1 && shape::order(yShapeInfo) == shape::order(zShapeInfo)) {
            transformCuda<OpType>(len, vscalar, vy, yEWS, vparams, vz, zEWS, allocationBuffer);    
    }
    else {
        for (Nd4jLong i = tid; i < len; i+= totalThreads)                        
            z[shape::getIndexOffset(i, zShapeInfo, len)] = OpType::op(y[shape::getIndexOffset(i, yShapeInfo, len)], scalar, params);
    }
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void  ScalarBoolTransform<X, Z>::transformCuda(Nd4jLong len, 
                                                          void* vx, 
                                                          void *vy, Nd4jLong yEWS, 
                                                          void *vparams, 
                                                          void *vz, Nd4jLong zEWS, 
                                                          int *allocationBuffer) {

    auto x = reinterpret_cast<X*>(vx)[0];
    auto y = reinterpret_cast<X*>(vy);
    auto z = reinterpret_cast<Z*>(vz);
    auto params = reinterpret_cast<X*>(vparams);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    Nd4jLong i = tid;
    if(yEWS == 1 && zEWS == 1) {
        for (; i < len; i += totalThreads)
            z[i] = OpType::op(y[i], x, params);
    }
    else {
        for (; i < len; i += totalThreads)
            z[i * zEWS] = OpType::op(y[i * yEWS], x, params);
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
__device__ void  ScalarBoolTransform<X, Z>::transformCuda(void *vx, Nd4jLong *xShapeInfo, 
                                                        void *vextraParams, 
                                                        void *vz, Nd4jLong *zShapeInfo, 
                                                        void *vscalars, 
                                                        int *dimension, int dimensionLength, 
                                                        Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, 
                                                        Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
    auto x = reinterpret_cast<X*>(vx);
    auto scalars = reinterpret_cast<X*>(vscalars);
    auto z = reinterpret_cast<Z*>(vz);
    auto extraParams = reinterpret_cast<X*>(vextraParams);
    
    if (tadShapeInfoZ == nullptr) {
        tadShapeInfoZ = tadShapeInfo;
        tadOffsetsZ = tadOffsets;
    }

    // tad preparation
    auto tadEWS = shape::elementWiseStride(tadShapeInfo);
    auto zEWS = shape::elementWiseStride(tadShapeInfo);
    auto tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
    auto numTads =shape::length(xShapeInfo) / tadLength;

    // main loop, rolling over tads
    for (int r = blockIdx.x; r < numTads; r+=gridDim.x) {
        auto offset = tadOffsets[r];
        auto offsetZ = tadOffsetsZ[r];
        X scalar = scalars[r];

        if (tadEWS >= 1 && zEWS >= 1) {
            Z *oZ = z + offsetZ;
            X *oX = x + offset;

            for (int f = threadIdx.x; f < tadLength; f+= blockDim.x)
                oZ[f] = OpType::op(oX[f], scalar, extraParams);
        } 
        else        
            printf("Super-bad loop visited. Shouldn't ever happen\n");
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template <typename OpType>
_CUDA_H void ScalarBoolTransform<X, Z>::intermediateAlongDimension(dim3& launchDims, hipStream_t *stream,
                                                                void *x, Nd4jLong *xShapeInfo, 
                                                                void *z, Nd4jLong *zShapeInfo, 
                                                                void *scalars, 
                                                                void *extraParams, 
                                                                int *dimension, int dimensionLength, 
                                                                Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, 
                                                                Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    scalarAlongDimension<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
template<typename OpType>
void _CUDA_H ScalarBoolTransform<X,Z>::intermediateShaped(dim3& launchDims, hipStream_t *stream,
                                                            void *vx, Nd4jLong *xShapeInfo, 
                                                            void *vz, Nd4jLong *zShapeInfo, 
                                                            void* vscalar, 
                                                            void *vextraParams, int *allocPointer){
    
    scalarSimpleShaped<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vscalar, xShapeInfo, vextraParams, vz, zShapeInfo, allocPointer);    
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
void ScalarBoolTransform<X,Y>::executeCudaShaped(dim3& launchDims, hipStream_t *stream,
                                                int opNum, 
                                                void *vx, Nd4jLong *xShapeInfo, 
                                                void *vz, Nd4jLong *zShapeInfo, 
                                                void* vscalar, 
                                                void *vextraParams) {

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("H14 opNum:[%i]\n", opNum);

    DISPATCH_BY_OPNUM_TT(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalar, vextraParams, nullptr), SCALAR_BOOL_OPS);
}

////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
void ScalarBoolTransform<X,Y>::executeCudaAlongDimension(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *vscalars, void *vextraParams, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
    DISPATCH_BY_OPNUM_TT(intermediateAlongDimension, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalars, vextraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), SCALAR_BOOL_OPS);
}

    BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT ScalarBoolTransform, , LIBND4J_TYPES, BOOL_TYPES);
}
}

