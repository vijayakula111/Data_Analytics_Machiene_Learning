#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//


#include <pointercast.h>
#include <types/types.h>
#include <types/float16.h>
#include <op_boilerplate.h>
#include <loops/summarystatsreduce.h>
#include <helpers/shape.h>
#include <helpers/TAD.h>
#include <dll.h>
#include <Environment.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>
#include <helpers/DebugHelper.h>

using namespace simdOps;

namespace functions {
    namespace summarystats {

template <typename X, typename Z>
void _CUDA_G summaryStatsReduceT(int op, void *dx, Nd4jLong *xShapeInfo, int xRank, void *extraParams, void *result, Nd4jLong *resultShapeInfo, int zRank, int *dimension, int dimensionLength, int postProcessOrNot,bool biasCorrected,int *allocationBuffer, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {
            
    functions::summarystats::SummaryStatsReduce<X,Z>::transform(op,dx,xShapeInfo,extraParams,result,resultShapeInfo,dimension,dimensionLength,biasCorrected,allocationBuffer,reductionBuffer,tadOnlyShapeInfo,tadOffsets);
}

        /**
		 *
		 * @param sPartialsRef
		 * @param tid
		 * @param extraParams
		 */
        template<typename X, typename Z>
        template<typename OpType>
        _CUDA_D void SummaryStatsReduce<X,Z>::aggregatePartials(SummaryStatsData<X> **sPartialsRef, Nd4jLong tid, Nd4jLong numElements, void *vextraParams) {
            // start the shared memory loop on the next power of 2 less
            // than the block size.  If block size is not a power of 2,
            // accumulate the intermediate sums in the remainder range.
            auto extraParams = static_cast<Z*>(vextraParams);
            SummaryStatsData<X> *sPartials = *sPartialsRef;
            Nd4jLong floorPow2 = blockDim.x;

            if (floorPow2 & (floorPow2 - 1)) {
                while (floorPow2 & (floorPow2 - 1)) {
                    floorPow2 &= floorPow2 - 1;
                }

                if (tid >= floorPow2) {
                    SummaryStatsData<X> prev = sPartials[tid - floorPow2];
                    SummaryStatsData<X> curr = sPartials[tid];
                    sPartials[tid - floorPow2] = update(prev, curr, extraParams);
                }
                __syncthreads();
            }

            for (Nd4jLong activeThreads = floorPow2 >> 1; activeThreads; activeThreads >>= 1) {
                if (tid < activeThreads && tid + activeThreads < numElements) {
                    SummaryStatsData<X> curr = sPartials[tid];
                    SummaryStatsData<X> next = sPartials[tid + activeThreads];
                    sPartials[tid] = update(curr, next, extraParams);
                }
                __syncthreads();
            }
        };

        /**
			 * @param n n is the number of
			 *        elements to loop through
			 * @param dx the data to operate on
			 * @param xVectorInfo the meta data for the vector:
			 *                              0 is the offset
			 *                              1 is the increment/stride
			 *                              2 is the real length of the buffer (n and dx.length won't always be the same)
			 *                              3 is the element wise stride for the buffer
			 *                              4 is the number of elements it takes to get to the next row/column/tensor
			 * @param gpuInformation
			 *                              0 is the block size
			 *                              1 is the grid size
			 *                              2 is the shared memory size
			 * @param problemDefinition
			 *                          0 is the number of elements per vector
			 *                          1 is the number of vectors
			 */
        template<typename X, typename Z>
        template<typename OpType>
        _CUDA_D void SummaryStatsReduce<X,Z>::transform(void *vdx, Nd4jLong *xShapeInfo, void *vextraParams, void *vresult, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, int postProcessOrNot, int *allocationBuffer, void *vreductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {

            /**
             * Gpu information for the problem
             */
            auto dx = static_cast<X*>(vdx);
            auto result = static_cast<Z*>(vresult);
            auto extraParams = static_cast<Z*>(vextraParams);
            auto reductionBuffer = static_cast<Z*>(vreductionBuffer);

            int tid = blockIdx.x * blockDim.x + threadIdx.x;
            __shared__ volatile int resultScalar;

            __shared__ int xElementWiseStride;

            int numElements = blockDim.x;
            //shared memory space for storing intermediate results
            SummaryStatsData<X> *sPartials;
            if(threadIdx.x == 0) {
                extern __shared__ unsigned char shmem[];
                sPartials = reinterpret_cast<SummaryStatsData<X>*>(shmem);
            }
            __syncthreads();

            Z startingVal = startingValue(dx);

            SummaryStatsData<X> val;
            val.initWithValue(startingVal);
            val.n = 0;
            sPartials[threadIdx.x] = val;


            //length for the tad
            __shared__ volatile int xLength;

            __shared__ volatile int resultLength;


            SummaryStatsData<X> reduction;
            reduction.initWithValue(0.0);
            reduction.n = 0;
            if (threadIdx.x == 0) {
                if (resultShapeInfo != nullptr)
                    resultLength = shape::length(resultShapeInfo);
                else resultLength = 1;

                if (dimensionLength == 1) {
                    if (dimension == nullptr || dimension[0] == MAX_DIMENSION)
                        resultScalar = 1;
                    else
                        resultScalar = 0;
                }
                else
                    resultScalar = 0;

                if (resultLength == 1)
                    resultScalar = 1;

                auto xStride = shape::stride(xShapeInfo);
                auto xOrder = shape::order(xShapeInfo);

                if (dimension != nullptr && (dimension[0] != MAX_DIMENSION && dimensionLength == 1)) {
                    xElementWiseStride = xStride[dimension[0]];
                }
                else {
                    xElementWiseStride = shape::elementWiseStride(xShapeInfo);
                }


                xLength = shape::length(xShapeInfo);


            }
            __syncthreads();
            if (!resultScalar) {

                __shared__ int tadLength;
                __shared__ int tadEWS;
                __shared__ int numTads;

                if (threadIdx.x == 0) {
                    tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
                    tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                    numTads = shape::length(xShapeInfo) / tadLength;
                }
                __syncthreads();

                if (dimensionLength > 1) {                    

                    for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
                        auto tadOffsetForBlock = tadOffsets[r];

                        val.initWithValue(startingVal);
                        val.n = 0;
                        sPartials[threadIdx.x] = val;

                        for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {                            

                            auto xOffset = tadOffsetForBlock + shape::getIndexOffset(i, tadOnlyShapeInfo, tadLength);
                            SummaryStatsData<X> indexVal2;
                            indexVal2.initWithValue(dx[xOffset]);

                            sPartials[threadIdx.x] = update(sPartials[threadIdx.x], OpType::op(indexVal2, extraParams), extraParams);
                        }
                        __syncthreads();
                        aggregatePartials<OpType>(&sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, tadLength), extraParams);

                        __syncthreads();
                        if (threadIdx.x == 0) {
                            result[r] = OpType::getValue(postProcessOrNot, sPartials[threadIdx.x]);
                        }

                    }
                }
                else {


                    for (int i = blockIdx.x; i < numTads; i += gridDim.x) {
                        auto tadOffsetForBlock = tadOffsets[i];

                        val.initWithValue(startingVal);
                        val.n = 0;
                        sPartials[threadIdx.x] = val;

                        auto indexX = tadOffsetForBlock + (xElementWiseStride * threadIdx.x);

                        if (threadIdx.x < tadLength) {
                            SummaryStatsData<X> indexVal;
                            indexVal.initWithValue(dx[indexX]);
                            sPartials[threadIdx.x] = OpType::op(indexVal, extraParams);
                        }

                        for (int x = threadIdx.x + blockDim.x; x < tadLength; x += blockDim.x) {
                            indexX = tadOffsetForBlock + x * tadEWS;
                            SummaryStatsData<X> indexVal2;
                            indexVal2.initWithValue(dx[indexX]);
                            sPartials[threadIdx.x] = update(sPartials[threadIdx.x], OpType::op(indexVal2, extraParams), extraParams);
                        }

                        __syncthreads();
                        aggregatePartials<OpType>(&sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, tadLength), extraParams);

                        __syncthreads();
                        if (threadIdx.x == 0) {
                            result[i] = OpType::getValue(postProcessOrNot, sPartials[threadIdx.x]); //postProcess(sPartials[0],tadLength ,extraParams);
                        }
                    }
                }
            }
            else if (resultScalar) {
                __shared__ int n;
                if (threadIdx.x == 0) {
                    xElementWiseStride = shape::elementWiseStride(xShapeInfo);
                    n = shape::length(xShapeInfo);
                }
                __syncthreads();

                if (xElementWiseStride >= 1) {
                    for (Nd4jLong i = tid; i < n; i += (blockDim.x * gridDim.x)) {
                        SummaryStatsData<X> indexVal2;
                        indexVal2.initWithValue(dx[i * xElementWiseStride]);
                        reduction = update(reduction, indexVal2, extraParams);
                    }
                }
                else {

                    for (Nd4jLong i = tid; i < n; i += blockDim.x * gridDim.x) {
                        
                        auto offset = shape::getIndexOffset(i, xShapeInfo, n);                        
                        SummaryStatsData<X> indexVal2;
                        indexVal2.initWithValue(dx[offset]);
                        reduction = update(reduction, indexVal2, extraParams);
                    }
                }
                sPartials[threadIdx.x] = reduction;

                __syncthreads();
                aggregatePartials<OpType>(&sPartials, threadIdx.x, blockDim.x, extraParams);
                __syncthreads();

                if (gridDim.x > 1) {
                    __shared__ bool amLast;
                    unsigned int *tc = (unsigned int *)reductionBuffer;                    
                    tid = threadIdx.x;
                    if (threadIdx.x == 0) {
                        SummaryStatsData<X> *pBuffer = (SummaryStatsData<X>*) reductionBuffer;
                        pBuffer[blockIdx.x] = sPartials[0];
                    }
                    __syncthreads();
                    __threadfence();

                    if (tid == 0) {
                        unsigned int ticket = atomicInc(&tc[16384], gridDim.x);
                        amLast = (ticket == gridDim.x - 1);
                    }

                    __syncthreads();

                    if (amLast) {
                        tc[16384] = 0;
                        SummaryStatsData<X>* pBuffer = (SummaryStatsData<X>*) reductionBuffer;

                        Z startingVal = startingValue(dx);

                        SummaryStatsData<X> val;
                        val.initWithValue(startingVal);
                        val.n = 0;
                        sPartials[threadIdx.x] = val;

                        for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) {
                            sPartials[threadIdx.x] = update(sPartials[threadIdx.x], pBuffer[i], extraParams);
                        }

                        __syncthreads();
                        aggregatePartials<OpType>(&sPartials, threadIdx.x, gridDim.x, extraParams);
                        __syncthreads();

                        if (tid == 0) {
                            result[0] = OpType::getValue(postProcessOrNot, sPartials[0]);
                        }
                    }
                }
                else {
                    if (tid == 0) {
                        unsigned int *tc = (unsigned *)reductionBuffer;
                        tc[16384] = 0;
                        result[0] = result[0] = OpType::getValue(postProcessOrNot, sPartials[0]);
                    }
                }
            }
        };


        template <typename X, typename Y>
        _CUDA_D void SummaryStatsReduce<X,Y>::transform(const int opNum, void *dx, Nd4jLong *xShapeInfo, void *extraParams, void *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, int postProcessOrNot, int *allocationBuffer, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {
            DISPATCH_BY_OPNUM_TT(transform, PARAMS(dx, xShapeInfo, extraParams, result, resultShapeInfo, dimension, dimensionLength, postProcessOrNot, allocationBuffer, reductionBuffer, tadOnlyShapeInfo, tadOffsets), SUMMARY_STATS_OPS);
        };


        template <typename X, typename Z>
        _CUDA_H void SummaryStatsReduce<X,Z>::execSummaryStatsReduceScalar(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo, void *vextraParams, void *vz, Nd4jLong *zShapeInfo, Nd4jLong *hzShapeInfo, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, bool biasCorrected, void *reductionBuffer) {
            
            auto x = static_cast<X*>(vx);
            auto extraParams = static_cast<Z*>(vextraParams);                                        
            auto z = reinterpret_cast<Z*>(vz);
            auto reductionPointerA = reinterpret_cast<Z*>(reductionBuffer);

            if (nd4j::Environment::getInstance()->isDebugAndVerbose())
                printf("D16 opNum:[%i]\n", opNum);

            summaryStatsReduceT<X,Z><<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
                            opNum,
                            x,
                            xShapeInfo, shape::rank(hxShapeInfo),
                            extraParams,
                            z,
                            zShapeInfo, shape::rank(hzShapeInfo),
                            nullptr,
                            1,
                            1,biasCorrected, nullptr, reductionPointerA, tadShapeInfo, tadOffsets);

            // this is blocking method since method should return scalar
            nd4j::DebugHelper::checkErrorCode(stream, "execSSReduceScalar(...) failed");
        }

        template <typename X, typename Z>
        _CUDA_H void SummaryStatsReduce<X,Z>::execSummaryStatsReduce(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo, void *vextraParams, void *vresult, Nd4jLong *resultShapeInfo, Nd4jLong *hzShapeInfo, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, bool biasCorrected, void *reductionBuffer) {

            auto x = static_cast<X*>(vx);
            auto result = static_cast<Z*>(vresult);
            auto extraParams = static_cast<Z*>(vextraParams);

            if (nd4j::Environment::getInstance()->isDebugAndVerbose())
                printf("F17 opNum:[%i]\n", opNum);

            auto reductionPointerA = reinterpret_cast<Z*>(reductionBuffer);

            summaryStatsReduceT<X,Z><<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
                    opNum,
                            x,
                            xShapeInfo, shape::rank(hxShapeInfo),
                            extraParams,
                            result,
                            resultShapeInfo, shape::rank(hzShapeInfo),
                            nullptr,
                            1,
                            1,biasCorrected, nullptr, reductionPointerA, tadShapeInfo, tadOffsets);

            DEBUG_KERNEL(stream, opNum);
        }


        template<typename X, typename Z>
        _CUDA_H void SummaryStatsReduce<X,Z>::execSummaryStatsReduce(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, Nd4jLong *hxShapeInfo, void *vextraParams, void *vresult, Nd4jLong *resultShapeInfo, Nd4jLong *hzShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, bool biasCorrected, void *reductionBuffer) {

            auto x = static_cast<X*>(vx);
            auto result = static_cast<Z*>(vresult);
            auto extraParams = static_cast<Z*>(vextraParams);

            if (nd4j::Environment::getInstance()->isDebugAndVerbose())
                printf("D18 opNum:[%i]\n", opNum);

            summaryStatsReduceT<X, Z><<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
                    opNum,
                            x,
                            xShapeInfo, shape::rank(hxShapeInfo),
                            extraParams,
                            result,
                            resultShapeInfo, shape::rank(hzShapeInfo),
                            dimension,
                            dimensionLength,
                            1, biasCorrected, nullptr, reinterpret_cast<Z*>(reductionBuffer), tadShapeInfo, tadOffsets);

            DEBUG_KERNEL(stream, opNum);
        }

        BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT SummaryStatsReduce, , LIBND4J_TYPES, FLOAT_TYPES);
    }
}